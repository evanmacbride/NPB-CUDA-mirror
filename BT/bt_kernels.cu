#include "hip/hip_runtime.h"
#include <stdio.h>
#include "main.h"

#define aa 0
#define bb 1
#define cc 2

namespace gpu_mod {
__constant__ double tx1, tx2, tx3, ty1, ty2, ty3, tz1, tz2, tz3;
__constant__ double dt, dtdssp;
__constant__ double xxcon1, xxcon2, xxcon3, xxcon4, xxcon5, dx1tx1, dx2tx1, dx3tx1, dx4tx1, dx5tx1;
__constant__ double yycon1, yycon2, yycon3, yycon4, yycon5, dy1ty1, dy2ty1, dy3ty1, dy4ty1, dy5ty1;
__constant__ double zzcon1, zzcon2, zzcon3, zzcon4, zzcon5, dz1tz1, dz2tz1, dz3tz1, dz4tz1, dz5tz1;
__constant__ double dnxm1, dnym1, dnzm1;
__constant__ double dttx1, dttx2, dtty1, dtty2, dttz1, dttz2, c2dttx1, c2dtty1, c2dttz1;
__constant__ double comz1, comz4, comz5, comz6, c3c4tx3, c3c4ty3, c3c4tz3;
__constant__ double ce[13][5];
}

static void inline HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define START_TIMER(timer) if (timeron) { HANDLE_ERROR(hipDeviceSynchronize()); timers->timer_start(timer); }
#define STOP_TIMER(timer) if (timeron) { HANDLE_ERROR(hipDeviceSynchronize()); timers->timer_stop(timer); }

void BT::adi (bool singlestep) {
	for (int i = 0; i < t_last; i++) timers->timer_clear(i);
	HANDLE_ERROR(hipDeviceSynchronize());
	timers->timer_start(0);

	int itmax = singlestep ? 1 : niter;
	for (int step = 1; step <= itmax; step++) {
		if (step % 20 == 0 || step == 1 && !singlestep)
			printf(" Time step %4d\n", step);

		compute_rhs();
		x_solve();
		y_solve();
		z_solve();
		add();
	}

	HANDLE_ERROR(hipDeviceSynchronize());
	timers->timer_stop(0);
	tmax = timers->timer_read(0);
}

//---------------------------------------------------------------------
//     addition of update to the vector u
//---------------------------------------------------------------------
__global__ static void add_kernel (double *u, const double *rhs, const int nx, const int ny, const int nz) {
	int i, j, k, m;

	k = blockIdx.y+1;
	j = blockIdx.x+1;
	i = threadIdx.x+1;
	m = threadIdx.y;

	u(m,i,j,k) += rhs(m,i,j,k);
}

void BT::add () {
	dim3 grid(ny-2,nz-2);
	dim3 block(nx-2,5);

	START_TIMER(t_add);
	add_kernel<<<grid,block>>>(u, rhs, nx, ny, nz);
	STOP_TIMER(t_add);
}

//---------------------------------------------------------------------
//      compute the reciprocal of density, and the kinetic energy
//---------------------------------------------------------------------
__global__ static void compute_rhs_kernel_1 (double *rho_i, double *us, double *vs, double *ws, double *qs, double *square, double *rhs, const double *forcing, const double *u, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	k = blockIdx.y;
	j = blockIdx.x;
	i = threadIdx.x;

	double rho_inv = 1.0/u(0,i,j,k);
	rho_i(i,j,k) = rho_inv;
	us(i,j,k) = u(1,i,j,k) * rho_inv;
	vs(i,j,k) = u(2,i,j,k) * rho_inv;
	ws(i,j,k) = u(3,i,j,k) * rho_inv;
	square(i,j,k) = 0.5*(u(1,i,j,k)*u(1,i,j,k) + u(2,i,j,k)*u(2,i,j,k) + u(3,i,j,k)*u(3,i,j,k)) * rho_inv;
	qs(i,j,k) = square(i,j,k) * rho_inv;

	//---------------------------------------------------------------------
	// copy the exact forcing term to the right hand side;  because 
	// this forcing term is known, we can store it on the whole zone
	// including the boundary                   
	//---------------------------------------------------------------------
	for (m = 0; m < 5; m++) rhs(m,i,j,k) = forcing(m,i,j,k);
}

__global__ static void compute_rhs_kernel_2 (const double *rho_i, const double *us, const double *vs, const double *ws, const double *qs, const double *square, double *rhs, const double *u, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double rtmp[5];
	k = blockIdx.y+1;
	j = blockIdx.x+1;
	i = threadIdx.x+1;
	
	using namespace gpu_mod;

	//---------------------------------------------------------------------
	//      compute xi-direction fluxes 
	//---------------------------------------------------------------------
	double uijk = us(i,j,k);
	double up1 = us(i+1,j,k);
	double um1 = us(i-1,j,k);
				
	rtmp[0] = rhs(0,i,j,k) + dx1tx1*(u(0,i+1,j,k) - 2.0*u(0,i,j,k) + u(0,i-1,j,k)) - tx2*(u(1,i+1,j,k)-u(1,i-1,j,k));
	rtmp[1] = rhs(1,i,j,k) + dx2tx1*(u(1,i+1,j,k) - 2.0*u(1,i,j,k) + u(1,i-1,j,k)) + xxcon2*con43*(up1-2.0*uijk+um1) - tx2*(u(1,i+1,j,k)*up1 - u(1,i-1,j,k)*um1 + (u(4,i+1,j,k)-square(i+1,j,k)-u(4,i-1,j,k)+square(i-1,j,k))*c2);
	rtmp[2] = rhs(2,i,j,k) + dx3tx1*(u(2,i+1,j,k) - 2.0*u(2,i,j,k) + u(2,i-1,j,k)) + xxcon2*(vs(i+1,j,k)-2.0*vs(i,j,k)+vs(i-1,j,k)) - tx2*(u(2,i+1,j,k)*up1 - u(2,i-1,j,k)*um1);
	rtmp[3] = rhs(3,i,j,k) + dx4tx1*(u(3,i+1,j,k) - 2.0*u(3,i,j,k) + u(3,i-1,j,k)) + xxcon2*(ws(i+1,j,k)-2.0*ws(i,j,k)+ws(i-1,j,k)) - tx2*(u(3,i+1,j,k)*up1 - u(3,i-1,j,k)*um1);
	rtmp[4] = rhs(4,i,j,k) + dx5tx1*(u(4,i+1,j,k) - 2.0*u(4,i,j,k) + u(4,i-1,j,k)) + xxcon3*(qs(i+1,j,k)-2.0*qs(i,j,k)+qs(i-1,j,k))+ xxcon4*(up1*up1-2.0*uijk*uijk+um1*um1) +
				xxcon5*(u(4,i+1,j,k)*rho_i(i+1,j,k) - 2.0*u(4,i,j,k)*rho_i(i,j,k) + u(4,i-1,j,k)*rho_i(i-1,j,k)) - tx2*((c1*u(4,i+1,j,k) - c2*square(i+1,j,k))*up1 - (c1*u(4,i-1,j,k) - c2*square(i-1,j,k))*um1 );
	//---------------------------------------------------------------------
	//      add fourth order xi-direction dissipation               
	//---------------------------------------------------------------------
	if (i == 1) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (5.0*u(m,i,j,k)-4.0*u(m,i+1,j,k)+u(m,i+2,j,k));
	} else if (i == 2) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (-4.0*u(m,i-1,j,k)+6.0*u(m,i,j,k)-4.0*u(m,i+1,j,k)+u(m,i+2,j,k));
	} else if (i >= 3 && i < nx-3) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * ( u(m,i-2,j,k)-4.0*u(m,i-1,j,k)+6.0*u(m,i,j,k)-4.0*u(m,i+1,j,k)+u(m,i+2,j,k));
	} else if (i == nx-3) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (u(m,i-2,j,k)-4.0*u(m,i-1,j,k)+6.0*u(m,i,j,k)-4.0*u(m,i+1,j,k) );
	} else if (i == nx-2) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (u(m,i-2,j,k)-4.0*u(m,i-1,j,k) + 5.0*u(m,i,j,k));
	}

	//---------------------------------------------------------------------
	//      compute eta-direction fluxes 
	//---------------------------------------------------------------------
	double vijk = vs(i,j,k);
	double vp1 = vs(i,j+1,k);
	double vm1 = vs(i,j-1,k);
	rtmp[0] = rtmp[0] + dy1ty1*(u(0,i,j+1,k) - 2.0*u(0,i,j,k) + u(0,i,j-1,k)) - ty2*(u(2,i,j+1,k)-u(2,i,j-1,k));
	rtmp[1] = rtmp[1] + dy2ty1*(u(1,i,j+1,k) - 2.0*u(1,i,j,k) + u(1,i,j-1,k)) + yycon2*(us(i,j+1,k)-2.0*us(i,j,k)+us(i,j-1,k)) - ty2*(u(1,i,j+1,k)*vp1-u(1,i,j-1,k)*vm1);
	rtmp[2] = rtmp[2] + dy3ty1*(u(2,i,j+1,k) - 2.0*u(2,i,j,k) + u(2,i,j-1,k)) + yycon2*con43*(vp1-2.0*vijk+vm1) - ty2*(u(2,i,j+1,k)*vp1-u(2,i,j-1,k)*vm1+(u(4,i,j+1,k)-square(i,j+1,k)-u(4,i,j-1,k)+square(i,j-1,k))*c2);
	rtmp[3] = rtmp[3] + dy4ty1*(u(3,i,j+1,k) - 2.0*u(3,i,j,k) + u(3,i,j-1,k)) + yycon2*(ws(i,j+1,k)-2.0*ws(i,j,k)+ws(i,j-1,k))-ty2*(u(3,i,j+1,k)*vp1-u(3,i,j-1,k)*vm1);
	rtmp[4] = rtmp[4] + dy5ty1*(u(4,i,j+1,k) - 2.0*u(4,i,j,k) + u(4,i,j-1,k)) + yycon3*(qs(i,j+1,k)-2.0*qs(i,j,k)+qs(i,j-1,k)) + yycon4*(vp1*vp1-2.0*vijk*vijk+vm1*vm1) +
				yycon5*(u(4,i,j+1,k)*rho_i(i,j+1,k)-2.0*u(4,i,j,k)*rho_i(i,j,k)+u(4,i,j-1,k)*rho_i(i,j-1,k)) - ty2*((c1*u(4,i,j+1,k)-c2*square(i,j+1,k))*vp1 - (c1*u(4,i,j-1,k)-c2*square(i,j-1,k))*vm1);
	//---------------------------------------------------------------------
	//      add fourth order eta-direction dissipation         
	//---------------------------------------------------------------------
	if (j == 1) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(5.0*u(m,i,j,k)-4.0*u(m,i,j+1,k)+u(m,i,j+2,k));
	} else if (j == 2) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(-4.0*u(m,i,j-1,k)+6.0*u(m,i,j,k)-4.0*u(m,i,j+1,k)+u(m,i,j+2,k));
	} else if (j >= 3 && j < ny-3) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j-2,k)-4.0*u(m,i,j-1,k)+6.0*u(m,i,j,k)-4.0*u(m,i,j+1,k)+u(m,i,j+2,k));
	} else if (j == ny-3) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j-2,k)-4.0*u(m,i,j-1,k)+6.0*u(m,i,j,k)-4.0*u(m,i,j+1,k));
	} else if (j == ny-2) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j-2,k)-4.0*u(m,i,j-1,k)+5.0*u(m,i,j,k));
	}

	//---------------------------------------------------------------------
	//      compute zeta-direction fluxes 
	//---------------------------------------------------------------------
	double wijk = ws(i,j,k);
	double wp1 = ws(i,j,k+1);
	double wm1 = ws(i,j,k-1);

	rtmp[0] = rtmp[0] + dz1tz1*(u(0,i,j,k+1)-2.0*u(0,i,j,k)+u(0,i,j,k-1)) - tz2*(u(3,i,j,k+1)-u(3,i,j,k-1));
	rtmp[1] = rtmp[1] + dz2tz1*(u(1,i,j,k+1)-2.0*u(1,i,j,k)+u(1,i,j,k-1)) + zzcon2*(us(i,j,k+1)-2.0*us(i,j,k)+us(i,j,k-1)) - tz2*(u(1,i,j,k+1)*wp1-u(1,i,j,k-1)*wm1);
	rtmp[2] = rtmp[2] + dz3tz1*(u(2,i,j,k+1)-2.0*u(2,i,j,k)+u(2,i,j,k-1)) + zzcon2*(vs(i,j,k+1)-2.0*vs(i,j,k)+vs(i,j,k-1)) - tz2*(u(2,i,j,k+1)*wp1-u(2,i,j,k-1)*wm1);
	rtmp[3] = rtmp[3] + dz4tz1*(u(3,i,j,k+1)-2.0*u(3,i,j,k)+u(3,i,j,k-1)) + zzcon2*con43*(wp1-2.0*wijk+wm1) - tz2*(u(3,i,j,k+1)*wp1-u(3,i,j,k-1)*wm1+(u(4,i,j,k+1)-square(i,j,k+1)-u(4,i,j,k-1)+square(i,j,k-1))*c2);
	rtmp[4] = rtmp[4] + dz5tz1*(u(4,i,j,k+1)-2.0*u(4,i,j,k)+u(4,i,j,k-1)) + zzcon3*(qs(i,j,k+1)-2.0*qs(i,j,k)+qs(i,j,k-1)) + zzcon4*(wp1*wp1-2.0*wijk*wijk+wm1*wm1) +
				zzcon5*(u(4,i,j,k+1)*rho_i(i,j,k+1)-2.0*u(4,i,j,k)*rho_i(i,j,k)+u(4,i,j,k-1)*rho_i(i,j,k-1)) - tz2*((c1*u(4,i,j,k+1)-c2*square(i,j,k+1))*wp1-(c1*u(4,i,j,k-1)-c2*square(i,j,k-1))*wm1);
	//---------------------------------------------------------------------
	//      add fourth order zeta-direction dissipation                
	//---------------------------------------------------------------------
	if (k == 1) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(5.0*u(m,i,j,k)-4.0*u(m,i,j,k+1)+u(m,i,j,k+2));
	} else if (k == 2) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(-4.0*u(m,i,j,k-1)+6.0*u(m,i,j,k)-4.0*u(m,i,j,k+1)+u(m,i,j,k+2));
	} else if (k >= 3 && k < nz-3) {
		for (m = 0; m < 5; m++)	rtmp[m] = rtmp[m] - dssp*(u(m,i,j,k-2)-4.0*u(m,i,j,k-1)+6.0*u(m,i,j,k)-4.0*u(m,i,j,k+1)+u(m,i,j,k+2));
	} else if (k == nz-3) {
		for (m = 0; m < 5; m++)	rtmp[m] = rtmp[m] - dssp*(u(m,i,j,k-2)-4.0*u(m,i,j,k-1)+6.0*u(m,i,j,k)-4.0*u(m,i,j,k+1));
	} else if (k == nz-2) {
		for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j,k-2)-4.0*u(m,i,j,k-1)+5.0*u(m,i,j,k));
	}

	for (m = 0; m < 5; m++) rhs(m,i,j,k) = rtmp[m] * dt;
}

void BT::compute_rhs () {
	dim3 grid1(ny,nz);
	dim3 grid2(ny-2,nz-2);

	START_TIMER(t_rhs);
	compute_rhs_kernel_1<<<grid1,nx>>>(rho_i, us, vs, ws, qs, square, rhs, forcing, u, nx, ny, nz);
	START_TIMER(t_rhsx);
	compute_rhs_kernel_2<<<grid2,nx-2>>>(rho_i, us, vs, ws, qs, square, rhs, u, nx, ny, nz);
	STOP_TIMER(t_rhsx);
	STOP_TIMER(t_rhs);
}

//---------------------------------------------------------------------
//     subtracts bvec=bvec - ablock*avec
//---------------------------------------------------------------------
__device__ static void matvec_sub_kernel (const int m, const double *ablock, const double *avec, double *bvec) {
	//---------------------------------------------------------------------
	//            rhs(i,ic,jc,kc) = rhs(i,ic,jc,kc) - lhs(i,1,ablock,ia)*
	//---------------------------------------------------------------------
	bvec[m] = bvec[m] - ablock[m+0*5]*avec[0] - ablock[m+1*5]*avec[1] - ablock[m+2*5]*avec[2] - ablock[m+3*5]*avec[3] - ablock[m+4*5]*avec[4];
}

//---------------------------------------------------------------------
// subtracts a(i,j,k) X b(i,j,k) from c(i,j,k)
//---------------------------------------------------------------------
__device__ static void matmul_sub_kernel (const int m, const double *ablock, const double *bblock, double *cblock) {
	cblock[m+0*5] = cblock[m+0*5] - ablock[m+0*5]*bblock[0+0*5] - ablock[m+1*5]*bblock[1+0*5] - ablock[m+2*5]*bblock[2+0*5] - ablock[m+3*5]*bblock[3+0*5] - ablock[m+4*5]*bblock[4+0*5];
	cblock[m+1*5] = cblock[m+1*5] - ablock[m+0*5]*bblock[0+1*5] - ablock[m+1*5]*bblock[1+1*5] - ablock[m+2*5]*bblock[2+1*5] - ablock[m+3*5]*bblock[3+1*5] - ablock[m+4*5]*bblock[4+1*5];
	cblock[m+2*5] = cblock[m+2*5] - ablock[m+0*5]*bblock[0+2*5] - ablock[m+1*5]*bblock[1+2*5] - ablock[m+2*5]*bblock[2+2*5] - ablock[m+3*5]*bblock[3+2*5] - ablock[m+4*5]*bblock[4+2*5];
	cblock[m+3*5] = cblock[m+3*5] - ablock[m+0*5]*bblock[0+3*5] - ablock[m+1*5]*bblock[1+3*5] - ablock[m+2*5]*bblock[2+3*5] - ablock[m+3*5]*bblock[3+3*5] - ablock[m+4*5]*bblock[4+3*5];
	cblock[m+4*5] = cblock[m+4*5] - ablock[m+0*5]*bblock[0+4*5] - ablock[m+1*5]*bblock[1+4*5] - ablock[m+2*5]*bblock[2+4*5] - ablock[m+3*5]*bblock[3+4*5] - ablock[m+4*5]*bblock[4+4*5];
}

__device__ static void binvcrhs_kernel (const int m, double *lhs, double *c, double *r) {
	double pivot;

	pivot = 1.0 / lhs[0+0*5];
	c[0+m*5] *= pivot;
	if (m > 0) lhs[0+m*5] *= pivot;
	else r[0] *= pivot;
	__syncthreads();

	c[1+m*5] -= lhs[1+0*5] * c[0+m*5];
	c[2+m*5] -= lhs[2+0*5] * c[0+m*5];
	c[3+m*5] -= lhs[3+0*5] * c[0+m*5];
	c[4+m*5] -= lhs[4+0*5] * c[0+m*5];
	if (m != 0) {
		lhs[m+1*5] -= lhs[m+0*5] * lhs[0+1*5];
		lhs[m+2*5] -= lhs[m+0*5] * lhs[0+2*5];
		lhs[m+3*5] -= lhs[m+0*5] * lhs[0+3*5];
		lhs[m+4*5] -= lhs[m+0*5] * lhs[0+4*5];
		r[m] -= lhs[m+0*5] * r[0];
	} 
	__syncthreads();

	pivot = 1.0/lhs[1+1*5];
	c[1+m*5] *= pivot;
	if (m > 1) lhs[1+m*5] *= pivot;
	else if (m == 0) r[1] *= pivot;
	__syncthreads();

	c[0+m*5] -= lhs[0+1*5] * c[1+m*5];
	c[2+m*5] -= lhs[2+1*5] * c[1+m*5];
	c[3+m*5] -= lhs[3+1*5] * c[1+m*5];
	c[4+m*5] -= lhs[4+1*5] * c[1+m*5];
	if (m != 1) {
		lhs[m+2*5] -= lhs[m+1*5] * lhs[1+2*5];
		lhs[m+3*5] -= lhs[m+1*5] * lhs[1+3*5];
		lhs[m+4*5] -= lhs[m+1*5] * lhs[1+4*5];
		r[m] -= lhs[m+1*5] * r[1];
	}
	__syncthreads();
	pivot = 1.0 / lhs[2+2*5];
	c[2+m*5] *= pivot;
	if (m > 2) lhs[2+m*5] *= pivot;
	else if (m == 0) r[2] *= pivot;
	__syncthreads();

	c[0+m*5] -= lhs[0+2*5] * c[2+m*5];
	c[1+m*5] -= lhs[1+2*5] * c[2+m*5];
	c[3+m*5] -= lhs[3+2*5] * c[2+m*5];
	c[4+m*5] -= lhs[4+2*5] * c[2+m*5];
	if (m != 2) {
		lhs[m+3*5] -= lhs[m+2*5] * lhs[2+3*5];
		lhs[m+4*5] -= lhs[m+2*5] * lhs[2+4*5];
		r[m] -= lhs[m+2*5] * r[2];
	}
	__syncthreads();

	pivot = 1.0/lhs[3+3*5];
	c[3+m*5] *= pivot;
	if (m == 4) lhs[3+4*5] *= pivot;
	else if (m == 0) r[3] *= pivot;
	__syncthreads();

	c[0+m*5] -= lhs[0+3*5] * c[3+m*5];
	c[1+m*5] -= lhs[1+3*5] * c[3+m*5];
	c[2+m*5] -= lhs[2+3*5] * c[3+m*5];
	c[4+m*5] -= lhs[4+3*5] * c[3+m*5];
	if (m != 3) {
		lhs[m+4*5] -= lhs[m+3*5] * lhs[3+4*5];
		r[m] -= lhs[m+3*5] * r[3];
	}
	__syncthreads();

	pivot = 1.0/lhs[4+4*5];
	c[4+m*5] *= pivot;
	if (m == 0) r[4] *= pivot;
	__syncthreads();

	c[0+m*5] -= lhs[0+4*5] * c[4+m*5];
	c[1+m*5] -= lhs[1+4*5] * c[4+m*5];
	c[2+m*5] -= lhs[2+4*5] * c[4+m*5];
	c[3+m*5] -= lhs[3+4*5] * c[4+m*5];
	if (m != 4) r[m] -= lhs[m+4*5] * r[4];
}

__device__ static void binvrhs_kernel (const int m, double *lhs, double *r) {
	double pivot;

	pivot = 1.0/lhs[0+0*5];
	if (m > 0) lhs[0+m*5] *= pivot;
	else r[0] *= pivot;
	__syncthreads();

	if (m != 0) {
		lhs[m+1*5] -= lhs[m+0*5] * lhs[0+1*5];
		lhs[m+2*5] -= lhs[m+0*5] * lhs[0+2*5];
		lhs[m+3*5] -= lhs[m+0*5] * lhs[0+3*5];
		lhs[m+4*5] -= lhs[m+0*5] * lhs[0+4*5];
		r[m] -= lhs[m+0*5] * r[0];
	}

	__syncthreads();
	pivot = 1.0/lhs[1+1*5];
	if (m > 1) lhs[1+m*5] *= pivot;
	else if (m == 0) r[1] *= pivot;
	__syncthreads();

	if (m != 1) {
		lhs[m+2*5] -= lhs[m+1*5] * lhs[1+2*5];
		lhs[m+3*5] -= lhs[m+1*5] * lhs[1+3*5];
		lhs[m+4*5] -= lhs[m+1*5] * lhs[1+4*5];
		r[m] -= lhs[m+1*5] * r[1];
	}

	__syncthreads();
	pivot = 1.0/lhs[2+2*5];
	if (m > 2) lhs[2+m*5] *= pivot;
	else if (m == 0) r[2] *= pivot;
	__syncthreads();

	if (m != 2) {
		lhs[m+3*5] -= lhs[m+2*5] * lhs[2+3*5];
		lhs[m+4*5] -= lhs[m+2*5] * lhs[2+4*5];
		r[m] -= lhs[m+2*5] * r[2];
	}

	__syncthreads();
	pivot = 1.0/lhs[3+3*5];
	if (m > 3) lhs[3+m*5] *= pivot;
	else if (m == 0) r[3] *= pivot;
	__syncthreads();

	if (m != 3) {
		lhs[m+4*5] -= lhs[m+3*5] * lhs[3+4*5];
		r[m] -= lhs[m+3*5] * r[3];
	}
	
	__syncthreads();
	if (m == 0) {
		pivot = 1.0/lhs[4+4*5];
		r[4] *= pivot;
	}
	__syncthreads();
	if (m != 4) r[m] -= lhs[m+4*5] * r[4];
}

//---------------------------------------------------------------------
//     
//     Performs line solves in X direction by first factoring
//     the block-tridiagonal matrix into an upper triangular matrix, 
//     and then performing back substitution to solve for the unknow
//     vectors of each line.  
//     
//     Make sure we treat elements zero to cell_size in the direction
//     of the sweep.
//     
//---------------------------------------------------------------------
#define lhs(a,b,c,i) lhs[(a)+5*((i)+nx*((b)+5*(c)))]
__global__ static void x_solve_kernel_1 (const double *rho_i, const double *qs, const double *square, const double *u, double *rhs, double *lhs, const int nx, const int ny, const int nz) {
	int j, k, jacofs, lhsofs, jacofs2;
	double tmp1, tmp2, tmp3, utmp[5];
	k = blockIdx.x+1;
	j = blockIdx.y+1;
	__shared__ double fjac[2*5*5*SOLVE_BLOCK];
	__shared__ double njac[2*5*5*SOLVE_BLOCK];
	lhs += (k-1+(j-1)*nz)*5*5*3*nx;

	using namespace gpu_mod;

	lhsofs = threadIdx.x-1;
	jacofs = threadIdx.x;
	while (lhsofs < nx-1) {
		//---------------------------------------------------------------------
		//     This function computes the left hand side in the xi-direction
		//     isize = nx-1
		//---------------------------------------------------------------------
		//---------------------------------------------------------------------
		//     determine a (labeled f) and n jacobians
		//---------------------------------------------------------------------
		jacofs2 = lhsofs+1;
		tmp1 = rho_i(jacofs2,j,k);
		tmp2 = tmp1 * tmp1;
		tmp3 = tmp1 * tmp2;
		utmp[0] = u(0,jacofs2,j,k);
		utmp[1] = u(1,jacofs2,j,k);
		utmp[2] = u(2,jacofs2,j,k);
		utmp[3] = u(3,jacofs2,j,k);
		utmp[4] = u(4,jacofs2,j,k);
		//---------------------------------------------------------------------
		//     
		//---------------------------------------------------------------------
		fjac(0,0,jacofs) = 0.0;
		fjac(0,1,jacofs) = 1.0;
		fjac(0,2,jacofs) = 0.0;
		fjac(0,3,jacofs) = 0.0;
		fjac(0,4,jacofs) = 0.0;

		fjac(1,0,jacofs) = -(utmp[1] * tmp2 * utmp[1]) + c2*qs(jacofs2,j,k);
		fjac(1,1,jacofs) = (2.0-c2) *(utmp[1]/utmp[0]);
		fjac(1,2,jacofs) = -c2 * (utmp[2] * tmp1);
		fjac(1,3,jacofs) = -c2 * (utmp[3] * tmp1);
		fjac(1,4,jacofs) = c2;

		fjac(2,0,jacofs) = -(utmp[1]*utmp[2]) * tmp2;
		fjac(2,1,jacofs) = utmp[2] * tmp1;
		fjac(2,2,jacofs) = utmp[1] * tmp1;
		fjac(2,3,jacofs) = 0.0;
		fjac(2,4,jacofs) = 0.0;

		fjac(3,0,jacofs) = -(utmp[1]*utmp[3]) * tmp2;
		fjac(3,1,jacofs) = utmp[3] * tmp1;
		fjac(3,2,jacofs) = 0.0;
		fjac(3,3,jacofs) = utmp[1] * tmp1;
		fjac(3,4,jacofs) = 0.0;

		fjac(4,0,jacofs) = (c2*2.0*square(jacofs2,j,k) - c1*utmp[4]) * (utmp[1]*tmp2);
		fjac(4,1,jacofs) = c1*utmp[4]*tmp1 - c2*(utmp[1]*utmp[1]*tmp2 + qs(jacofs2,j,k));
		fjac(4,2,jacofs) = -c2 * (utmp[2]*utmp[1]) * tmp2;
		fjac(4,3,jacofs) = -c2 * (utmp[3]*utmp[1]) * tmp2;
		fjac(4,4,jacofs) = c1 * (utmp[1] * tmp1);

		njac(0,0,jacofs) = 0.0;
		njac(0,1,jacofs) = 0.0;
		njac(0,2,jacofs) = 0.0;
		njac(0,3,jacofs) = 0.0;
		njac(0,4,jacofs) = 0.0;

		njac(1,0,jacofs) = -con43 * c3c4 * tmp2 * utmp[1];
		njac(1,1,jacofs) = con43 * c3c4 * tmp1;
		njac(1,2,jacofs) = 0.0;
		njac(1,3,jacofs) = 0.0;
		njac(1,4,jacofs) = 0.0;

		njac(2,0,jacofs) = -c3c4 * tmp2 * utmp[2];
		njac(2,1,jacofs) = 0.0;
		njac(2,2,jacofs) = c3c4 * tmp1;
		njac(2,3,jacofs) = 0.0;
		njac(2,4,jacofs) = 0.0;

		njac(3,0,jacofs) = -c3c4 * tmp2 * utmp[3];
		njac(3,1,jacofs) = 0.0;
		njac(3,2,jacofs) = 0.0;
		njac(3,3,jacofs) = c3c4 * tmp1;
		njac(3,4,jacofs) = 0.0;

		njac(4,0,jacofs) = -(con43*c3c4-c1345)*tmp3*(utmp[1]*utmp[1]) - (c3c4-c1345)*tmp3*(utmp[2]*utmp[2]) - (c3c4-c1345)*tmp3*(utmp[3]*utmp[3]) - c1345*tmp2*utmp[4];
		njac(4,1,jacofs) = (con43*c3c4-c1345)*tmp2*utmp[1];
		njac(4,2,jacofs) = (c3c4-c1345)*tmp2*utmp[2];
		njac(4,3,jacofs) = (c3c4-c1345)*tmp2*utmp[3];
		njac(4,4,jacofs) = c1345*tmp1;
	
		//---------------------------------------------------------------------
		//     now jacobians set, so form left hand side in x direction
		//---------------------------------------------------------------------
		__syncthreads();
		if (lhsofs < 1) {
			jacofs2 = lhsofs == 0 ? 0 : nx-1;
			for (int m = 0; m < 5; m++) {
				for (int n = 0; n < 5; n++) {
					lhs(m,n,aa,jacofs2) = 0.0;
					lhs(m,n,bb,jacofs2) = 0.0;
					lhs(m,n,cc,jacofs2) = 0.0;
				}
				lhs(m,m,bb,jacofs2) = 1.0;
			}
		} else {
			tmp1 = dt * tx1;
			tmp2 = dt * tx2;

			jacofs2 = (2*SOLVE_BLOCK+jacofs-2) % (2*SOLVE_BLOCK);
			lhs(0,0,aa,lhsofs) = -tmp2*fjac(0,0,jacofs2) - tmp1*njac(0,0,jacofs2) - tmp1*dx1;
			lhs(0,1,aa,lhsofs) = -tmp2*fjac(0,1,jacofs2) - tmp1*njac(0,1,jacofs2);
			lhs(0,2,aa,lhsofs) = -tmp2*fjac(0,2,jacofs2) - tmp1*njac(0,2,jacofs2);
			lhs(0,3,aa,lhsofs) = -tmp2*fjac(0,3,jacofs2) - tmp1*njac(0,3,jacofs2);
			lhs(0,4,aa,lhsofs) = -tmp2*fjac(0,4,jacofs2) - tmp1*njac(0,4,jacofs2);

			lhs(1,0,aa,lhsofs) = -tmp2*fjac(1,0,jacofs2) - tmp1*njac(1,0,jacofs2);
			lhs(1,1,aa,lhsofs) = -tmp2*fjac(1,1,jacofs2) - tmp1*njac(1,1,jacofs2) - tmp1*dx2;
			lhs(1,2,aa,lhsofs) = -tmp2*fjac(1,2,jacofs2) - tmp1*njac(1,2,jacofs2);
			lhs(1,3,aa,lhsofs) = -tmp2*fjac(1,3,jacofs2) - tmp1*njac(1,3,jacofs2);
			lhs(1,4,aa,lhsofs) = -tmp2*fjac(1,4,jacofs2) - tmp1*njac(1,4,jacofs2);

			lhs(2,0,aa,lhsofs) = -tmp2*fjac(2,0,jacofs2) - tmp1*njac(2,0,jacofs2);
			lhs(2,1,aa,lhsofs) = -tmp2*fjac(2,1,jacofs2) - tmp1*njac(2,1,jacofs2);
			lhs(2,2,aa,lhsofs) = -tmp2*fjac(2,2,jacofs2) - tmp1*njac(2,2,jacofs2) - tmp1*dx3;
			lhs(2,3,aa,lhsofs) = -tmp2*fjac(2,3,jacofs2) - tmp1*njac(2,3,jacofs2);
			lhs(2,4,aa,lhsofs) = -tmp2*fjac(2,4,jacofs2) - tmp1*njac(2,4,jacofs2);

			lhs(3,0,aa,lhsofs) = -tmp2*fjac(3,0,jacofs2) - tmp1*njac(3,0,jacofs2);
			lhs(3,1,aa,lhsofs) = -tmp2*fjac(3,1,jacofs2) - tmp1*njac(3,1,jacofs2);
			lhs(3,2,aa,lhsofs) = -tmp2*fjac(3,2,jacofs2) - tmp1*njac(3,2,jacofs2);
			lhs(3,3,aa,lhsofs) = -tmp2*fjac(3,3,jacofs2) - tmp1*njac(3,3,jacofs2) - tmp1*dx4;
			lhs(3,4,aa,lhsofs) = -tmp2*fjac(3,4,jacofs2) - tmp1*njac(3,4,jacofs2);

			lhs(4,0,aa,lhsofs) = -tmp2*fjac(4,0,jacofs2) - tmp1*njac(4,0,jacofs2);
			lhs(4,1,aa,lhsofs) = -tmp2*fjac(4,1,jacofs2) - tmp1*njac(4,1,jacofs2);
			lhs(4,2,aa,lhsofs) = -tmp2*fjac(4,2,jacofs2) - tmp1*njac(4,2,jacofs2);
			lhs(4,3,aa,lhsofs) = -tmp2*fjac(4,3,jacofs2) - tmp1*njac(4,3,jacofs2);
			lhs(4,4,aa,lhsofs) = -tmp2*fjac(4,4,jacofs2) - tmp1*njac(4,4,jacofs2) - tmp1*dx5;

			jacofs2 = (jacofs2+1) % (2*SOLVE_BLOCK);
			lhs(0,0,bb,lhsofs) = 1.0 + tmp1*2.0*njac(0,0,jacofs2) + tmp1*2.0*dx1;
			lhs(0,1,bb,lhsofs) = tmp1*2.0*njac(0,1,jacofs2);
			lhs(0,2,bb,lhsofs) = tmp1*2.0*njac(0,2,jacofs2);
			lhs(0,3,bb,lhsofs) = tmp1*2.0*njac(0,3,jacofs2);
			lhs(0,4,bb,lhsofs) = tmp1*2.0*njac(0,4,jacofs2);

			lhs(1,0,bb,lhsofs) = tmp1*2.0*njac(1,0,jacofs2);
			lhs(1,1,bb,lhsofs) = 1.0 + tmp1*2.0*njac(1,1,jacofs2) + tmp1*2.0*dx2;
			lhs(1,2,bb,lhsofs) = tmp1*2.0*njac(1,2,jacofs2);
			lhs(1,3,bb,lhsofs) = tmp1*2.0*njac(1,3,jacofs2);
			lhs(1,4,bb,lhsofs) = tmp1*2.0*njac(1,4,jacofs2);

			lhs(2,0,bb,lhsofs) = tmp1*2.0*njac(2,0,jacofs2);
			lhs(2,1,bb,lhsofs) = tmp1*2.0*njac(2,1,jacofs2);
			lhs(2,2,bb,lhsofs) = 1.0 + tmp1*2.0*njac(2,2,jacofs2) + tmp1*2.0*dx3;
			lhs(2,3,bb,lhsofs) = tmp1*2.0*njac(2,3,jacofs2);
			lhs(2,4,bb,lhsofs) = tmp1*2.0*njac(2,4,jacofs2);

			lhs(3,0,bb,lhsofs) = tmp1*2.0*njac(3,0,jacofs2);
			lhs(3,1,bb,lhsofs) = tmp1*2.0*njac(3,1,jacofs2);
			lhs(3,2,bb,lhsofs) = tmp1*2.0*njac(3,2,jacofs2);
			lhs(3,3,bb,lhsofs) = 1.0 + tmp1*2.0*njac(3,3,jacofs2) + tmp1*2.0*dx4;
			lhs(3,4,bb,lhsofs) = tmp1*2.0*njac(3,4,jacofs2);

			lhs(4,0,bb,lhsofs) = tmp1*2.0*njac(4,0,jacofs2);
			lhs(4,1,bb,lhsofs) = tmp1*2.0*njac(4,1,jacofs2);
			lhs(4,2,bb,lhsofs) = tmp1*2.0*njac(4,2,jacofs2);
			lhs(4,3,bb,lhsofs) = tmp1*2.0*njac(4,3,jacofs2);
			lhs(4,4,bb,lhsofs) = 1.0 + tmp1*2.0*njac(4,4,jacofs2) + tmp1*2.0*dx5;

			jacofs2 = (jacofs2+1) % (2*SOLVE_BLOCK);
			lhs(0,0,cc,lhsofs) = tmp2*fjac(0,0,jacofs2) - tmp1*njac(0,0,jacofs2) - tmp1*dx1;
			lhs(0,1,cc,lhsofs) = tmp2*fjac(0,1,jacofs2) - tmp1*njac(0,1,jacofs2);
			lhs(0,2,cc,lhsofs) = tmp2*fjac(0,2,jacofs2) - tmp1*njac(0,2,jacofs2);
			lhs(0,3,cc,lhsofs) = tmp2*fjac(0,3,jacofs2) - tmp1*njac(0,3,jacofs2);
			lhs(0,4,cc,lhsofs) = tmp2*fjac(0,4,jacofs2) - tmp1*njac(0,4,jacofs2);

			lhs(1,0,cc,lhsofs) = tmp2*fjac(1,0,jacofs2) - tmp1*njac(1,0,jacofs2);
			lhs(1,1,cc,lhsofs) = tmp2*fjac(1,1,jacofs2) - tmp1*njac(1,1,jacofs2) - tmp1*dx2;
			lhs(1,2,cc,lhsofs) = tmp2*fjac(1,2,jacofs2) - tmp1*njac(1,2,jacofs2);
			lhs(1,3,cc,lhsofs) = tmp2*fjac(1,3,jacofs2) - tmp1*njac(1,3,jacofs2);
			lhs(1,4,cc,lhsofs) = tmp2*fjac(1,4,jacofs2) - tmp1*njac(1,4,jacofs2);

			lhs(2,0,cc,lhsofs) = tmp2*fjac(2,0,jacofs2) - tmp1*njac(2,0,jacofs2);
			lhs(2,1,cc,lhsofs) = tmp2*fjac(2,1,jacofs2) - tmp1*njac(2,1,jacofs2);
			lhs(2,2,cc,lhsofs) = tmp2*fjac(2,2,jacofs2) - tmp1*njac(2,2,jacofs2) - tmp1*dx3;
			lhs(2,3,cc,lhsofs) = tmp2*fjac(2,3,jacofs2) - tmp1*njac(2,3,jacofs2);
			lhs(2,4,cc,lhsofs) = tmp2*fjac(2,4,jacofs2) - tmp1*njac(2,4,jacofs2);

			lhs(3,0,cc,lhsofs) = tmp2*fjac(3,0,jacofs2) - tmp1*njac(3,0,jacofs2);
			lhs(3,1,cc,lhsofs) = tmp2*fjac(3,1,jacofs2) - tmp1*njac(3,1,jacofs2);
			lhs(3,2,cc,lhsofs) = tmp2*fjac(3,2,jacofs2) - tmp1*njac(3,2,jacofs2);
			lhs(3,3,cc,lhsofs) = tmp2*fjac(3,3,jacofs2) - tmp1*njac(3,3,jacofs2) - tmp1*dx4;
			lhs(3,4,cc,lhsofs) = tmp2*fjac(3,4,jacofs2) - tmp1*njac(3,4,jacofs2);

			lhs(4,0,cc,lhsofs) = tmp2*fjac(4,0,jacofs2) - tmp1*njac(4,0,jacofs2);
			lhs(4,1,cc,lhsofs) = tmp2*fjac(4,1,jacofs2) - tmp1*njac(4,1,jacofs2);
			lhs(4,2,cc,lhsofs) = tmp2*fjac(4,2,jacofs2) - tmp1*njac(4,2,jacofs2);
			lhs(4,3,cc,lhsofs) = tmp2*fjac(4,3,jacofs2) - tmp1*njac(4,3,jacofs2);
			lhs(4,4,cc,lhsofs) = tmp2*fjac(4,4,jacofs2) - tmp1*njac(4,4,jacofs2) - tmp1*dx5;
		} 
		lhsofs += SOLVE_BLOCK;
		jacofs = (jacofs + SOLVE_BLOCK) % (2*SOLVE_BLOCK);
	}
}

__global__ static void x_solve_kernel_2 (double *rhs, double *lhs, const int nx, const int ny, const int nz) {
	int j, k, m;
	k = blockIdx.x+1;
	j = blockIdx.y+1;
	m = threadIdx.x;
	lhs += (k-1+(j-1)*nz)*5*5*3*nx;
	__shared__ double rtmp[2][5];
	__shared__ double lhsbtmp[5*5], lhsctmp[5*5], lhsatmp[5*5];

	rtmp[0][m] = rhs(m,0,j,k);
	for (int n = 0; n < 5; n++) {
		lhsbtmp[m+5*n] = lhs(m,n,bb,0);
		lhsctmp[m+5*n] = lhs(m,n,cc,0);
	}
	__syncthreads();
	binvcrhs_kernel (m, lhsbtmp, lhsctmp, rtmp[0]);
	for (int n = 0; n < 5; n++) lhs(m,n,cc,0) = lhsctmp[m+5*n];
	for (int i = 1; i < nx-1; i++) {
		rtmp[1][m] = rhs(m,i,j,k);
		for (int n = 0; n < 5; n++) {
			lhsatmp[m+5*n] = lhs(m,n,aa,i);
			lhsbtmp[m+5*n] = lhs(m,n,bb,i);
		}
		__syncthreads();
		matvec_sub_kernel (m, lhsatmp, rtmp[0], rtmp[1]);
		matmul_sub_kernel (m, lhsatmp, lhsctmp, lhsbtmp);
		for (int n = 0; n < 5; n++) lhsctmp[m+5*n] = lhs(m,n,cc,i);
		__syncthreads();
		binvcrhs_kernel (m, lhsbtmp, lhsctmp, rtmp[1]);
		for (int n = 0; n < 5; n++) lhs(m,n,cc,i) = lhsctmp[m+5*n];
		rhs(m,i-1,j,k) = rtmp[0][m];
		rtmp[0][m] = rtmp[1][m];
	}
	rtmp[1][m] = rhs(m,nx-1,j,k);
	for (int n = 0; n < 5; n++) {
		lhsatmp[m+5*n] = lhs(m,n,aa,nx-1);
		lhsbtmp[m+5*n] = lhs(m,n,bb,nx-1);
	}
	__syncthreads();
	matvec_sub_kernel (m, lhsatmp, rtmp[0], rtmp[1]);
	matvec_sub_kernel (m, lhsatmp, lhsctmp, lhsbtmp);
	binvrhs_kernel (m, lhsbtmp, rtmp[1]);
	for (int i = nx-2; i >= 0; i--) {
		for (int n = 0; n < 5; n++) rtmp[0][m] -= lhs(m,n,cc,i)*rtmp[1][n];
		rhs(m,i,j,k) = rtmp[1][m] = rtmp[0][m];
		if (i > 0) rtmp[0][m] = rhs(m,i-1,j,k);
		__syncthreads();
	}
}
#undef lhs

void BT::x_solve () {
	dim3 grid(nz-2,ny-2);

	START_TIMER(t_xsolve);
	x_solve_kernel_1<<<grid,SOLVE_BLOCK>>>(rho_i, qs, square, u, rhs, lhs, nx, ny, nz); 
	x_solve_kernel_2<<<grid,5>>>(rhs, lhs, nx, ny, nz); 
	STOP_TIMER(t_xsolve);
}

//---------------------------------------------------------------------
//     Performs line solves in Y direction by first factoring
//     the block-tridiagonal matrix into an upper triangular matrix, 
//     and then performing back substitution to solve for the unknow
//     vectors of each line.  
//     
//     Make sure we treat elements zero to cell_size in the direction
//     of the sweep.
//---------------------------------------------------------------------
#define lhs(a,b,c,i) lhs[(a)+5*((i)+ny*((b)+5*(c)))]
__global__ static void y_solve_kernel_1 (const double *rho_i, const double *qs, const double *square, const double *u, double *rhs, double *lhs, const int nx, const int ny, const int nz) {
	int i, k, jacofs, jacofs2, lhsofs;
	double tmp1, tmp2, tmp3, utmp[5];
	k = blockIdx.x+1;
	i = blockIdx.y+1;
	__shared__ double fjac[2*5*5*SOLVE_BLOCK];
	__shared__ double njac[2*5*5*SOLVE_BLOCK];
	lhs += (k-1+(i-1)*nz)*5*5*3*ny;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	//     This function computes the left hand side for the three y-factors   
	//     jsize = ny-1
	//---------------------------------------------------------------------
	//---------------------------------------------------------------------
	//     Compute the indices for storing the tri-diagonal matrix;
	//     determine a (labeled f) and n jacobians for cell c
	//---------------------------------------------------------------------
	jacofs = threadIdx.x;
	lhsofs = jacofs-1;
	while (lhsofs < ny-1) {
		jacofs2 = lhsofs+1;
		tmp1 = rho_i(i,jacofs2,k);
		tmp2 = tmp1 * tmp1;
		tmp3 = tmp1 * tmp2;
		utmp[0] = u(0,i,jacofs2,k);
		utmp[1] = u(1,i,jacofs2,k);
		utmp[2] = u(2,i,jacofs2,k);
		utmp[3] = u(3,i,jacofs2,k);
		utmp[4] = u(4,i,jacofs2,k);

		fjac(0,0,jacofs) = 0.0;
		fjac(0,1,jacofs) = 0.0;
		fjac(0,2,jacofs) = 1.0;
		fjac(0,3,jacofs) = 0.0;
		fjac(0,4,jacofs) = 0.0;

		fjac(1,0,jacofs) = -(utmp[1]*utmp[2])*tmp2;
		fjac(1,1,jacofs) = utmp[2]* tmp1;
		fjac(1,2,jacofs) = utmp[1]* tmp1;
		fjac(1,3,jacofs) = 0.0;
		fjac(1,4,jacofs) = 0.0;

		fjac(2,0,jacofs) = -(utmp[2]*utmp[2]*tmp2) + c2*qs(i,jacofs2,k);
		fjac(2,1,jacofs) = -c2*utmp[1]*tmp1;
		fjac(2,2,jacofs) = (2.0-c2) * utmp[2] * tmp1;
		fjac(2,3,jacofs) = -c2 * utmp[3] * tmp1;
		fjac(2,4,jacofs) = c2;

		fjac(3,0,jacofs) = -(utmp[2]*utmp[3]) * tmp2;
		fjac(3,1,jacofs) = 0.0;
		fjac(3,2,jacofs) = utmp[3] * tmp1;
		fjac(3,3,jacofs) = utmp[2] * tmp1;
		fjac(3,4,jacofs) = 0.0;

		fjac(4,0,jacofs) = (c2*2.0*square(i,jacofs2,k) - c1*utmp[4]) * utmp[2] * tmp2;
		fjac(4,1,jacofs) = -c2 * utmp[1] * utmp[2] * tmp2;
		fjac(4,2,jacofs) = c1 * utmp[4] * tmp1 - c2 * (qs(i,jacofs2,k)+utmp[2]*utmp[2]*tmp2);
		fjac(4,3,jacofs) = -c2 * (utmp[2]*utmp[3]) * tmp2;
		fjac(4,4,jacofs) = c1 * utmp[2] * tmp1;

		njac(0,0,jacofs) = 0.0;
		njac(0,1,jacofs) = 0.0;
		njac(0,2,jacofs) = 0.0;
		njac(0,3,jacofs) = 0.0;
		njac(0,4,jacofs) = 0.0;

		njac(1,0,jacofs) = -c3c4 * tmp2 * utmp[1];
		njac(1,1,jacofs) = c3c4 * tmp1;
		njac(1,2,jacofs) = 0.0;
		njac(1,3,jacofs) = 0.0;
		njac(1,4,jacofs) = 0.0;

		njac(2,0,jacofs) = -con43 * c3c4 * tmp2 * utmp[2];
		njac(2,1,jacofs) = 0.0;
		njac(2,2,jacofs) = con43 * c3c4 * tmp1;
		njac(2,3,jacofs) = 0.0;
		njac(2,4,jacofs) = 0.0;

		njac(3,0,jacofs) = -c3c4 * tmp2 * utmp[3];
		njac(3,1,jacofs) = 0.0;
		njac(3,2,jacofs) = 0.0;
		njac(3,3,jacofs) = c3c4 * tmp1;
		njac(3,4,jacofs) = 0.0;

		njac(4,0,jacofs) = -(c3c4-c1345)*tmp3*(utmp[1]*utmp[1]) - (con43*c3c4-c1345)*tmp3*(utmp[2]*utmp[2]) - (c3c4-c1345)*tmp3*(utmp[3]*utmp[3]) - c1345*tmp2*utmp[4];
		njac(4,1,jacofs) = (c3c4-c1345)*tmp2*utmp[1];
		njac(4,2,jacofs) = (con43*c3c4-c1345) * tmp2 * utmp[2];
		njac(4,3,jacofs) = (c3c4-c1345) * tmp2 * utmp[3];
		njac(4,4,jacofs) = (c1345) * tmp1;
		//---------------------------------------------------------------------
		//     now joacobians set, so form left hand side in y direction
		//---------------------------------------------------------------------
		__syncthreads();
		if (lhsofs < 1) {
			jacofs2 = lhsofs == 0 ? 0 : ny-1;
			for (int m = 0; m < 5; m++) {
				for (int n = 0; n < 5; n++) {
					lhs(m,n,aa,jacofs2) = 0.0;
					lhs(m,n,bb,jacofs2) = 0.0;
					lhs(m,n,cc,jacofs2) = 0.0;
				}
				lhs(m,m,bb,jacofs2) = 1.0;
			}
		} else {
			tmp1 = dt * ty1;
			tmp2 = dt * ty2;

			jacofs2 = (2*SOLVE_BLOCK+jacofs-2) % (2*SOLVE_BLOCK);
			lhs(0,0,aa,lhsofs) = -tmp2*fjac(0,0,jacofs2) - tmp1*njac(0,0,jacofs2) - tmp1*dy1;
			lhs(0,1,aa,lhsofs) = -tmp2*fjac(0,1,jacofs2) - tmp1*njac(0,1,jacofs2);
			lhs(0,2,aa,lhsofs) = -tmp2*fjac(0,2,jacofs2) - tmp1*njac(0,2,jacofs2);
			lhs(0,3,aa,lhsofs) = -tmp2*fjac(0,3,jacofs2) - tmp1*njac(0,3,jacofs2);
			lhs(0,4,aa,lhsofs) = -tmp2*fjac(0,4,jacofs2) - tmp1*njac(0,4,jacofs2);

			lhs(1,0,aa,lhsofs) = -tmp2*fjac(1,0,jacofs2) - tmp1*njac(1,0,jacofs2);
			lhs(1,1,aa,lhsofs) = -tmp2*fjac(1,1,jacofs2) - tmp1*njac(1,1,jacofs2) - tmp1*dy2;
			lhs(1,2,aa,lhsofs) = -tmp2*fjac(1,2,jacofs2) - tmp1*njac(1,2,jacofs2);
			lhs(1,3,aa,lhsofs) = -tmp2*fjac(1,3,jacofs2) - tmp1*njac(1,3,jacofs2);
			lhs(1,4,aa,lhsofs) = -tmp2*fjac(1,4,jacofs2) - tmp1*njac(1,4,jacofs2);

			lhs(2,0,aa,lhsofs) = -tmp2*fjac(2,0,jacofs2) - tmp1*njac(2,0,jacofs2);
			lhs(2,1,aa,lhsofs) = -tmp2*fjac(2,1,jacofs2) - tmp1*njac(2,1,jacofs2);
			lhs(2,2,aa,lhsofs) = -tmp2*fjac(2,2,jacofs2) - tmp1*njac(2,2,jacofs2) - tmp1*dy3;
			lhs(2,3,aa,lhsofs) = -tmp2*fjac(2,3,jacofs2) - tmp1*njac(2,3,jacofs2);
			lhs(2,4,aa,lhsofs) = -tmp2*fjac(2,4,jacofs2) - tmp1*njac(2,4,jacofs2);

			lhs(3,0,aa,lhsofs) = -tmp2*fjac(3,0,jacofs2) - tmp1*njac(3,0,jacofs2);
			lhs(3,1,aa,lhsofs) = -tmp2*fjac(3,1,jacofs2) - tmp1*njac(3,1,jacofs2);
			lhs(3,2,aa,lhsofs) = -tmp2*fjac(3,2,jacofs2) - tmp1*njac(3,2,jacofs2);
			lhs(3,3,aa,lhsofs) = -tmp2*fjac(3,3,jacofs2) - tmp1*njac(3,3,jacofs2) - tmp1*dy4;
			lhs(3,4,aa,lhsofs) = -tmp2*fjac(3,4,jacofs2) - tmp1*njac(3,4,jacofs2);

			lhs(4,0,aa,lhsofs) = -tmp2*fjac(4,0,jacofs2) - tmp1*njac(4,0,jacofs2);
			lhs(4,1,aa,lhsofs) = -tmp2*fjac(4,1,jacofs2) - tmp1*njac(4,1,jacofs2);
			lhs(4,2,aa,lhsofs) = -tmp2*fjac(4,2,jacofs2) - tmp1*njac(4,2,jacofs2);
			lhs(4,3,aa,lhsofs) = -tmp2*fjac(4,3,jacofs2) - tmp1*njac(4,3,jacofs2);
			lhs(4,4,aa,lhsofs) = -tmp2*fjac(4,4,jacofs2) - tmp1*njac(4,4,jacofs2) - tmp1*dy5;

			jacofs2 = (jacofs2+1) % (2*SOLVE_BLOCK);
			lhs(0,0,bb,lhsofs) = 1.0 + tmp1*2.0*njac(0,0,jacofs2) + tmp1*2.0*dy1;
			lhs(0,1,bb,lhsofs) = tmp1*2.0*njac(0,1,jacofs2);
			lhs(0,2,bb,lhsofs) = tmp1*2.0*njac(0,2,jacofs2);
			lhs(0,3,bb,lhsofs) = tmp1*2.0*njac(0,3,jacofs2);
			lhs(0,4,bb,lhsofs) = tmp1*2.0*njac(0,4,jacofs2);

			lhs(1,0,bb,lhsofs) = tmp1*2.0*njac(1,0,jacofs2);
			lhs(1,1,bb,lhsofs) = 1.0 + tmp1*2.0*njac(1,1,jacofs2) + tmp1*2.0*dy2;
			lhs(1,2,bb,lhsofs) = tmp1*2.0*njac(1,2,jacofs2);
			lhs(1,3,bb,lhsofs) = tmp1*2.0*njac(1,3,jacofs2);
			lhs(1,4,bb,lhsofs) = tmp1*2.0*njac(1,4,jacofs2);

			lhs(2,0,bb,lhsofs) = tmp1*2.0*njac(2,0,jacofs2);
			lhs(2,1,bb,lhsofs) = tmp1*2.0*njac(2,1,jacofs2);
			lhs(2,2,bb,lhsofs) = 1.0 + tmp1*2.0*njac(2,2,jacofs2) + tmp1*2.0*dy3;
			lhs(2,3,bb,lhsofs) = tmp1*2.0*njac(2,3,jacofs2);
			lhs(2,4,bb,lhsofs) = tmp1*2.0*njac(2,4,jacofs2);

			lhs(3,0,bb,lhsofs) = tmp1*2.0*njac(3,0,jacofs2);
			lhs(3,1,bb,lhsofs) = tmp1*2.0*njac(3,1,jacofs2);
			lhs(3,2,bb,lhsofs) = tmp1*2.0*njac(3,2,jacofs2);
			lhs(3,3,bb,lhsofs) = 1.0 + tmp1*2.0*njac(3,3,jacofs2) + tmp1*2.0*dy4;
			lhs(3,4,bb,lhsofs) = tmp1*2.0*njac(3,4,jacofs2);

			lhs(4,0,bb,lhsofs) = tmp1*2.0*njac(4,0,jacofs2);
			lhs(4,1,bb,lhsofs) = tmp1*2.0*njac(4,1,jacofs2);
			lhs(4,2,bb,lhsofs) = tmp1*2.0*njac(4,2,jacofs2);
			lhs(4,3,bb,lhsofs) = tmp1*2.0*njac(4,3,jacofs2);
			lhs(4,4,bb,lhsofs) = 1.0 + tmp1*2.0*njac(4,4,jacofs2) + tmp1*2.0*dy5;

			jacofs2 = (jacofs2+1) % (2*SOLVE_BLOCK);
			lhs(0,0,cc,lhsofs) = tmp2*fjac(0,0,jacofs2) - tmp1*njac(0,0,jacofs2) - tmp1*dy1;
			lhs(0,1,cc,lhsofs) = tmp2*fjac(0,1,jacofs2) - tmp1*njac(0,1,jacofs2);
			lhs(0,2,cc,lhsofs) = tmp2*fjac(0,2,jacofs2) - tmp1*njac(0,2,jacofs2);
			lhs(0,3,cc,lhsofs) = tmp2*fjac(0,3,jacofs2) - tmp1*njac(0,3,jacofs2);
			lhs(0,4,cc,lhsofs) = tmp2*fjac(0,4,jacofs2) - tmp1*njac(0,4,jacofs2);

			lhs(1,0,cc,lhsofs) = tmp2*fjac(1,0,jacofs2) - tmp1*njac(1,0,jacofs2);
			lhs(1,1,cc,lhsofs) = tmp2*fjac(1,1,jacofs2) - tmp1*njac(1,1,jacofs2) - tmp1*dy2;
			lhs(1,2,cc,lhsofs) = tmp2*fjac(1,2,jacofs2) - tmp1*njac(1,2,jacofs2);
			lhs(1,3,cc,lhsofs) = tmp2*fjac(1,3,jacofs2) - tmp1*njac(1,3,jacofs2);
			lhs(1,4,cc,lhsofs) = tmp2*fjac(1,4,jacofs2) - tmp1*njac(1,4,jacofs2);

			lhs(2,0,cc,lhsofs) = tmp2*fjac(2,0,jacofs2) - tmp1*njac(2,0,jacofs2);
			lhs(2,1,cc,lhsofs) = tmp2*fjac(2,1,jacofs2) - tmp1*njac(2,1,jacofs2);
			lhs(2,2,cc,lhsofs) = tmp2*fjac(2,2,jacofs2) - tmp1*njac(2,2,jacofs2) - tmp1*dy3;
			lhs(2,3,cc,lhsofs) = tmp2*fjac(2,3,jacofs2) - tmp1*njac(2,3,jacofs2);
			lhs(2,4,cc,lhsofs) = tmp2*fjac(2,4,jacofs2) - tmp1*njac(2,4,jacofs2);

			lhs(3,0,cc,lhsofs) = tmp2*fjac(3,0,jacofs2) - tmp1*njac(3,0,jacofs2);
			lhs(3,1,cc,lhsofs) = tmp2*fjac(3,1,jacofs2) - tmp1*njac(3,1,jacofs2);
			lhs(3,2,cc,lhsofs) = tmp2*fjac(3,2,jacofs2) - tmp1*njac(3,2,jacofs2);
			lhs(3,3,cc,lhsofs) = tmp2*fjac(3,3,jacofs2) - tmp1*njac(3,3,jacofs2) - tmp1*dy4;
			lhs(3,4,cc,lhsofs) = tmp2*fjac(3,4,jacofs2) - tmp1*njac(3,4,jacofs2);

			lhs(4,0,cc,lhsofs) = tmp2*fjac(4,0,jacofs2) - tmp1*njac(4,0,jacofs2);
			lhs(4,1,cc,lhsofs) = tmp2*fjac(4,1,jacofs2) - tmp1*njac(4,1,jacofs2);
			lhs(4,2,cc,lhsofs) = tmp2*fjac(4,2,jacofs2) - tmp1*njac(4,2,jacofs2);
			lhs(4,3,cc,lhsofs) = tmp2*fjac(4,3,jacofs2) - tmp1*njac(4,3,jacofs2);
			lhs(4,4,cc,lhsofs) = tmp2*fjac(4,4,jacofs2) - tmp1*njac(4,4,jacofs2) - tmp1*dy5;
		}
		lhsofs += SOLVE_BLOCK;
		jacofs = (jacofs + SOLVE_BLOCK) % (2*SOLVE_BLOCK);
	}
}

__global__ static void y_solve_kernel_2 (double *rhs, double *lhs, const int nx, const int ny, const int nz) {
	int i, k, m;
	k = blockIdx.x+1;
	i = blockIdx.y+1;
	m = threadIdx.x;
	lhs += (k-1+(i-1)*nz)*5*5*3*ny;
	__shared__ double rtmp[2][5];
	__shared__ double lhsbtmp[5*5], lhsctmp[5*5], lhsatmp[5*5];

	rtmp[0][m] = rhs(m,i,0,k);
	for (int n = 0; n < 5; n++) {
		lhsbtmp[m+5*n] = lhs(m,n,bb,0);
		lhsctmp[m+5*n] = lhs(m,n,cc,0);
	}
	__syncthreads();
	binvcrhs_kernel (m, lhsbtmp, lhsctmp, rtmp[0]);
	for (int n = 0; n < 5; n++) lhs(m,n,cc,0) = lhsctmp[m+5*n];
	for (int j = 1; j < ny-1; j++) {
		rtmp[1][m] = rhs(m,i,j,k);
		for (int n = 0; n < 5; n++) {
			lhsatmp[m+5*n] = lhs(m,n,aa,j);
			lhsbtmp[m+5*n] = lhs(m,n,bb,j);
		}
		__syncthreads();
		matvec_sub_kernel(m, lhsatmp, rtmp[0], rtmp[1]);
		matmul_sub_kernel(m, lhsatmp, lhsctmp, lhsbtmp);
		for (int n = 0; n < 5; n++) lhsctmp[m+5*n] = lhs(m,n,cc,j);
		__syncthreads();
		binvcrhs_kernel (m, lhsbtmp, lhsctmp, rtmp[1]);
		for (int n = 0; n < 5; n++) lhs(m,n,cc,j) = lhsctmp[m+5*n];
		rhs(m,i,j-1,k) = rtmp[0][m];
		rtmp[0][m] = rtmp[1][m];
	}
	rtmp[1][m] = rhs(m,i,ny-1,k);
	for (int n = 0; n < 5; n++) {
		lhsatmp[m+5*n] = lhs(m,n,aa,ny-1);
		lhsbtmp[m+5*n] = lhs(m,n,bb,ny-1);
	}
	__syncthreads();
	matvec_sub_kernel(m, lhsatmp, rtmp[0], rtmp[1]);
	matmul_sub_kernel(m, lhsatmp, lhsctmp, lhsbtmp);
	binvrhs_kernel(m, lhsbtmp, rtmp[1]);
	rhs(m,i,ny-1,k) = rtmp[1][m];
	for (int j = ny-2; j >= 0; j--) {
		for (int n = 0; n < 5; n++) rtmp[0][m] -= lhs(m,n,cc,j)*rtmp[1][n];
		rhs(m,i,j,k) = rtmp[1][m] = rtmp[0][m];
		if (j > 0) rtmp[0][m] = rhs(m,i,j-1,k);
		__syncthreads();
	}
}
#undef lhs

void BT::y_solve () {
	dim3 grid2(nz-2,nx-2);

	START_TIMER(t_ysolve);
	y_solve_kernel_1<<<grid2,SOLVE_BLOCK>>>(rho_i, qs, square, u, rhs, lhs, nx, ny, nz); 
	y_solve_kernel_2<<<grid2,5>>>(rhs, lhs, nx, ny, nz);
	STOP_TIMER(t_ysolve);
}

//---------------------------------------------------------------------
//     Performs line solves in Z direction by first factoring the block-tridiagonal matrix into an upper triangular matrix, 
//     and then performing back substitution to solve for the unknow vectors of each line.  
//     
//     Make sure we treat elements zero to cell_size in the direction of the sweep.
//---------------------------------------------------------------------
#define lhs(a,b,c,i) lhs[(a)+5*((i)+nz*((b)+5*(c)))]
__global__ static void z_solve_kernel_1 (const double *qs, const double *square, const double *u, double *rhs, double *lhs, const int nx, const int ny, const int nz) {
	int i, j, jacofs, jacofs2, lhsofs;
	double tmp1, tmp2, tmp3, utmp[5];
	i = blockIdx.x+1;
	j = blockIdx.y+1;
	__shared__ double fjac[2*5*5*SOLVE_BLOCK];
	__shared__ double njac[2*5*5*SOLVE_BLOCK];
	lhs += (i-1+(j-1)*nx)*5*5*3*nz;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	//     This function computes the left hand side for the three z-factors   
	//     ksize = nz-1
	//---------------------------------------------------------------------
	//---------------------------------------------------------------------
	//     Compute the indices for storing the block-diagonal matrix; determine c (labeled f) and s jacobians
	//---------------------------------------------------------------------
	lhsofs = threadIdx.x-1;
	jacofs = threadIdx.x;
	while (lhsofs < nz-1) {
		jacofs2 = lhsofs+1;
		utmp[0] = u(0,i,j,jacofs2);
		utmp[1] = u(1,i,j,jacofs2);
		utmp[2] = u(2,i,j,jacofs2);
		utmp[3] = u(3,i,j,jacofs2);
		utmp[4] = u(4,i,j,jacofs2);
		tmp1 = 1.0/utmp[0];
		tmp2 = tmp1*tmp1;
		tmp3 = tmp1*tmp2;

		fjac(0,0,jacofs) = 0.0;
		fjac(0,1,jacofs) = 0.0;
		fjac(0,2,jacofs) = 0.0;
		fjac(0,3,jacofs) = 1.0;
		fjac(0,4,jacofs) = 0.0;

		fjac(1,0,jacofs) = -(utmp[1]*utmp[3]) * tmp2;
		fjac(1,1,jacofs) = utmp[3] * tmp1;
		fjac(1,2,jacofs) = 0.0;
		fjac(1,3,jacofs) = utmp[1] * tmp1;
		fjac(1,4,jacofs) = 0.0;

		fjac(2,0,jacofs) = -(utmp[2]*utmp[3]) * tmp2;
		fjac(2,1,jacofs) = 0.0;
		fjac(2,2,jacofs) = utmp[3] * tmp1;
		fjac(2,3,jacofs) = utmp[2] * tmp1;
		fjac(2,4,jacofs) = 0.0;

		fjac(3,0,jacofs) = -(utmp[3]*utmp[3]*tmp2) + c2*qs(i,j,jacofs2);
		fjac(3,1,jacofs) = -c2*utmp[1]*tmp1;
		fjac(3,2,jacofs) = -c2*utmp[2]*tmp1;
		fjac(3,3,jacofs) = (2.0-c2) * utmp[3] * tmp1;
		fjac(3,4,jacofs) = c2;

		fjac(4,0,jacofs) = (c2*2.0*square(i,j,jacofs2) - c1*utmp[4]) * utmp[3] * tmp2;
		fjac(4,1,jacofs) = -c2 * (utmp[1]*utmp[3]) * tmp2;
		fjac(4,2,jacofs) = -c2 * (utmp[2]*utmp[3]) * tmp2;
		fjac(4,3,jacofs) = c1 * (utmp[4]*tmp1) - c2 * (qs(i,j,jacofs2)+utmp[3]*utmp[3]*tmp2);
		fjac(4,4,jacofs) = c1 * utmp[3] * tmp1;

		njac(0,0,jacofs) = 0.0;
		njac(0,1,jacofs) = 0.0;
		njac(0,2,jacofs) = 0.0;
		njac(0,3,jacofs) = 0.0;
		njac(0,4,jacofs) = 0.0;

		njac(1,0,jacofs) = -c3c4 * tmp2 * utmp[1];
		njac(1,1,jacofs) = c3c4 * tmp1;
		njac(1,2,jacofs) = 0.0;
		njac(1,3,jacofs) = 0.0;
		njac(1,4,jacofs) = 0.0;

		njac(2,0,jacofs) = -c3c4 * tmp2 * utmp[2];
		njac(2,1,jacofs) = 0.0;
		njac(2,2,jacofs) = c3c4 * tmp1;
		njac(2,3,jacofs) = 0.0;
		njac(2,4,jacofs) = 0.0;

		njac(3,0,jacofs) = -con43 * c3c4 * tmp2 * utmp[3];
		njac(3,1,jacofs) = 0.0;
		njac(3,2,jacofs) = 0.0;
		njac(3,3,jacofs) = con43 * c3 * c4 * tmp1;
		njac(3,4,jacofs) = 0.0;

		njac(4,0,jacofs) = -(c3c4-c1345)*tmp3*(utmp[1]*utmp[1]) - (c3c4-c1345)*tmp3*(utmp[2]*utmp[2]) - (con43*c3c4-c1345)*tmp3*(utmp[3]*utmp[3]) - c1345*tmp2*utmp[4];
		njac(4,1,jacofs) = (c3c4-c1345)*tmp2*utmp[1];
		njac(4,2,jacofs) = (c3c4-c1345)*tmp2*utmp[2];
		njac(4,3,jacofs) = (con43*c3c4-c1345)*tmp2*utmp[3];
		njac(4,4,jacofs) = c1345 * tmp1;
		//---------------------------------------------------------------------
		//     now jacobians set, so form left hand side in z direction
		//---------------------------------------------------------------------
		__syncthreads();
		if (lhsofs < 1) {
			jacofs2 = lhsofs == 0 ? 0 : nz-1;
			for (int m = 0; m < 5; m++) {
				for (int n = 0; n < 5; n++) {
					lhs(m,n,aa,jacofs2) = 0.0;
					lhs(m,n,bb,jacofs2) = 0.0;
					lhs(m,n,cc,jacofs2) = 0.0;
				}
				lhs(m,m,bb,jacofs2) = 1.0;
			}
		} else {
			tmp1 = dt*tz1;
			tmp2 = dt*tz2;

			jacofs2 = (2*SOLVE_BLOCK+jacofs-2) % (2*SOLVE_BLOCK);
			lhs(0,0,aa,lhsofs) = -tmp2*fjac(0,0,jacofs2) - tmp1*njac(0,0,jacofs2) - tmp1*dz1;
			lhs(0,1,aa,lhsofs) = -tmp2*fjac(0,1,jacofs2) - tmp1*njac(0,1,jacofs2);
			lhs(0,2,aa,lhsofs) = -tmp2*fjac(0,2,jacofs2) - tmp1*njac(0,2,jacofs2);
			lhs(0,3,aa,lhsofs) = -tmp2*fjac(0,3,jacofs2) - tmp1*njac(0,3,jacofs2);
			lhs(0,4,aa,lhsofs) = -tmp2*fjac(0,4,jacofs2) - tmp1*njac(0,4,jacofs2);

			lhs(1,0,aa,lhsofs) = -tmp2*fjac(1,0,jacofs2) - tmp1*njac(1,0,jacofs2);
			lhs(1,1,aa,lhsofs) = -tmp2*fjac(1,1,jacofs2) - tmp1*njac(1,1,jacofs2) - tmp1*dz2;
			lhs(1,2,aa,lhsofs) = -tmp2*fjac(1,2,jacofs2) - tmp1*njac(1,2,jacofs2);
			lhs(1,3,aa,lhsofs) = -tmp2*fjac(1,3,jacofs2) - tmp1*njac(1,3,jacofs2);
			lhs(1,4,aa,lhsofs) = -tmp2*fjac(1,4,jacofs2) - tmp1*njac(1,4,jacofs2);
				
			lhs(2,0,aa,lhsofs) = -tmp2*fjac(2,0,jacofs2) - tmp1*njac(2,0,jacofs2);
			lhs(2,1,aa,lhsofs) = -tmp2*fjac(2,1,jacofs2) - tmp1*njac(2,1,jacofs2);
			lhs(2,2,aa,lhsofs) = -tmp2*fjac(2,2,jacofs2) - tmp1*njac(2,2,jacofs2) - tmp1*dz3;
			lhs(2,3,aa,lhsofs) = -tmp2*fjac(2,3,jacofs2) - tmp1*njac(2,3,jacofs2);
			lhs(2,4,aa,lhsofs) = -tmp2*fjac(2,4,jacofs2) - tmp1*njac(2,4,jacofs2);

			lhs(3,0,aa,lhsofs) = -tmp2*fjac(3,0,jacofs2) - tmp1*njac(3,0,jacofs2);
			lhs(3,1,aa,lhsofs) = -tmp2*fjac(3,1,jacofs2) - tmp1*njac(3,1,jacofs2);
			lhs(3,2,aa,lhsofs) = -tmp2*fjac(3,2,jacofs2) - tmp1*njac(3,2,jacofs2);
			lhs(3,3,aa,lhsofs) = -tmp2*fjac(3,3,jacofs2) - tmp1*njac(3,3,jacofs2) - tmp1*dz4;
			lhs(3,4,aa,lhsofs) = -tmp2*fjac(3,4,jacofs2) - tmp1*njac(3,4,jacofs2);

			lhs(4,0,aa,lhsofs) = -tmp2*fjac(4,0,jacofs2) - tmp1*njac(4,0,jacofs2);
			lhs(4,1,aa,lhsofs) = -tmp2*fjac(4,1,jacofs2) - tmp1*njac(4,1,jacofs2);
			lhs(4,2,aa,lhsofs) = -tmp2*fjac(4,2,jacofs2) - tmp1*njac(4,2,jacofs2);
			lhs(4,3,aa,lhsofs) = -tmp2*fjac(4,3,jacofs2) - tmp1*njac(4,3,jacofs2);
			lhs(4,4,aa,lhsofs) = -tmp2*fjac(4,4,jacofs2) - tmp1*njac(4,4,jacofs2) - tmp1*dz5;

			jacofs2 = (jacofs2+1) % (2*SOLVE_BLOCK);
			lhs(0,0,bb,lhsofs) = 1.0 + tmp1*2.0*njac(0,0,jacofs2) + tmp1*2.0*dz1;
			lhs(0,1,bb,lhsofs) = tmp1*2.0*njac(0,1,jacofs2);
			lhs(0,2,bb,lhsofs) = tmp1*2.0*njac(0,2,jacofs2);
			lhs(0,3,bb,lhsofs) = tmp1*2.0*njac(0,3,jacofs2);
			lhs(0,4,bb,lhsofs) = tmp1*2.0*njac(0,4,jacofs2);

			lhs(1,0,bb,lhsofs) = tmp1*2.0*njac(1,0,jacofs2);
			lhs(1,1,bb,lhsofs) = 1.0 + tmp1*2.0*njac(1,1,jacofs2) + tmp1*2.0*dz2;
			lhs(1,2,bb,lhsofs) = tmp1*2.0*njac(1,2,jacofs2);
			lhs(1,3,bb,lhsofs) = tmp1*2.0*njac(1,3,jacofs2);
			lhs(1,4,bb,lhsofs) = tmp1*2.0*njac(1,4,jacofs2);

			lhs(2,0,bb,lhsofs) = tmp1*2.0*njac(2,0,jacofs2);
			lhs(2,1,bb,lhsofs) = tmp1*2.0*njac(2,1,jacofs2);
			lhs(2,2,bb,lhsofs) = 1.0 + tmp1*2.0*njac(2,2,jacofs2) + tmp1*2.0*dz3;
			lhs(2,3,bb,lhsofs) = tmp1*2.0*njac(2,3,jacofs2);
			lhs(2,4,bb,lhsofs) = tmp1*2.0*njac(2,4,jacofs2);

			lhs(3,0,bb,lhsofs) = tmp1*2.0*njac(3,0,jacofs2);
			lhs(3,1,bb,lhsofs) = tmp1*2.0*njac(3,1,jacofs2);
			lhs(3,2,bb,lhsofs) = tmp1*2.0*njac(3,2,jacofs2);
			lhs(3,3,bb,lhsofs) = 1.0 + tmp1*2.0*njac(3,3,jacofs2) + tmp1*2.0*dz4;
			lhs(3,4,bb,lhsofs) = tmp1*2.0*njac(3,4,jacofs2);

			lhs(4,0,bb,lhsofs) = tmp1*2.0*njac(4,0,jacofs2);
			lhs(4,1,bb,lhsofs) = tmp1*2.0*njac(4,1,jacofs2);
			lhs(4,2,bb,lhsofs) = tmp1*2.0*njac(4,2,jacofs2);
			lhs(4,3,bb,lhsofs) = tmp1*2.0*njac(4,3,jacofs2);
			lhs(4,4,bb,lhsofs) = 1.0 + tmp1*2.0*njac(4,4,jacofs2) + tmp1*2.0*dz5;

			jacofs2 = (jacofs2+1) % (2*SOLVE_BLOCK);
			lhs(0,0,cc,lhsofs) = tmp2*fjac(0,0,jacofs2) - tmp1*njac(0,0,jacofs2) - tmp1*dz1;
			lhs(0,1,cc,lhsofs) = tmp2*fjac(0,1,jacofs2) - tmp1*njac(0,1,jacofs2);
			lhs(0,2,cc,lhsofs) = tmp2*fjac(0,2,jacofs2) - tmp1*njac(0,2,jacofs2);
			lhs(0,3,cc,lhsofs) = tmp2*fjac(0,3,jacofs2) - tmp1*njac(0,3,jacofs2);
			lhs(0,4,cc,lhsofs) = tmp2*fjac(0,4,jacofs2) - tmp1*njac(0,4,jacofs2);

			lhs(1,0,cc,lhsofs) = tmp2*fjac(1,0,jacofs2) - tmp1*njac(1,0,jacofs2);
			lhs(1,1,cc,lhsofs) = tmp2*fjac(1,1,jacofs2) - tmp1*njac(1,1,jacofs2) - tmp1*dz2;
			lhs(1,2,cc,lhsofs) = tmp2*fjac(1,2,jacofs2) - tmp1*njac(1,2,jacofs2);
			lhs(1,3,cc,lhsofs) = tmp2*fjac(1,3,jacofs2) - tmp1*njac(1,3,jacofs2);
			lhs(1,4,cc,lhsofs) = tmp2*fjac(1,4,jacofs2) - tmp1*njac(1,4,jacofs2);

			lhs(2,0,cc,lhsofs) = tmp2*fjac(2,0,jacofs2) - tmp1*njac(2,0,jacofs2);
			lhs(2,1,cc,lhsofs) = tmp2*fjac(2,1,jacofs2) - tmp1*njac(2,1,jacofs2);
			lhs(2,2,cc,lhsofs) = tmp2*fjac(2,2,jacofs2) - tmp1*njac(2,2,jacofs2) - tmp1*dz3;
			lhs(2,3,cc,lhsofs) = tmp2*fjac(2,3,jacofs2) - tmp1*njac(2,3,jacofs2);
			lhs(2,4,cc,lhsofs) = tmp2*fjac(2,4,jacofs2) - tmp1*njac(2,4,jacofs2);

			lhs(3,0,cc,lhsofs) = tmp2*fjac(3,0,jacofs2) - tmp1*njac(3,0,jacofs2);
			lhs(3,1,cc,lhsofs) = tmp2*fjac(3,1,jacofs2) - tmp1*njac(3,1,jacofs2);
			lhs(3,2,cc,lhsofs) = tmp2*fjac(3,2,jacofs2) - tmp1*njac(3,2,jacofs2);
			lhs(3,3,cc,lhsofs) = tmp2*fjac(3,3,jacofs2) - tmp1*njac(3,3,jacofs2) - tmp1*dz4;
			lhs(3,4,cc,lhsofs) = tmp2*fjac(3,4,jacofs2) - tmp1*njac(3,4,jacofs2);

			lhs(4,0,cc,lhsofs) = tmp2*fjac(4,0,jacofs2) - tmp1*njac(4,0,jacofs2);
			lhs(4,1,cc,lhsofs) = tmp2*fjac(4,1,jacofs2) - tmp1*njac(4,1,jacofs2);
			lhs(4,2,cc,lhsofs) = tmp2*fjac(4,2,jacofs2) - tmp1*njac(4,2,jacofs2);
			lhs(4,3,cc,lhsofs) = tmp2*fjac(4,3,jacofs2) - tmp1*njac(4,3,jacofs2);
			lhs(4,4,cc,lhsofs) = tmp2*fjac(4,4,jacofs2) - tmp1*njac(4,4,jacofs2) - tmp1*dz5;
		}
		lhsofs += SOLVE_BLOCK;
		jacofs = (jacofs + SOLVE_BLOCK) % (2*SOLVE_BLOCK);
	}
}

__global__ static void z_solve_kernel_2 (double *rhs, double *lhs, const int nx, const int ny, const int nz) {
	int i, j, m;
	i = blockIdx.x+1;
	j = blockIdx.y+1;
	m = threadIdx.x;
	lhs += (i-1+(j-1)*nx)*5*5*3*nz;
	__shared__ double rtmp[2][5];
	__shared__ double lhsbtmp[5*5], lhsctmp[5*5], lhsatmp[5*5];

	// copy date to shared memory
	rtmp[0][m] = rhs(m,i,j,0);
	for (int n = 0; n < 5; n++) {
		lhsbtmp[m+5*n] = lhs(m,n,bb,0);
		lhsctmp[m+5*n] = lhs(m,n,cc,0);
	}
	__syncthreads();
	binvcrhs_kernel (m, lhsbtmp, lhsctmp, rtmp[0]);
	for (int n = 0; n < 5; n++) lhs(m,n,cc,0) = lhsctmp[m+5*n];
	for (int k = 1; k < nz-1; k++) {
		rtmp[1][m] = rhs(m,i,j,k);
		for (int n = 0; n < 5; n++) {
			lhsatmp[m+5*n] = lhs(m,n,aa,k);
			lhsbtmp[m+5*n] = lhs(m,n,bb,k);
		}
		__syncthreads();
		matvec_sub_kernel (m, lhsatmp, rtmp[0], rtmp[1]);
		matmul_sub_kernel (m, lhsatmp, lhsctmp, lhsbtmp);
		for (int n = 0; n < 5; n++) lhsctmp[m+5*n] = lhs(m,n,cc,k);
		__syncthreads();
		binvcrhs_kernel (m, lhsbtmp, lhsctmp, rtmp[1]);
		for (int n = 0; n < 5; n++) lhs(m,n,cc,k) = lhsctmp[m+5*n];
		rhs(m,i,j,k-1) = rtmp[0][m];
		rtmp[0][m] = rtmp[1][m];
	}
	rtmp[1][m] = rhs(m,i,j,nz-1);
	for (int n = 0; n < 5; n++) {
		lhsatmp[m+5*n] = lhs(m,n,aa,nz-1);
		lhsbtmp[m+5*n] = lhs(m,n,bb,nz-1);
	}
	__syncthreads();
	matvec_sub_kernel (m, lhsatmp, rtmp[0], rtmp[1]);
	matmul_sub_kernel (m, lhsatmp, lhsctmp, lhsbtmp);
	binvrhs_kernel (m, lhsbtmp, rtmp[1]);
	rhs(m,i,j,nz-1) = rtmp[1][m];
	for (int k = nz-2; k >= 0; k--) {
		for (int n = 0; n < 5; n++) rtmp[0][m] -= lhs(m,n,cc,k)*rtmp[1][n];
		rhs(m,i,j,k) = rtmp[1][m] = rtmp[0][m];
		if (k > 0) rtmp[0][m] = rhs(m,i,j,k-1);
		__syncthreads();
	}
}
#undef lhs

void BT::z_solve () {
	dim3 grid2(nx-2,ny-2);

	START_TIMER(t_zsolve);
	z_solve_kernel_1<<<grid2,SOLVE_BLOCK>>>(qs, square, u, rhs, lhs, nx, ny, nz);  
	z_solve_kernel_2<<<grid2,5>>>(rhs, lhs, nx, ny, nz);
	STOP_TIMER(t_zsolve);
}

//---------------------------------------------------------------------
//     this function returns the exact solution at point xi, eta, zeta  
//---------------------------------------------------------------------
__device__ static void exact_solution_kernel (const double xi, const double eta, const double zeta, double *dtemp) {
	using namespace gpu_mod;

	for (int m = 0; m < 5; m++) 
		dtemp[m] = ce[0][m] + xi*(ce[1][m] + xi*(ce[4][m] + xi*(ce[7][m] + xi*ce[10][m]))) + eta*(ce[2][m] + eta*(ce[5][m] + eta*(ce[8][m] + eta*ce[11][m])))+zeta*(ce[3][m] + zeta*(ce[6][m] + zeta*(ce[9][m] + zeta*ce[12][m])));
}

//---------------------------------------------------------------------
//     compute the right hand side based on exact solution
//---------------------------------------------------------------------
__global__ static void exact_rhs_kernel_init (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	k = blockIdx.y;
	j = blockIdx.x;
	i = threadIdx.x;
	for (m = 0; m < 5; m++) forcing(m,i,j,k) = 0.0;
}

__global__ static void exact_rhs_kernel_x (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, dtpp, dtemp[5];
	double ue[5][5], buf[3][5], cuf[3], q[3];

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	j = blockIdx.y*blockDim.y+threadIdx.y+1;

	if (k >= nz-1 || j >= ny-1) return;

	using namespace gpu_mod;

	zeta = (double)k * dnzm1;
	eta = (double)j * dnym1;
	//---------------------------------------------------------------------
	//      xi-direction flux differences                      
	//---------------------------------------------------------------------
	for (i = 0; i < 3; i++) {
		xi = (double)i * dnxm1;
		exact_solution_kernel(xi, eta, zeta, dtemp);
		for (m = 0; m < 5; m++) ue[i+1][m] = dtemp[m];
		dtpp = 1.0/dtemp[0];
		for (m = 1; m < 5; m++) buf[i][m] = dtpp*dtemp[m];
		cuf[i] = buf[i][1] * buf[i][1];
		buf[i][0] = cuf[i] + buf[i][2] * buf[i][2] + buf[i][3] * buf[i][3];
		q[i] = 0.5 * (buf[i][1]*ue[i+1][1] + buf[i][2]*ue[i+1][2] + buf[i][3]*ue[i+1][3]);
	}
	for (i = 1; i < nx-1; i++) {
		if (i+2 < nx) {
			xi = (double)(i+2) * dnxm1;
			exact_solution_kernel(xi, eta, zeta, dtemp);
			for (m = 0; m < 5; m++) ue[4][m] = dtemp[m];
		}

		dtemp[0] = 0.0 - tx2*(ue[3][1]-ue[1][1])+ dx1tx1*(ue[3][0]-2.0*ue[2][0]+ue[1][0]);
		dtemp[1] = 0.0 - tx2*((ue[3][1]*buf[2][1]+c2*(ue[3][4]-q[2]))-(ue[1][1]*buf[0][1]+c2*(ue[1][4]-q[0])))+xxcon1*(buf[2][1]-2.0*buf[1][1]+buf[0][1])+dx2tx1*(ue[3][1]-2.0*ue[2][1]+ue[1][1]);
		dtemp[2] = 0.0 - tx2*(ue[3][2]*buf[2][1]-ue[1][2]*buf[0][1])+xxcon2*(buf[2][2]-2.0*buf[1][2]+buf[0][2])+dx3tx1*(ue[3][2]-2.0*ue[2][2]+ue[1][2]);
		dtemp[3] = 0.0 - tx2*(ue[3][3]*buf[2][1]-ue[1][3]*buf[0][1])+xxcon2*(buf[2][3]-2.0*buf[1][3]+buf[0][3])+dx4tx1*(ue[3][3]-2.0*ue[2][3]+ue[1][3]);
		dtemp[4] = 0.0 - tx2*(buf[2][1]*(c1*ue[3][4]-c2*q[2])-buf[0][1]*(c1*ue[1][4]-c2*q[0]))+0.5*xxcon3*(buf[2][0]-2.0*buf[1][0]+buf[0][0])+xxcon4*(cuf[2]-2.0*cuf[1]+cuf[0])+
					xxcon5*(buf[2][4]-2.0*buf[1][4]+buf[0][4])+dx5tx1*(ue[3][4]-2.0*ue[2][4]+ ue[1][4]);

		//---------------------------------------------------------------------
		//            Fourth-order dissipation                         
		//---------------------------------------------------------------------
		if (i == 1) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(5.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (i == 2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(-4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (i >= 3 && i < nx-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m]+6.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (i == nx-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m] +6.0*ue[2][m] - 4.0*ue[3][m]);
		} else if (i == nx-2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m] + 5.0*ue[2][m]);
		}

		for (m = 0; m < 5; m++) {
			ue[0][m] = ue[1][m]; 
			ue[1][m] = ue[2][m];
			ue[2][m] = ue[3][m];
			ue[3][m] = ue[4][m];
			buf[0][m] = buf[1][m];
			buf[1][m] = buf[2][m];
		}
		cuf[0] = cuf[1]; cuf[1] = cuf[2];
		q[0] = q[1]; q[1] = q[2];

		if (i < nx-2) {
			dtpp = 1.0/ue[3][0];
			for (m = 1; m < 5; m++) buf[2][m] = dtpp*ue[3][m];
			cuf[2] = buf[2][1] * buf[2][1];
			buf[2][0] = cuf[2] + buf[2][2] * buf[2][2] + buf[2][3] * buf[2][3];
			q[2] = 0.5 * (buf[2][1]*ue[3][1] + buf[2][2]*ue[3][2] + buf[2][3]*ue[3][3]);
		}
	}
}

__global__ static void exact_rhs_kernel_y (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, dtpp, dtemp[5];
	double ue[5][5], buf[3][5], cuf[5], q[5];

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (k >= nz-1 || i >= nx-1) return;
	
	using namespace gpu_mod;

	zeta = (double)k * dnzm1;
	xi = (double)i * dnxm1;
	//---------------------------------------------------------------------
	//  eta-direction flux differences             
	//---------------------------------------------------------------------
	for (j = 0; j < 3; j++) {
		eta = (double)j * dnym1;
		exact_solution_kernel(xi, eta, zeta, dtemp);
		for (m = 0; m < 5; m++) ue[j+1][m] = dtemp[m];;
		dtpp = 1.0/dtemp[0];
		for (m = 1; m < 5; m++) buf[j][m] = dtpp * dtemp[m];
		cuf[j] = buf[j][2] * buf[j][2];
		buf[j][0] = cuf[j] + buf[j][1] * buf[j][1] + buf[j][3] * buf[j][3];
		q[j] = 0.5*(buf[j][1]*ue[j+1][1] + buf[j][2]*ue[j+1][2] + buf[j][3]*ue[j+1][3]);
	}
	for (j = 1; j < ny-1; j++) {
		if (j+2 < ny) {
			eta = (double)(j+2) * dnym1;
			exact_solution_kernel(xi, eta, zeta, dtemp);
			for (m = 0; m < 5; m++) ue[4][m] = dtemp[m];
		}

		dtemp[0] = forcing(0,i,j,k) - ty2*(ue[3][2]-ue[1][2])+ dy1ty1*(ue[3][0]-2.0*ue[2][0]+ue[1][0]);
		dtemp[1] = forcing(1,i,j,k) - ty2*(ue[3][1]*buf[2][2]-ue[1][1]*buf[0][2])+yycon2*(buf[2][1]-2.0*buf[1][1]+buf[0][1])+dy2ty1*(ue[3][1]-2.0*ue[2][1]+ ue[1][1]);
		dtemp[2] = forcing(2,i,j,k) - ty2*((ue[3][2]*buf[2][2]+c2*(ue[3][4]-q[2]))-(ue[1][2]*buf[0][2]+c2*(ue[1][4]-q[0])))+yycon1*(buf[2][2]-2.0*buf[1][2]+buf[0][2])+dy3ty1*(ue[3][2]-2.0*ue[2][2] +ue[1][2]);
		dtemp[3] = forcing(3,i,j,k) - ty2*(ue[3][3]*buf[2][2]-ue[1][3]*buf[0][2])+yycon2*(buf[2][3]-2.0*buf[1][3]+buf[0][3])+dy4ty1*(ue[3][3]-2.0*ue[2][3]+ ue[1][3]);
		dtemp[4] = forcing(4,i,j,k) - ty2*(buf[2][2]*(c1*ue[3][4]-c2*q[2])-buf[0][2]*(c1*ue[1][4]-c2*q[0]))+0.5*yycon3*(buf[2][0]-2.0*buf[1][0]+buf[0][0])+yycon4*(cuf[2]-2.0*cuf[1]+cuf[0])+
					yycon5*(buf[2][4]-2.0*buf[1][4]+buf[0][4])+dy5ty1*(ue[3][4]-2.0*ue[2][4]+ue[1][4]);
		//---------------------------------------------------------------------
		//            Fourth-order dissipation                      
		//---------------------------------------------------------------------
		if (j == 1) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (5.0*ue[2][m] - 4.0*ue[3][m] +ue[4][m]);
		} else if (j == 2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (-4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m] +       ue[4][m]);
		} else if (j >= 3 && j < ny-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (j == ny-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (ue[0][m] - 4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m]);
		} else if (j == ny-2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (ue[0][m] - 4.0*ue[1][m] + 5.0*ue[2][m]);
		}

		for (m = 0; m < 5; m++) {
			ue[0][m] = ue[1][m]; 
			ue[1][m] = ue[2][m];
			ue[2][m] = ue[3][m];
			ue[3][m] = ue[4][m];
			buf[0][m] = buf[1][m];
			buf[1][m] = buf[2][m];
		}
		cuf[0] = cuf[1]; cuf[1] = cuf[2];
		q[0] = q[1]; q[1] = q[2];

		if (j < ny-2) {
			dtpp = 1.0/ue[3][0];
			for (m = 1; m < 5; m++) buf[2][m] = dtpp * ue[3][m];
			cuf[2] = buf[2][2] * buf[2][2];
			buf[2][0] = cuf[2] + buf[2][1] * buf[2][1] + buf[2][3] * buf[2][3];
			q[2] = 0.5*(buf[2][1]*ue[3][1] + buf[2][2]*ue[3][2] + buf[2][3]*ue[3][3]);
		}
	}
}

__global__ static void exact_rhs_kernel_z (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, dtpp, dtemp[5];
	double ue[5][5], buf[3][5], cuf[3], q[3];

	j = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (j >= ny-1 || i >= nx-1) return;

	using namespace gpu_mod;

	eta = (double)j * dnym1;
	xi = (double)i * dnxm1;
	//---------------------------------------------------------------------
	//      zeta-direction flux differences                      
	//---------------------------------------------------------------------
	for (k = 0; k < 3; k++) {
		zeta = (double)k * dnzm1;
		exact_solution_kernel(xi, eta, zeta, dtemp);
		for (m = 0; m < 5; m++) ue[k+1][m] = dtemp[m];
		dtpp = 1.0/dtemp[0];
		for (m = 1; m < 5; m++) buf[k][m] = dtpp * dtemp[m];
		cuf[k] = buf[k][3] * buf[k][3];
		buf[k][0] = cuf[k] + buf[k][1] * buf[k][1] + buf[k][2] * buf[k][2];
		q[k] = 0.5*(buf[k][1]*ue[k+1][1] + buf[k][2]*ue[k+1][2] + buf[k][3]*ue[k+1][3]);
	}

	for (k = 1; k < nz-1; k++) {
		if (k+2 < nz) {
			zeta = (double)(k+2) * dnzm1;
			exact_solution_kernel(xi, eta, zeta, dtemp);
			for (m = 0; m < 5; m++) ue[4][m] = dtemp[m];
		}

		dtemp[0] = forcing(0,i,j,k) - tz2*(ue[3][3]-ue[1][3])+dz1tz1*(ue[3][0]-2.0*ue[2][0]+ue[1][0]);
		dtemp[1] = forcing(1,i,j,k) - tz2*(ue[3][1]*buf[2][3]-ue[1][1]*buf[0][3])+zzcon2*(buf[2][1]-2.0*buf[1][1]+buf[0][1])+dz2tz1*(ue[3][1]-2.0*ue[2][1]+ue[1][1]);
		dtemp[2] = forcing(2,i,j,k) - tz2*(ue[3][2]*buf[2][3]-ue[1][2]*buf[0][3])+zzcon2*(buf[2][2]-2.0*buf[1][2]+buf[0][2])+dz3tz1*(ue[3][2]-2.0*ue[2][2]+ue[1][2]);
		dtemp[3] = forcing(3,i,j,k) - tz2*((ue[3][3]*buf[2][3]+c2*(ue[3][4]-q[2]))-(ue[1][3]*buf[0][3]+c2*(ue[1][4]-q[0])))+zzcon1*(buf[2][3]-2.0*buf[1][3]+buf[0][3])+dz4tz1*(ue[3][3]-2.0*ue[2][3] +ue[1][3]);
		dtemp[4] = forcing(4,i,j,k) - tz2*(buf[2][3]*(c1*ue[3][4]-c2*q[2])-buf[0][3]*(c1*ue[1][4]-c2*q[0]))+0.5*zzcon3*(buf[2][0]-2.0*buf[1][0]+buf[0][0])+
					zzcon4*(cuf[2]-2.0*cuf[1]+cuf[0])+zzcon5*(buf[2][4]-2.0*buf[1][4]+buf[0][4])+dz5tz1*(ue[3][4]-2.0*ue[2][4]+ue[1][4]);
		//---------------------------------------------------------------------
		//            Fourth-order dissipation
		//---------------------------------------------------------------------
		if (k == 1) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(5.0*ue[2][m]-4.0*ue[3][m]+ue[4][m]);
		} else if (k == 2) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(-4.0*ue[1][m]+6.0*ue[2][m]-4.0*ue[3][m]+ue[4][m]);
		} else if (k >= 3 && k < nz-3) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(ue[0][m]-4.0*ue[1][m]+6.0*ue[2][m]-4.0*ue[3][m]+ue[4][m]);
		} else if (k == nz-3) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(ue[0][m]-4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m]);
		} else if (k == nz-2) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(ue[0][m]-4.0*ue[1][m]+5.0*ue[2][m]);
		}
		//---------------------------------------------------------------------
		// now change the sign of the forcing function, 
		//---------------------------------------------------------------------
		for (m = 0; m < 5; m++) forcing(m,i,j,k) = -1.0 * dtemp[m];

		for (m = 0; m < 5; m++) {
			ue[0][m] = ue[1][m]; 
			ue[1][m] = ue[2][m];
			ue[2][m] = ue[3][m];
			ue[3][m] = ue[4][m];
			buf[0][m] = buf[1][m];
			buf[1][m] = buf[2][m];
		}
		cuf[0] = cuf[1]; cuf[1] = cuf[2];
		q[0] = q[1]; q[1] = q[2];

		if (k < nz-2) {
			dtpp = 1.0/ue[3][0];
			for (m = 1; m < 5; m++) buf[2][m] = dtpp * ue[3][m];
			cuf[2] = buf[2][3] * buf[2][3];
			buf[2][0] = cuf[2] + buf[2][1] * buf[2][1] + buf[2][2] * buf[2][2];
			q[2] = 0.5*(buf[2][1]*ue[3][1] + buf[2][2]*ue[3][2] + buf[2][3]*ue[3][3]);
		}
	}
}

void BT::exact_rhs () {
	dim3 gridyz(ny,nz);
	exact_rhs_kernel_init<<<gridyz,nx>>>(forcing, nx, ny, nz);

	int yblock = min(ERHS_BLOCK,ny-2);
	int ygrid = (ny-2+yblock-1)/yblock;
	int zblock_y = min(ERHS_BLOCK/yblock,nz-2);
	int zgrid_y = (nz-2+zblock_y-1)/zblock_y;
	dim3 grid_x(zgrid_y,ygrid), block_x(zblock_y,yblock);
	exact_rhs_kernel_x<<<grid_x,block_x>>>(forcing, nx, ny, nz);

	int xblock = min(ERHS_BLOCK,nx-2);
	int xgrid = (nx-2+xblock-1)/xblock;
	int zblock_x = min(ERHS_BLOCK/xblock,nz-2);
	int zgrid_x = (nz-2+zblock_x-1)/zblock_x;
	dim3 grid_y(zgrid_x,xgrid), block_y(zblock_x,xblock);
	exact_rhs_kernel_y<<<grid_y,block_y>>>(forcing, nx, ny, nz);

	int yblock_x = min(ERHS_BLOCK/xblock,ny-2);
	int ygrid_x = (ny-2+yblock_x-1)/yblock_x;
	dim3 grid_z(ygrid_x,xgrid), block_z(yblock_x,xblock);
	exact_rhs_kernel_z<<<grid_z,block_z>>>(forcing, nx, ny, nz);
}

//---------------------------------------------------------------------
// This subroutine initializes the field variable u using 
// tri-linear transfinite interpolation of the boundary values     
//---------------------------------------------------------------------
__global__ static void initialize_kernel (double *u, const int nx, const int ny, const int nz) {
	int i, j, k;
	double xi, eta, zeta, temp[5], Pxi, Peta, Pzeta;
	double Pface11[5], Pface12[5], Pface21[5], Pface22[5], Pface31[5], Pface32[5];

	k = blockIdx.x;
	j = blockIdx.y;
	i = threadIdx.x;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	//  Later (in compute_rhs) we compute 1/u for every element. A few of 
	//  the corner elements are not used, but it convenient (and faster) 
	//  to compute the whole thing with a simple loop. Make sure those 
	//  values are nonzero by initializing the whole thing here. 
	//---------------------------------------------------------------------
	for (int m = 0; m < 5; m++) u(m,i,j,k) = 1.0;

	//---------------------------------------------------------------------
	// first store the "interpolated" values everywhere on the zone    
	//---------------------------------------------------------------------
	zeta = (double)k * dnzm1;
	eta = (double)j * dnym1;
	xi = (double)i * dnxm1;
	exact_solution_kernel (0.0, eta, zeta, Pface11);
	exact_solution_kernel (1.0, eta, zeta, Pface12);
	exact_solution_kernel (xi, 0.0, zeta, Pface21);
	exact_solution_kernel (xi, 1.0, zeta, Pface22);
	exact_solution_kernel (xi, eta, 0.0, Pface31);
	exact_solution_kernel (xi, eta, 1.0, Pface32);
	for (int m = 0; m < 5; m++) {
		Pxi = xi * Pface12[m] + (1.0-xi)*Pface11[m];
		Peta = eta * Pface22[m] + (1.0-eta)*Pface21[m];
		Pzeta = zeta * Pface32[m] + (1.0-zeta)*Pface31[m];
		u(m,i,j,k) = Pxi + Peta + Pzeta - Pxi*Peta - Pxi*Pzeta - Peta*Pzeta + Pxi*Peta*Pzeta;
	}

	//---------------------------------------------------------------------
	// now store the exact values on the boundaries        
	//---------------------------------------------------------------------

	//---------------------------------------------------------------------
	// west face                                                  
	//---------------------------------------------------------------------
	xi = 0.0;
	if (i == 0) {
		zeta = (double)k * dnzm1;
		eta = (double)j * dnym1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (int m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}

	//---------------------------------------------------------------------
	// east face                                                      
	//---------------------------------------------------------------------
	xi = 1.0;
	if (i == nx-1) {
		zeta = (double)k * dnzm1;
		eta = (double)j * dnym1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (int m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}

	//---------------------------------------------------------------------
	// south face                                                 
	//---------------------------------------------------------------------
	eta = 0.0;
	if (j == 0) {
		zeta = (double)k * dnzm1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi,eta,zeta,temp);
		for (int m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}

	//---------------------------------------------------------------------
	// north face                                    
	//---------------------------------------------------------------------
	eta = 1.0;
	if (j == ny-1) {
		zeta = (double)k * dnzm1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi,eta,zeta,temp);
		for (int m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}

	//---------------------------------------------------------------------
	// bottom face                                       
	//---------------------------------------------------------------------
	zeta = 0.0;
	if (k == 0) {
		eta = (double)j * dnym1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (int m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}

	//---------------------------------------------------------------------
	// top face     
	//---------------------------------------------------------------------
	zeta = 1.0;
	if (k == nz-1) {
		eta = (double)j * dnym1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (int m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
}

void BT::initialize() {
	dim3 grid(nz,ny);

	initialize_kernel<<<grid,nx>>>(u, nx, ny, nz);
}

//---------------------------------------------------------------------
//     this function computes the norm of the difference between the
//     computed solution and the exact solution
//---------------------------------------------------------------------
__global__ static void error_norm_kernel (double *rms, const double *u, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, u_exact[5], rms_loc[5];

	j = blockIdx.x*blockDim.x+threadIdx.x;
	i = blockIdx.y*blockDim.y+threadIdx.y;
	if (j >= ny || i >= nx) return;

	using namespace gpu_mod;

	for (m = 0; m < 5; m++) rms_loc[m] = 0.0;

	xi = (double)i * dnxm1;
	eta = (double)j * dnym1;

	for (k = 0; k < nz; k++) {
		zeta = (double)k * dnzm1;
		exact_solution_kernel (xi, eta, zeta, u_exact);
		for (m = 0; m < 5; m++) {
			double add = u(m,i,j,k) - u_exact[m];
			rms_loc[m] += add*add;
		}
	}

	for (m = 0; m < 5; m++) rms[i+nx*(j+ny*m)] = rms_loc[m];
}

__global__ static void reduce_norm_kernel (double *rms, const int nx, const int ny, const int nz) {
	int i, m, maxpos, dist;
	__shared__ double buffer[NORM_BLOCK][5];

	i = threadIdx.x;
	for (m = 0; m < 5; m++) buffer[i][m] = 0.0;

	while (i < nx*ny) {
		for (m = 0; m < 5; m++) buffer[threadIdx.x][m] += rms[i+nx*ny*m];
		i += blockDim.x;
	}

	maxpos = blockDim.x;
	dist = (maxpos+1)/2;
	i = threadIdx.x;
	__syncthreads();
	while (maxpos > 1) {
		if (i < dist && i+dist < maxpos)
			for (m = 0; m < 5; m++) buffer[i][m] += buffer[i+dist][m];
		maxpos = dist;
		dist = (dist+1)/2;
		__syncthreads();
	}
	
	m = threadIdx.x;
	if (m < 5) rms[m] = sqrt(buffer[0][m]/((double)(nz-2)*(double)(ny-2)*(double)(nx-2)));
}

void BT::error_norm () {
	int xblock = min(64,nx);
	int xgrid = (nx+xblock-1)/xblock;
	int yblock = min(64/xblock,ny);
	int ygrid = (ny+yblock-1)/yblock;
	dim3 grid(ygrid,xgrid), block(yblock,xblock);

	error_norm_kernel<<<grid,block>>>(rmsbuf, u, nx, ny, nz);
	reduce_norm_kernel<<<1,NORM_BLOCK>>>(rmsbuf, nx, ny, nz);
	HANDLE_ERROR(hipMemcpy(xce, rmsbuf, 5*sizeof(double), hipMemcpyDeviceToHost));
}

__global__ static void rhs_norm_kernel (double *rms, const double *rhs, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double rms_loc[5];

	j = blockIdx.x*blockDim.x+threadIdx.x;
	i = blockIdx.y*blockDim.y+threadIdx.y;
	if (j >= ny || i >= nx) return;

	for (m = 0; m < 5; m++) rms_loc[m] = 0.0;
	if (i >= 1 && i < nx-1 && j >= 1 && j < ny-1) {
		for (k = 1; k < nz-1; k++) {
			for (int m = 0; m < 5; m++) {
				double add = rhs(m,i,j,k);
				rms_loc[m] += add*add;
			}
		}
	}

	for (m = 0; m < 5; m++) rms[i+nx*(j+ny*m)] = rms_loc[m];

}
void BT::rhs_norm () {
	int xblock = min(64,nx);
	int xgrid = (nx+xblock-1)/xblock;
	int yblock = min(64/xblock,ny);
	int ygrid = (ny+yblock-1)/yblock;
	dim3 grid(ygrid,xgrid), block(yblock,xblock);

	rhs_norm_kernel<<<grid,block>>>(rmsbuf, rhs, nx, ny, nz);
	reduce_norm_kernel<<<1,NORM_BLOCK>>>(rmsbuf, nx, ny, nz);

	HANDLE_ERROR(hipMemcpy(xcr, rmsbuf, 5*sizeof(double), hipMemcpyDeviceToHost));
}

void BT::set_constants() {

	double ce[13][5];
	ce[0][0] = 2.0;
	ce[1][0] = 0.0;
	ce[2][0] = 0.0;
	ce[3][0] = 4.0;
	ce[4][0] = 5.0;
	ce[5][0] = 3.0;
	ce[6][0] = 0.5;
	ce[7][0] = 0.02;
	ce[8][0] = 0.01;
	ce[9][0] = 0.03;
	ce[10][0] = 0.5;
	ce[11][0] = 0.4;
	ce[12][0] = 0.3;

	ce[0][1] = 1.0;
	ce[1][1] = 0.0;
	ce[2][1] = 0.0;
	ce[3][1] = 0.0;
	ce[4][1] = 1.0;
	ce[5][1] = 2.0;
	ce[6][1] = 3.0;
	ce[7][1] = 0.01;
	ce[8][1] = 0.03;
	ce[9][1] = 0.02;
	ce[10][1] = 0.4;
	ce[11][1] = 0.3;
	ce[12][1] = 0.5;

	ce[0][2] = 2.0;
	ce[1][2] = 2.0;
	ce[2][2] = 0.0;
	ce[3][2] = 0.0;
	ce[4][2] = 0.0;
	ce[5][2] = 2.0;
	ce[6][2] = 3.0;
	ce[7][2] = 0.04;
	ce[8][2] = 0.03;
	ce[9][2] = 0.05;
	ce[10][2] = 0.3;
	ce[11][2] = 0.5;
	ce[12][2] = 0.4;

	ce[0][3] = 2.0;
	ce[1][3] = 2.0;
	ce[2][3] = 0.0;
	ce[3][3] = 0.0;
	ce[4][3] = 0.0;
	ce[5][3] = 2.0;
	ce[6][3] = 3.0;
	ce[7][3] = 0.03;
	ce[8][3] = 0.05;
	ce[9][3] = 0.04;
	ce[10][3] = 0.2;
	ce[11][3] = 0.1;
	ce[12][3] = 0.3;

	ce[0][4] = 5.0;
	ce[1][4] = 4.0;
	ce[2][4] = 3.0;
	ce[3][4] = 2.0;
	ce[4][4] = 0.1;
	ce[5][4] = 0.4;
	ce[6][4] = 0.3;
	ce[7][4] = 0.05;
	ce[8][4] = 0.04;
	ce[9][4] = 0.03;
	ce[10][4] = 0.1;
	ce[11][4] = 0.3;
	ce[12][4] = 0.2;

	double dnxm1 = 1.0/((double)nx-1.0);
	double dnym1 = 1.0/((double)ny-1.0);
	double dnzm1 = 1.0/((double)nz-1.0);

	double tx1 = 1.0 / (dnxm1 * dnxm1);
	double tx2 = 1.0 / (2.0 * dnxm1);
	double tx3 = 1.0 / dnxm1;

	double ty1 = 1.0 / (dnym1 * dnym1);
	double ty2 = 1.0 / (2.0 * dnym1);
	double ty3 = 1.0 / dnym1;
 
	double tz1 = 1.0 / (dnzm1 * dnzm1);
	double tz2 = 1.0 / (2.0 * dnzm1);
	double tz3 = 1.0 / dnzm1;

	double dttx1 = dt*tx1;
	double dttx2 = dt*tx2;
	double dtty1 = dt*ty1;
	double dtty2 = dt*ty2;
	double dttz1 = dt*tz1;
	double dttz2 = dt*tz2;

	double c2dttx1 = 2.0*dttx1;
	double c2dtty1 = 2.0*dtty1;
	double c2dttz1 = 2.0*dttz1;

	double dtdssp = dt*dssp;

	double comz1  = dtdssp;
	double comz4  = 4.0*dtdssp;
	double comz5  = 5.0*dtdssp;
	double comz6  = 6.0*dtdssp;

	double c3c4tx3 = c3c4*tx3;
	double c3c4ty3 = c3c4*ty3;
	double c3c4tz3 = c3c4*tz3;

	double dx1tx1 = dx1*tx1;
	double dx2tx1 = dx2*tx1;
	double dx3tx1 = dx3*tx1;
	double dx4tx1 = dx4*tx1;
	double dx5tx1 = dx5*tx1;

	double dy1ty1 = dy1*ty1;
	double dy2ty1 = dy2*ty1;
	double dy3ty1 = dy3*ty1;
	double dy4ty1 = dy4*ty1;
	double dy5ty1 = dy5*ty1;

	double dz1tz1 = dz1*tz1;
	double dz2tz1 = dz2*tz1;
	double dz3tz1 = dz3*tz1;
	double dz4tz1 = dz4*tz1;
	double dz5tz1 = dz5*tz1;

	double xxcon1 = c3c4tx3*con43*tx3;
	double xxcon2 = c3c4tx3*tx3;
	double xxcon3 = c3c4tx3*conz1*tx3;
	double xxcon4 = c3c4tx3*con16*tx3;
	double xxcon5 = c3c4tx3*c1c5*tx3;

	double yycon1 = c3c4ty3*con43*ty3;
	double yycon2 = c3c4ty3*ty3;
	double yycon3 = c3c4ty3*conz1*ty3;
	double yycon4 = c3c4ty3*con16*ty3;
	double yycon5 = c3c4ty3*c1c5*ty3;

	double zzcon1 = c3c4tz3*con43*tz3;
	double zzcon2 = c3c4tz3*tz3;
	double zzcon3 = c3c4tz3*conz1*tz3;
	double zzcon4 = c3c4tz3*con16*tz3;
	double zzcon5 = c3c4tz3*c1c5*tz3;

	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ce), &ce, 13*5*sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dnxm1), &dnxm1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dnym1), &dnym1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dnzm1), &dnzm1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tx1), &tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tx2), &tx2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tx3), &tx3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ty1), &ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ty2), &ty2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ty3), &ty3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tz1), &tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tz2), &tz2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tz3), &tz3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dt), &dt, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dttx1), &dttx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dttx2), &dttx2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dtty1), &dtty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dtty2), &dtty2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dttz1), &dttz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dttz2), &dttz2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c2dttx1), &c2dttx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c2dtty1), &c2dtty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c2dttz1), &c2dttz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dtdssp), &dtdssp, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::comz1), &comz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::comz4), &comz4, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::comz5), &comz5, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::comz6), &comz6, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c3c4tx3), &c3c4tx3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c3c4ty3), &c3c4ty3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c3c4tz3), &c3c4tz3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx1tx1), &dx1tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx2tx1), &dx2tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx3tx1), &dx3tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx4tx1), &dx4tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx5tx1), &dx5tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy1ty1), &dy1ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy2ty1), &dy2ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy3ty1), &dy3ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy4ty1), &dy4ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy5ty1), &dy5ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz1tz1), &dz1tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz2tz1), &dz2tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz3tz1), &dz3tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz4tz1), &dz4tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz5tz1), &dz5tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon1), &xxcon1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon2), &xxcon2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon3), &xxcon3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon4), &xxcon4, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon5), &xxcon5, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon1), &yycon1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon2), &yycon2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon3), &yycon3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon4), &yycon4, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon5), &yycon5, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon1), &zzcon1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon2), &zzcon2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon3), &zzcon3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon4), &zzcon4, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon5), &zzcon5, sizeof(double)));
}

void BT::allocate_device_memory() {
	int gridsize = nx*ny*nz;
	int facesize = max(max(nx*ny, nx*nz), ny*nz);

	HANDLE_ERROR(hipMalloc((void **)&u, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&forcing, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rhs, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&lhs, 5*5*3*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rho_i, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&us, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&vs, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&ws, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&qs, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&square, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rmsbuf, 5*facesize*sizeof(double)));
}

void BT::free_device_memory() {
	HANDLE_ERROR(hipFree(u));
	HANDLE_ERROR(hipFree(forcing));
	HANDLE_ERROR(hipFree(rhs));
	HANDLE_ERROR(hipFree(lhs));
	HANDLE_ERROR(hipFree(rho_i));
	HANDLE_ERROR(hipFree(us));
	HANDLE_ERROR(hipFree(vs));
	HANDLE_ERROR(hipFree(ws));
	HANDLE_ERROR(hipFree(qs));
	HANDLE_ERROR(hipFree(square));
	HANDLE_ERROR(hipFree(rmsbuf));
}

void BT::get_cuda_info() {
	int count;
	hipDeviceProp_t prop;

	HANDLE_ERROR(hipGetDeviceCount(&count));
	if (count == 0) {
		printf ("No CUDA devices found.\n");
		exit(EXIT_FAILURE);
	}

	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	strncpy (CUDAname, prop.name, 256);
	CUDAmp = prop.multiProcessorCount;
	CUDAclock = prop.clockRate;
	CUDAmem = prop.totalGlobalMem;
	CUDAmemclock = prop.memoryClockRate;
	CUDAl2cache = prop.l2CacheSize;
}
