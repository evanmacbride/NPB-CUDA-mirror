#include "hip/hip_runtime.h"
#include <stdio.h>
#include "main.h"

namespace gpu_mod {
// constants for LU method
__constant__ double dxi, deta, dzeta;
__constant__ double tx1, tx2, tx3, ty1, ty2, ty3, tz1, tz2, tz3;
__constant__ double ce[13*5];
__constant__ double dt, omega;
}

// error handling
static void inline HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define START_TIMER(timer) if (timeron) { HANDLE_ERROR(hipDeviceSynchronize()); timers->timer_start(timer); }
#define STOP_TIMER(timer) if (timeron) { HANDLE_ERROR(hipDeviceSynchronize()); timers->timer_stop(timer); }

__global__ static void jacld_blts_kernel (const int plane, const int klower, const int jlower, const double *u, const double *rho_i, const double *qs, double *v, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double tmp1, tmp2, tmp3, tmat[5*5], tv[5];
	double r43, c1345, c34;

	k = klower+blockIdx.x+1;
	j = jlower+threadIdx.x+1;
	i = plane-k-j+3;
	if (j > ny-2 || i > nx-2 || i < 1) return;

	r43 = 4.0/3.0;
	c1345 = c1 * c3 * c4 * c5;
	c34 = c3 * c4;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	//   form the first block sub-diagonal
	//---------------------------------------------------------------------
	tmp1 = rho_i(i,j,k-1);
	tmp2 = tmp1*tmp1;
	tmp3 = tmp1*tmp2;

	tmat[0+5*0] = -dt*tz1*dz1;
	tmat[0+5*1] = 0.0;
	tmat[0+5*2] = 0.0;
	tmat[0+5*3] = -dt*tz2;
	tmat[0+5*4] = 0.0;

	tmat[1+5*0] = -dt*tz2*(-(u(1,i,j,k-1)*u(3,i,j,k-1))*tmp2) - dt*tz1*(-c34*tmp2*u(1,i,j,k-1));
	tmat[1+5*1] = -dt*tz2*(u(3,i,j,k-1)*tmp1) - dt*tz1*c34*tmp1 - dt*tz1*dz2;
	tmat[1+5*2] = 0.0;
	tmat[1+5*3] = -dt*tz2*(u(1,i,j,k-1)*tmp1);
	tmat[1+5*4] = 0.0;

	tmat[2+5*0] = -dt*tz2*(-(u(2,i,j,k-1)*u(3,i,j,k-1))*tmp2) - dt*tz1*(-c34*tmp2*u(2,i,j,k-1));
	tmat[2+5*1] = 0.0;
	tmat[2+5*2] = -dt*tz2*(u(3,i,j,k-1)*tmp1) - dt*tz1*(c34*tmp1) - dt*tz1*dz3;
	tmat[2+5*3] = -dt*tz2*(u(2,i,j,k-1)*tmp1);
	tmat[2+5*4] = 0.0;

	tmat[3+5*0] = -dt*tz2*(-(u(3,i,j,k-1)*tmp1)*(u(3,i,j,k-1)*tmp1) + c2*qs(i,j,k-1)*tmp1) - dt*tz1*(-r43*c34*tmp2*u(3,i,j,k-1));
	tmat[3+5*1] = -dt*tz2*(-c2*(u(1,i,j,k-1)*tmp1));
	tmat[3+5*2] = -dt*tz2*(-c2*(u(2,i,j,k-1)*tmp1));
	tmat[3+5*3] = -dt*tz2*(2.0-c2)*(u(3,i,j,k-1)*tmp1) - dt*tz1*(r43*c34*tmp1) - dt*tz1*dz4;
	tmat[3+5*4] = -dt*tz2*c2;

	tmat[4+5*0] = -dt*tz2*((c2*2.0*qs(i,j,k-1)-c1*u(4,i,j,k-1))*u(3,i,j,k-1)*tmp2) - dt*tz1*(-(c34-c1345)*tmp3*(u(1,i,j,k-1)*u(1,i,j,k-1))-(c34-c1345)*tmp3*(u(2,i,j,k-1)*u(2,i,j,k-1))-(r43*c34-c1345)*tmp3*(u(3,i,j,k-1)*u(3,i,j,k-1))-c1345*tmp2*u(4,i,j,k-1));
	tmat[4+5*1] = -dt*tz2*(-c2*(u(1,i,j,k-1)*u(3,i,j,k-1))*tmp2) - dt*tz1*(c34-c1345)*tmp2*u(1,i,j,k-1);
	tmat[4+5*2] = -dt*tz2*(-c2*(u(2,i,j,k-1)*u(3,i,j,k-1))*tmp2) - dt*tz1*(c34-c1345)*tmp2*u(2,i,j,k-1);
	tmat[4+5*3] = -dt*tz2*(c1*(u(4,i,j,k-1)*tmp1)-c2*(qs(i,j,k-1)*tmp1+u(3,i,j,k-1)*u(3,i,j,k-1)*tmp2)) - dt*tz1*(r43*c34-c1345)*tmp2*u(3,i,j,k-1);
	tmat[4+5*4] = -dt*tz2*(c1*(u(3,i,j,k-1)*tmp1)) - dt*tz1*c1345*tmp1 - dt*tz1*dz5;

	for (m = 0; m < 5; m++) tv[m] = v(m,i,j,k) - omega*(tmat[m+5*0]*v(0,i,j,k-1) + tmat[m+5*1]*v(1,i,j,k-1) + tmat[m+5*2]*v(2,i,j,k-1) + tmat[m+5*3]*v(3,i,j,k-1) + tmat[m+5*4]*v(4,i,j,k-1));

	//---------------------------------------------------------------------
	//   form the second block sub-diagonal
	//---------------------------------------------------------------------
	tmp1 = rho_i(i,j-1,k);
	tmp2 = tmp1*tmp1;
	tmp3 = tmp1*tmp2;

	tmat[0+5*0] = -dt*ty1*dy1;
	tmat[0+5*1] = 0.0;
	tmat[0+5*2] = -dt*ty2;
	tmat[0+5*3] = 0.0;
	tmat[0+5*4] = 0.0;

	tmat[1+5*0] = -dt*ty2*(-(u(1,i,j-1,k)*u(2,i,j-1,k))*tmp2) - dt*ty1*(-c34*tmp2*u(1,i,j-1,k));
	tmat[1+5*1] = -dt*ty2*(u(2,i,j-1,k)*tmp1) - dt*ty1*(c34*tmp1) - dt*ty1*dy2;
	tmat[1+5*2] = -dt*ty2*(u(1,i,j-1,k)*tmp1);
	tmat[1+5*3] = 0.0;
	tmat[1+5*4] = 0.0;

	tmat[2+5*0] = -dt*ty2*(-(u(2,i,j-1,k)*tmp1)*(u(2,i,j-1,k)*tmp1) + c2*(qs(i,j-1,k)*tmp1)) - dt*ty1*(-r43*c34*tmp2*u(2,i,j-1,k));
	tmat[2+5*1] = -dt*ty2*(-c2*(u(1,i,j-1,k)*tmp1));
	tmat[2+5*2] = -dt*ty2*((2.0-c2)*(u(2,i,j-1,k)*tmp1)) - dt*ty1*(r43*c34*tmp1) - dt*ty1*dy3;
	tmat[2+5*3] = -dt*ty2*(-c2*(u(3,i,j-1,k)*tmp1));
	tmat[2+5*4] = -dt*ty2*c2;

	tmat[3+5*0] = -dt*ty2*(-(u(2,i,j-1,k)*u(3,i,j-1,k))*tmp2) - dt*ty1*(-c34*tmp2*u(3,i,j-1,k));
	tmat[3+5*1] = 0.0;
	tmat[3+5*2] = -dt*ty2*(u(3,i,j-1,k)*tmp1);
	tmat[3+5*3] = -dt*ty2*(u(2,i,j-1,k)*tmp1) - dt*ty1*(c34*tmp1) - dt*ty1*dy4;
	tmat[3+5*4] = 0.0;

	tmat[4+5*0] = -dt*ty2*((c2*2.0*qs(i,j-1,k)-c1*u(4,i,j-1,k))*(u(2,i,j-1,k)*tmp2)) - dt*ty1*(-(c34-c1345)*tmp3*(u(1,i,j-1,k)*u(1,i,j-1,k))-(r43*c34-c1345)*tmp3*(u(2,i,j-1,k)*u(2,i,j-1,k))-(c34-c1345)*tmp3*(u(3,i,j-1,k)*u(3,i,j-1,k))-c1345*tmp2*u(4,i,j-1,k));
	tmat[4+5*1] = -dt*ty2*(-c2*(u(1,i,j-1,k)*u(2,i,j-1,k))*tmp2) - dt*ty1*(c34-c1345)*tmp2*u(1,i,j-1,k);
	tmat[4+5*2] = -dt*ty2*(c1*(u(4,i,j-1,k)*tmp1)-c2*(qs(i,j-1,k)*tmp1+u(2,i,j-1,k)*u(2,i,j-1,k)*tmp2)) - dt*ty1*(r43*c34-c1345)*tmp2*u(2,i,j-1,k);
	tmat[4+5*3] = -dt*ty2*(-c2*(u(2,i,j-1,k)*u(3,i,j-1,k))*tmp2) - dt*ty1*(c34-c1345)*tmp2*u(3,i,j-1,k);
	tmat[4+5*4] = -dt*ty2*(c1*(u(2,i,j-1,k)*tmp1)) - dt*ty1*c1345*tmp1 - dt*ty1*dy5;

	for (m = 0; m < 5; m++) tv[m] = tv[m] - omega*(tmat[m+5*0]*v(0,i,j-1,k) + tmat[m+5*1]*v(1,i,j-1,k) + tmat[m+5*2]*v(2,i,j-1,k) + tmat[m+5*3]*v(3,i,j-1,k) + tmat[m+5*4]*v(4,i,j-1,k));

	//---------------------------------------------------------------------
	//   form the third block sub-diagonal
	//---------------------------------------------------------------------
	tmp1 = rho_i(i-1,j,k);
	tmp2 = tmp1*tmp1;
	tmp3 = tmp1*tmp2;

	tmat[0+5*0] = -dt*tx1*dx1;
	tmat[0+5*1] = -dt*tx2;
	tmat[0+5*2] = 0.0;
	tmat[0+5*3] = 0.0;
	tmat[0+5*4] = 0.0;

	tmat[1+5*0] = -dt*tx2*(-(u(1,i-1,j,k)*tmp1)*(u(1,i-1,j,k)*tmp1)+c2*qs(i-1,j,k)*tmp1) -dt*tx1*(-r43*c34*tmp2*u(1,i-1,j,k));
	tmat[1+5*1] = -dt*tx2*((2.0-c2)*(u(1,i-1,j,k)*tmp1)) - dt*tx1*(r43*c34*tmp1) - dt*tx1*dx2;
	tmat[1+5*2] = -dt*tx2*(-c2*(u(2,i-1,j,k)*tmp1));
	tmat[1+5*3] = -dt*tx2*(-c2*(u(3,i-1,j,k)*tmp1));
	tmat[1+5*4] = -dt*tx2*c2;

	tmat[2+5*0] = -dt*tx2*(-(u(1,i-1,j,k)*u(2,i-1,j,k))*tmp2) - dt*tx1*(-c34*tmp2*u(2,i-1,j,k));
	tmat[2+5*1] = -dt*tx2*(u(2,i-1,j,k)*tmp1);
	tmat[2+5*2] = -dt*tx2*(u(1,i-1,j,k)*tmp1) - dt*tx1*(c34*tmp1) - dt*tx1*dx3;
	tmat[2+5*3] = 0.0;
	tmat[2+5*4] = 0.0;

	tmat[3+5*0] = -dt*tx2*(-(u(1,i-1,j,k)*u(3,i-1,j,k))*tmp2) - dt*tx1*(-c34*tmp2*u(3,i-1,j,k));
	tmat[3+5*1] = -dt*tx2*(u(3,i-1,j,k)*tmp1);
	tmat[3+5*2] = 0.0;
	tmat[3+5*3] = -dt*tx2*(u(1,i-1,j,k)*tmp1) - dt*tx1*(c34*tmp1) - dt*tx1*dx4;
	tmat[3+5*4] = 0.0;

	tmat[4+5*0] = -dt*tx2*((c2*2.0*qs(i-1,j,k)-c1*u(4,i-1,j,k))*u(1,i-1,j,k)*tmp2) - dt*tx1*(-(r43*c34-c1345)*tmp3*(u(1,i-1,j,k)*u(1,i-1,j,k))-(c34-c1345)*tmp3*(u(2,i-1,j,k)*u(2,i-1,j,k))-(c34-c1345)*tmp3*(u(3,i-1,j,k)*u(3,i-1,j,k))-c1345*tmp2*u(4,i-1,j,k));
	tmat[4+5*1] = -dt*tx2*(c1*(u(4,i-1,j,k)*tmp1)-c2*(u(1,i-1,j,k)*u(1,i-1,j,k)*tmp2+qs(i-1,j,k)*tmp1)) - dt*tx1*(r43*c34-c1345)*tmp2*u(1,i-1,j,k);
	tmat[4+5*2] = -dt*tx2*(-c2*(u(2,i-1,j,k)*u(1,i-1,j,k))*tmp2) - dt*tx1*(c34-c1345)*tmp2*u(2,i-1,j,k);
	tmat[4+5*3] = -dt*tx2*(-c2*(u(3,i-1,j,k)*u(1,i-1,j,k))*tmp2) - dt*tx1*(c34-c1345)*tmp2*u(3,i-1,j,k);
	tmat[4+5*4] = -dt*tx2*(c1*(u(1,i-1,j,k)*tmp1)) - dt*tx1*c1345*tmp1 - dt*tx1*dx5;

	for (m = 0; m < 5; m++) tv[m] = tv[m] - omega*(tmat[m+0*5]*v(0,i-1,j,k) + tmat[m+5*1]*v(1,i-1,j,k) + tmat[m+5*2]*v(2,i-1,j,k) + tmat[m+5*3]*v(3,i-1,j,k) + tmat[m+5*4]*v(4,i-1,j,k));

	//---------------------------------------------------------------------
	//   form the block diagonal
	//---------------------------------------------------------------------
	tmp1 = rho_i(i,j,k);
	tmp2 = tmp1*tmp1;
	tmp3 = tmp1*tmp2;

	tmat[0+5*0] = 1.0 + dt*2.0*(tx1*dx1+ty1*dy1+tz1*dz1);
	tmat[0+5*1] = 0.0;
	tmat[0+5*2] = 0.0;
	tmat[0+5*3] = 0.0;
	tmat[0+5*4] = 0.0;

	tmat[1+5*0] = -dt*2.0*(tx1*r43+ty1+tz1)*c34*tmp2*u(1,i,j,k);
	tmat[1+5*1] = 1.0 + dt*2.0*c34*tmp1*(tx1*r43+ty1+tz1) + dt*2.0*(tx1*dx2+ty1*dy2+tz1*dz2);
	tmat[1+5*2] = 0.0;
	tmat[1+5*3] = 0.0;
	tmat[1+5*4] = 0.0;

	tmat[2+5*0] = -dt*2.0*(tx1+ty1*r43+tz1)*c34*tmp2*u(2,i,j,k);
	tmat[2+5*1] = 0.0;
	tmat[2+5*2] = 1.0 + dt*2.0*c34*tmp1*(tx1+ty1*r43+tz1) + dt*2.0*(tx1*dx3+ty1*dy3+tz1*dz3);
	tmat[2+5*3] = 0.0;
	tmat[2+5*4] = 0.0;

	tmat[3+5*0] = -dt*2.0*(tx1+ty1+tz1*r43)*c34*tmp2*u(3,i,j,k);
	tmat[3+5*1] = 0.0;
	tmat[3+5*2] = 0.0;
	tmat[3+5*3] = 1.0 + dt*2.0*c34*tmp1*(tx1+ty1+tz1*r43) + dt*2.0*(tx1*dx4+ty1*dy4+tz1*dz4);
	tmat[3+5*4] = 0.0;

	tmat[4+5*0] = -dt*2.0*(((tx1*(r43*c34-c1345)+ty1*(c34-c1345)+tz1*(c34-c1345))*(u(1,i,j,k)*u(1,i,j,k))+(tx1*(c34-c1345)+ty1*(r43*c34-c1345)+tz1*(c34-c1345))*(u(2,i,j,k)*u(2,i,j,k))+
			(tx1*(c34-c1345)+ty1*(c34-c1345)+tz1*(r43*c34-c1345))*(u(3,i,j,k)*u(3,i,j,k)))*tmp3+(tx1+ty1+tz1)*c1345*tmp2*u(4,i,j,k));
	tmat[4+5*1] = dt*2.0*tmp2*u(1,i,j,k)*(tx1*(r43*c34-c1345)+ty1*(c34-c1345)+tz1*(c34-c1345));
	tmat[4+5*2] = dt*2.0*tmp2*u(2,i,j,k)*(tx1*(c34-c1345)+ty1*(r43*c34-c1345)+tz1*(c34-c1345));
	tmat[4+5*3] = dt*2.0*tmp2*u(3,i,j,k)*(tx1*(c34-c1345)+ty1*(c34-c1345)+tz1*(r43*c34-c1345));
	tmat[4+5*4] = 1.0 + dt*2.0*(tx1+ty1+tz1)*c1345*tmp1 + dt*2.0*(tx1*dx5+ty1*dy5+tz1*dz5);

	//---------------------------------------------------------------------
	//   diagonal block inversion;  forward elimination
	//---------------------------------------------------------------------
	tmp1 = 1.0/tmat[0+0*5];
	tmp2 = tmp1*tmat[1+0*5];
	tmat[1+1*5] -= tmp2*tmat[0+1*5];
	tmat[1+2*5] -= tmp2*tmat[0+2*5];
	tmat[1+3*5] -= tmp2*tmat[0+3*5];
	tmat[1+4*5] -= tmp2*tmat[0+4*5];
	tv[1] -= tmp2*tv[0];

	tmp2 = tmp1*tmat[2+0*5];
	tmat[2+1*5] -= tmp2*tmat[0+1*5];
	tmat[2+2*5] -= tmp2*tmat[0+2*5];
	tmat[2+3*5] -= tmp2*tmat[0+3*5];
	tmat[2+4*5] -= tmp2*tmat[0+4*5];
	tv[2] -= tmp2*tv[0];

	tmp2 = tmp1*tmat[3+0*5];
	tmat[3+1*5] -= tmp2*tmat[0+1*5];
	tmat[3+2*5] -= tmp2*tmat[0+2*5];
	tmat[3+3*5] -= tmp2*tmat[0+3*5];
	tmat[3+4*5] -= tmp2*tmat[0+4*5];
	tv[3] -= tmp2*tv[0];

	tmp2 = tmp1*tmat[4+0*5];
	tmat[4+1*5] -= tmp2*tmat[0+1*5];
	tmat[4+2*5] -= tmp2*tmat[0+2*5];
	tmat[4+3*5] -= tmp2*tmat[0+3*5];
	tmat[4+4*5] -= tmp2*tmat[0+4*5];
	tv[4] -= tmp2*tv[0];

	tmp1 = 1.0/tmat[1+1*5];
	tmp2 = tmp1*tmat[2+1*5];
	tmat[2+2*5] -= tmp2*tmat[1+2*5];
	tmat[2+3*5] -= tmp2*tmat[1+3*5];
	tmat[2+4*5] -= tmp2*tmat[1+4*5];
	tv[2] -= tmp2*tv[1];

	tmp2 = tmp1*tmat[3+1*5];
	tmat[3+2*5] -= tmp2*tmat[1+2*5];
	tmat[3+3*5] -= tmp2*tmat[1+3*5];
	tmat[3+4*5] -= tmp2*tmat[1+4*5];
	tv[3] -= tmp2*tv[1];

	tmp2 = tmp1*tmat[4+1*5];
	tmat[4+2*5] -= tmp2*tmat[1+2*5];
	tmat[4+3*5] -= tmp2*tmat[1+3*5];
	tmat[4+4*5] -= tmp2*tmat[1+4*5];
	tv[4] -= tmp2*tv[1];

	tmp1 = 1.0/tmat[2+2*5];
	tmp2 = tmp1*tmat[3+2*5];
	tmat[3+3*5] -= tmp2*tmat[2+3*5];
	tmat[3+4*5] -= tmp2*tmat[2+4*5];
	tv[3] -= tmp2*tv[2];

	tmp2 = tmp1*tmat[4+2*5];
	tmat[4+3*5] -= tmp2*tmat[2+3*5];
	tmat[4+4*5] -= tmp2*tmat[2+4*5];
	tv[4] -= tmp2*tv[2];

	tmp1 = 1.0/tmat[3+3*5];
	tmp2 = tmp1*tmat[4+3*5];
	tmat[4+4*5] -= tmp2*tmat[3+4*5];
	tv[4] -= tmp2*tv[3];

	//---------------------------------------------------------------------
	//   back substitution
	//---------------------------------------------------------------------
	v(4,i,j,k) = tv[4]/tmat[4+4*5];

	tv[3] = tv[3] - tmat[3+4*5]*v(4,i,j,k);
	v(3,i,j,k) = tv[3]/tmat[3+3*5];

	tv[2] = tv[2] - tmat[2+3*5]*v(3,i,j,k) - tmat[2+4*5]*v(4,i,j,k);
	v(2,i,j,k) = tv[2]/tmat[2+2*5];

	tv[1] = tv[1] - tmat[1+2*5]*v(2,i,j,k) - tmat[1+3*5]*v(3,i,j,k) - tmat[1+4*5]*v(4,i,j,k);
	v(1,i,j,k) = tv[1]/tmat[1+1*5];

	tv[0] = tv[0] - tmat[0+1*5]*v(1,i,j,k) - tmat[0+2*5]*v(2,i,j,k) - tmat[0+3*5]*v(3,i,j,k) - tmat[0+4*5]*v(4,i,j,k);
	v(0,i,j,k) = tv[0]/tmat[0+0*5];
}

__global__ static void jacu_buts_kernel (const int plane, const int klower, const int jlower, const double *u, const double *rho_i, const double *qs, double *v, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double tmp, tmp1, tmp2, tmp3, tmat[5*5], tv[5];
	double r43, c1345, c34;

	k = klower+blockIdx.x+1;
	j = jlower+threadIdx.x+1;
	i = plane-j-k+3;
	if (i < 1 || i > nx-2 || j > ny-2) return;

	using namespace gpu_mod;

	r43 = 4.0/3.0;
	c1345 = c1*c3*c4*c5;
	c34 = c3*c4;

	//---------------------------------------------------------------------
	//   form the first block sub-diagonal
	//---------------------------------------------------------------------
	tmp1 = rho_i(i+1,j,k);
	tmp2 = tmp1*tmp1;
	tmp3 = tmp1*tmp2;

	tmat[0+5*0] = -dt*tx1*dx1;
	tmat[0+5*1] = dt*tx2;
	tmat[0+5*2] = 0.0;
	tmat[0+5*3] = 0.0;
	tmat[0+5*4] = 0.0;

	tmat[1+5*0] = dt*tx2*(-(u(1,i+1,j,k)*tmp1)*(u(1,i+1,j,k)*tmp1)+c2*qs(i+1,j,k)*tmp1) - dt*tx1*(-r43*c34*tmp2*u(1,i+1,j,k));
	tmat[1+5*1] = dt*tx2*((2.0-c2)*(u(1,i+1,j,k)*tmp1)) - dt*tx1*(r43*c34*tmp1) - dt*tx1*dx2;
	tmat[1+5*2] = dt*tx2*(-c2*(u(2,i+1,j,k)*tmp1));
	tmat[1+5*3] = dt*tx2*(-c2*(u(3,i+1,j,k)*tmp1));
	tmat[1+5*4] = dt*tx2*c2;

	tmat[2+5*0] = dt*tx2*(-(u(1,i+1,j,k)*u(2,i+1,j,k))*tmp2) - dt*tx1*(-c34*tmp2*u(2,i+1,j,k));
	tmat[2+5*1] = dt*tx2*(u(2,i+1,j,k)*tmp1);
	tmat[2+5*2] = dt*tx2*(u(1,i+1,j,k)*tmp1) - dt*tx1*(c34*tmp1) - dt*tx1*dx3;
	tmat[2+5*3] = 0.0;
	tmat[2+5*4] = 0.0;

	tmat[3+5*0] = dt*tx2*(-(u(1,i+1,j,k)*u(3,i+1,j,k))*tmp2) - dt*tx1*(-c34*tmp2*u(3,i+1,j,k));
	tmat[3+5*1] = dt*tx2*(u(3,i+1,j,k)*tmp1);
	tmat[3+5*2] = 0.0;
	tmat[3+5*3] = dt*tx2*(u(1,i+1,j,k)*tmp1) - dt*tx1*(c34*tmp1) - dt*tx1*dx4;
	tmat[3+5*4] = 0.0;

	tmat[4+5*0] = dt*tx2*((c2*2.0*qs(i+1,j,k)-c1*u(4,i+1,j,k))*(u(1,i+1,j,k)*tmp2)) - dt*tx1*(-(r43*c34-c1345)*tmp3*(u(1,i+1,j,k)*u(1,i+1,j,k))-(c34-c1345)*tmp3*(u(2,i+1,j,k)*u(2,i+1,j,k))-(c34-c1345)*tmp3*(u(3,i+1,j,k)*u(3,i+1,j,k))-c1345*tmp2*u(4,i+1,j,k));
	tmat[4+5*1] = dt*tx2*(c1*(u(4,i+1,j,k)*tmp1)-c2*(u(1,i+1,j,k)*u(1,i+1,j,k)*tmp2+qs(i+1,j,k)*tmp1)) - dt*tx1*(r43*c34-c1345)*tmp2*u(1,i+1,j,k);
	tmat[4+5*2] = dt*tx2*(-c2*(u(2,i+1,j,k)*u(1,i+1,j,k))*tmp2) - dt*tx1*(c34-c1345)*tmp2*u(2,i+1,j,k);
	tmat[4+5*3] = dt*tx2*(-c2*(u(3,i+1,j,k)*u(1,i+1,j,k))*tmp2) - dt*tx1*(c34-c1345)*tmp2*u(3,i+1,j,k);
	tmat[4+5*4] = dt*tx2*(c1*(u(1,i+1,j,k)*tmp1)) - dt*tx1*c1345*tmp1 - dt*tx1*dx5;

	for (m = 0; m < 5; m++) tv[m] = omega*(tmat[m+5*0]*v(0,i+1,j,k) + tmat[m+5*1]*v(1,i+1,j,k) + tmat[m+5*2]*v(2,i+1,j,k) + tmat[m+5*3]*v(3,i+1,j,k) + tmat[m+5*4]*v(4,i+1,j,k));

	//---------------------------------------------------------------------
	//   form the second block sub-diagonal
	//---------------------------------------------------------------------
	tmp1 = rho_i(i,j+1,k);
	tmp2 = tmp1*tmp1;
	tmp3 = tmp1*tmp2;

	tmat[0+5*0] = -dt*ty1*dy1;
	tmat[0+5*1] = 0.0;
	tmat[0+5*2] = dt*ty2;
	tmat[0+5*3] = 0.0;
	tmat[0+5*4] = 0.0;

	tmat[1+5*0] = dt*ty2*(-(u(1,i,j+1,k)*u(2,i,j+1,k))*tmp2) - dt*ty1*(-c34*tmp2*u(1,i,j+1,k));
	tmat[1+5*1] = dt*ty2*(u(2,i,j+1,k)*tmp1) - dt*ty1*(c34*tmp1) - dt*ty1*dy2;
	tmat[1+5*2] = dt*ty2*(u(1,i,j+1,k)*tmp1);
	tmat[1+5*3] = 0.0;
	tmat[1+5*4] = 0.0;

	tmat[2+5*0] = dt*ty2*(-(u(2,i,j+1,k)*tmp1)*(u(2,i,j+1,k)*tmp1)+c2*(qs(i,j+1,k)*tmp1)) - dt*ty1*(-r43*c34*tmp2*u(2,i,j+1,k));
	tmat[2+5*1] = dt*ty2*(-c2*(u(1,i,j+1,k)*tmp1));
	tmat[2+5*2] = dt*ty2*((2.0-c2)*(u(2,i,j+1,k)*tmp1)) - dt*ty1*(r43*c34*tmp1) - dt*ty1*dy3;
	tmat[2+5*3] = dt*ty2*(-c2*(u(3,i,j+1,k)*tmp1));
	tmat[2+5*4] = dt*ty2*c2;

	tmat[3+5*0] = dt*ty2*(-(u(2,i,j+1,k)*u(3,i,j+1,k))*tmp2) - dt*ty1*(-c34*tmp2*u(3,i,j+1,k));
	tmat[3+5*1] = 0.0;
	tmat[3+5*2] = dt*ty2*(u(3,i,j+1,k)*tmp1);
	tmat[3+5*3] = dt*ty2*(u(2,i,j+1,k)*tmp1) - dt*ty1*(c34*tmp1) - dt*ty1*dy4;
	tmat[3+5*4] = 0.0;

	tmat[4+5*0] = dt*ty2*((c2*2.0*qs(i,j+1,k)-c1*u(4,i,j+1,k))*(u(2,i,j+1,k)*tmp2)) - dt*ty1*(-(c34-c1345)*tmp3*(u(1,i,j+1,k)*u(1,i,j+1,k))-(r43*c34-c1345)*tmp3*(u(2,i,j+1,k)*u(2,i,j+1,k))-(c34-c1345)*tmp3*(u(3,i,j+1,k)*u(3,i,j+1,k))-c1345*tmp2*u(4,i,j+1,k));
	tmat[4+5*1] = dt*ty2*(-c2*(u(1,i,j+1,k)*u(2,i,j+1,k))*tmp2) - dt*ty1*(c34-c1345)*tmp2*u(1,i,j+1,k);
	tmat[4+5*2] = dt*ty2*(c1*(u(4,i,j+1,k)*tmp1)-c2*(qs(i,j+1,k)*tmp1+u(2,i,j+1,k)*u(2,i,j+1,k)*tmp2)) - dt*ty1*(r43*c34-c1345)*tmp2*u(2,i,j+1,k);
	tmat[4+5*3] = dt*ty2*(-c2*(u(2,i,j+1,k)*u(3,i,j+1,k))*tmp2) - dt*ty1*(c34-c1345)*tmp2*u(3,i,j+1,k);
	tmat[4+5*4] = dt*ty2*(c1*(u(2,i,j+1,k)*tmp1)) - dt*ty1*c1345*tmp1 - dt*ty1*dy5;

	for (m = 0; m < 5; m++) tv[m]= tv[m] + omega*(tmat[m+5*0]*v(0,i,j+1,k) + tmat[m+5*1]*v(1,i,j+1,k) + tmat[m+5*2]*v(2,i,j+1,k) + tmat[m+5*3]*v(3,i,j+1,k) + tmat[m+5*4]*v(4,i,j+1,k));

	//---------------------------------------------------------------------
	//   form the third block sub-diagonal
	//---------------------------------------------------------------------
	tmp1 = rho_i(i,j,k+1);
	tmp2 = tmp1*tmp1;
	tmp3 = tmp1*tmp2;

	tmat[0+5*0] = -dt*tz1*dz1;
	tmat[0+5*1] = 0.0;
	tmat[0+5*2] = 0.0;
	tmat[0+5*3] = dt*tz2;
	tmat[0+5*4] = 0.0;

	tmat[1+5*0] = dt*tz2*(-(u(1,i,j,k+1)*u(3,i,j,k+1))*tmp2) - dt*tz1*(-c34*tmp2*u(1,i,j,k+1));
	tmat[1+5*1] = dt*tz2*(u(3,i,j,k+1)*tmp1) - dt*tz1*c34*tmp1 - dt*tz1*dz2;
	tmat[1+5*2] = 0.0;
	tmat[1+5*3] = dt*tz2*(u(1,i,j,k+1)*tmp1);
	tmat[1+5*4] = 0.0;

	tmat[2+5*0] = dt*tz2*(-(u(2,i,j,k+1)*u(3,i,j,k+1))*tmp2) - dt*tz1*(-c34*tmp2*u(2,i,j,k+1));
	tmat[2+5*1] = 0.0;
	tmat[2+5*2] = dt*tz2*(u(3,i,j,k+1)*tmp1) - dt*tz1*(c34*tmp1) - dt*tz1*dz3;
	tmat[2+5*3] = dt*tz2*(u(2,i,j,k+1)*tmp1);
	tmat[2+5*4] = 0.0;

	tmat[3+5*0] = dt*tz2*(-(u(3,i,j,k+1)*tmp1)*(u(3,i,j,k+1)*tmp1)+c2*(qs(i,j,k+1)*tmp1)) - dt*tz1*(-r43*c34*tmp2*u(3,i,j,k+1));
	tmat[3+5*1] = dt*tz2*(-c2*(u(1,i,j,k+1)*tmp1));
	tmat[3+5*2] = dt*tz2*(-c2*(u(2,i,j,k+1)*tmp1));
	tmat[3+5*3] = dt*tz2*(2.0-c2)*(u(3,i,j,k+1)*tmp1) - dt*tz1*(r43*c34*tmp1) - dt*tz1*dz4;
	tmat[3+5*4] = dt*tz2*c2;

	tmat[4+5*0] = dt*tz2*((c2*2.0*qs(i,j,k+1)-c1*u(4,i,j,k+1))*(u(3,i,j,k+1)*tmp2)) - dt*tz1*(-(c34-c1345)*tmp3*(u(1,i,j,k+1)*u(1,i,j,k+1))-(c34-c1345)*tmp3*(u(2,i,j,k+1)*u(2,i,j,k+1))-(r43*c34-c1345)*tmp3*(u(3,i,j,k+1)*u(3,i,j,k+1))-c1345*tmp2*u(4,i,j,k+1));
	tmat[4+5*1] = dt*tz2*(-c2*(u(1,i,j,k+1)*u(3,i,j,k+1))*tmp2) - dt*tz1*(c34-c1345)*tmp2*u(1,i,j,k+1);
	tmat[4+5*2] = dt*tz2*(-c2*(u(2,i,j,k+1)*u(3,i,j,k+1))*tmp2) - dt*tz1*(c34-c1345)*tmp2*u(2,i,j,k+1);
	tmat[4+5*3] = dt*tz2*(c1*(u(4,i,j,k+1)*tmp1)-c2*(qs(i,j,k+1)*tmp1+u(3,i,j,k+1)*u(3,i,j,k+1)*tmp2)) - dt*tz1*(r43*c34-c1345)*tmp2*u(3,i,j,k+1);
	tmat[4+5*4] = dt*tz2*(c1*(u(3,i,j,k+1)*tmp1)) - dt*tz1*c1345*tmp1 - dt*tz1*dz5;

	for (m = 0; m < 5; m++) tv[m] = tv[m] + omega*(tmat[m+5*0]*v(0,i,j,k+1) + tmat[m+5*1]*v(1,i,j,k+1) + tmat[m+5*2]*v(2,i,j,k+1) + tmat[m+5*3]*v(3,i,j,k+1) + tmat[m+5*4]*v(4,i,j,k+1));
		
	//---------------------------------------------------------------------
	//   form the block daigonal
	//---------------------------------------------------------------------
	tmp1 = rho_i(i,j,k);
	tmp2 = tmp1*tmp1;
	tmp3 = tmp1*tmp2;

	tmat[0+5*0] = 1.0 + dt*2.0*(tx1*dx1+ty1*dy1+tz1*dz1);
	tmat[0+5*1] = 0.0;
	tmat[0+5*2] = 0.0;
	tmat[0+5*3] = 0.0;
	tmat[0+5*4] = 0.0;

	tmat[1+5*0] = dt*2.0*(-tx1*r43-ty1-tz1)*(c34*tmp2*u(1,i,j,k));
	tmat[1+5*1] = 1.0 + dt*2.0*c34*tmp1*(tx1*r43+ty1+tz1) + dt*2.0*(tx1*dx2+ty1*dy2+tz1*dz2);
	tmat[1+5*2] = 0.0;
	tmat[1+5*3] = 0.0;
	tmat[1+5*4] = 0.0;

	tmat[2+5*0] = dt*2.0*(-tx1-ty1*r43-tz1)*(c34*tmp2*u(2,i,j,k));
	tmat[2+5*1] = 0.0;
	tmat[2+5*2] = 1.0 + dt*2.0*c34*tmp1*(tx1+ty1*r43+tz1) + dt*2.0*(tx1*dx3+ty1*dy3+tz1*dz3);
	tmat[2+5*3] = 0.0;
	tmat[2+5*4] = 0.0;

	tmat[3+5*0] = dt*2.0*(-tx1-ty1-tz1*r43)*(c34*tmp2*u(3,i,j,k));
	tmat[3+5*1] = 0.0;
	tmat[3+5*2] = 0.0;
	tmat[3+5*3] = 1.0 + dt*2.0*c34*tmp1*(tx1+ty1+tz1*r43) + dt*2.0*(tx1*dx4+ty1*dy4+tz1*dz4);
	tmat[3+5*4] = 0.0;

	tmat[4+5*0] = -dt*2.0*(((tx1*(r43*c34-c1345)+ty1*(c34-c1345)+tz1*(c34-c1345))*(u(1,i,j,k)*u(1,i,j,k))+(tx1*(c34-c1345)+ty1*(r43*c34-c1345)+tz1*(c34-c1345))*(u(2,i,j,k)*u(2,i,j,k))+
			(tx1*(c34-c1345)+ty1*(c34-c1345)+tz1*(r43*c34-c1345))*(u(3,i,j,k)*u(3,i,j,k)))*tmp3 + (tx1+ty1+tz1)*c1345*tmp2*u(4,i,j,k));
	tmat[4+5*1] = dt*2.0*(tx1*(r43*c34-c1345)+ty1*(c34-c1345)+tz1*(c34-c1345))*tmp2*u(1,i,j,k);
	tmat[4+5*2] = dt*2.0*(tx1*(c34-c1345)+ty1*(r43*c34-c1345)+tz1*(c34-c1345))*tmp2*u(2,i,j,k);
	tmat[4+5*3] = dt*2.0*(tx1*(c34-c1345)+ty1*(c34-c1345)+tz1*(r43*c34-c1345))*tmp2*u(3,i,j,k);
	tmat[4+5*4] = 1.0 + dt*2.0*(tx1+ty1+tz1)*c1345*tmp1 + dt*2.0*(tx1*dx5+ty1*dy5+tz1*dz5);

	//---------------------------------------------------------------------
	//   diagonal block inversion
	//---------------------------------------------------------------------
	tmp1 = 1.0/tmat[0+0*5];
	tmp = tmp1*tmat[1+0*5];
	tmat[1+1*5] -= tmp*tmat[0+1*5];
	tmat[1+2*5] -= tmp*tmat[0+2*5];
	tmat[1+3*5] -= tmp*tmat[0+3*5];
	tmat[1+4*5] -= tmp*tmat[0+4*5];
	tv[1] -= tmp*tv[0];

	tmp = tmp1*tmat[2+0*5];
	tmat[2+1*5] -= tmp*tmat[0+1*5];
	tmat[2+2*5] -= tmp*tmat[0+2*5];
	tmat[2+3*5] -= tmp*tmat[0+3*5];
	tmat[2+4*5] -= tmp*tmat[0+4*5];
	tv[2] -= tmp*tv[0];

	tmp = tmp1*tmat[3+0*5];
	tmat[3+1*5] -= tmp*tmat[0+1*5];
	tmat[3+2*5] -= tmp*tmat[0+2*5];
	tmat[3+3*5] -= tmp*tmat[0+3*5];
	tmat[3+4*5] -= tmp*tmat[0+4*5];
	tv[3] -= tmp*tv[0];

	tmp = tmp1*tmat[4+0*5];
	tmat[4+1*5] -= tmp*tmat[0+1*5];
	tmat[4+2*5] -= tmp*tmat[0+2*5];
	tmat[4+3*5] -= tmp*tmat[0+3*5];
	tmat[4+4*5] -= tmp*tmat[0+4*5];
	tv[4] -= tmp*tv[0];

	tmp1 = 1.0/tmat[1+1*5];
	tmp = tmp1*tmat[2+1*5];
	tmat[2+2*5] -= tmp*tmat[1+2*5];
	tmat[2+3*5] -= tmp*tmat[1+3*5];
	tmat[2+4*5] -= tmp*tmat[1+4*5];
	tv[2] -= tmp*tv[1];

	tmp = tmp1*tmat[3+1*5];
	tmat[3+2*5] -= tmp*tmat[1+2*5];
	tmat[3+3*5] -= tmp*tmat[1+3*5];
	tmat[3+4*5] -= tmp*tmat[1+4*5];
	tv[3] -= tmp*tv[1];

	tmp = tmp1*tmat[4+1*5];
	tmat[4+2*5] -= tmp*tmat[1+2*5];
	tmat[4+3*5] -= tmp*tmat[1+3*5];
	tmat[4+4*5] -= tmp*tmat[1+4*5];
	tv[4] -= tmp*tv[1];

	tmp1 = 1.0/tmat[2+2*5];
	tmp = tmp1*tmat[3+2*5];
	tmat[3+3*5] -= tmp*tmat[2+3*5];
	tmat[3+4*5] -= tmp*tmat[2+4*5];
	tv[3] -= tmp*tv[2];

	tmp = tmp1*tmat[4+2*5];
	tmat[4+3*5] -= tmp*tmat[2+3*5];
	tmat[4+4*5] -= tmp*tmat[2+4*5];
	tv[4] -= tmp*tv[2];

	tmp1 = 1.0/tmat[3+3*5];
	tmp = tmp1 * tmat[4+3*5];
	tmat[4+4*5] -= tmp*tmat[3+4*5];
	tv[4] -= tmp*tv[3];

	//---------------------------------------------------------------------
	//   back substitution
	//---------------------------------------------------------------------
	tv[4] = tv[4]/tmat[4+4*5];

	tv[3] = tv[3] - tmat[3+4*5]*tv[4];
	tv[3] = tv[3]/tmat[3+3*5];

	tv[2] = tv[2] - tmat[2+3*5]*tv[3] - tmat[2+4*5]*tv[4];
	tv[2] = tv[2]/tmat[2+2*5];

	tv[1] = tv[1] - tmat[1+2*5]*tv[2] - tmat[1+3*5]*tv[3] - tmat[1+4*5]*tv[4];
	tv[1] = tv[1]/tmat[1+1*5];

	tv[0] = tv[0] - tmat[0+1*5]*tv[1] - tmat[0+2*5]*tv[2] - tmat[0+3*5]*tv[3] - tmat[0+4*5]*tv[4];
	tv[0] = tv[0]/tmat[0+0*5];

	v(0,i,j,k) -= tv[0];
	v(1,i,j,k) -= tv[1];
	v(2,i,j,k) -= tv[2];
	v(3,i,j,k) -= tv[3];
	v(4,i,j,k) -= tv[4];
}

__global__ static void ssor_kernel1 (double *rsd, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	
	i = threadIdx.x+1;
	j = blockIdx.y+1;
	k = blockIdx.x+1;
	m = threadIdx.y;
	using namespace gpu_mod;

	rsd(m,i,j,k) *= dt;
}

__global__ static void ssor_kernel2 (double *u, double *rsd, const double tmp, const int nx, const int ny, const int nz) {
	int i, j, k, m;

	i = threadIdx.x+1;
	j = blockIdx.y+1;
	k = blockIdx.x+1;

	for (m = 0; m < 5; m++) u(m,i,j,k) += tmp*rsd(m,i,j,k);
}

void LU::ssor(int niter) {
	dim3 grid_yz(nz-2,ny-2);
	dim3 grid_x(nx-2,5);
	double tmp = 1.0/(omega*(2.0-omega));

	//---------------------------------------------------------------------
	//   compute the steady-state residuals
	//---------------------------------------------------------------------
	rhs();

	//---------------------------------------------------------------------
	//   compute the L2 norms of newton iteration residuals
	//---------------------------------------------------------------------
	l2norm(rsd, rsdnm);

	for (int i = 0; i < t_last; i++) timers->timer_clear(i);
	HANDLE_ERROR(hipDeviceSynchronize());
	timers->timer_start(0);

	//---------------------------------------------------------------------
	//   the timestep loop
	//---------------------------------------------------------------------
	for (int istep = 1; istep <= niter; istep++) {
		if ((istep % 20 == 0 || istep == itmax || istep == 1) && niter > 1)
			printf(" Time step %4d\n", istep);
		//---------------------------------------------------------------------
		//   perform SSOR iteration
		//---------------------------------------------------------------------
		START_TIMER(t_rhs);
		ssor_kernel1<<<grid_yz, grid_x>>>(rsd, nx, ny, nz);
		STOP_TIMER(t_rhs);

		//---------------------------------------------------------------------
		//   form the lower triangular part of the jacobian matrix
		//   perform the lower triangular solution
		//---------------------------------------------------------------------
		START_TIMER(t_jacld);
		for (int plane = 0; plane <= nx+ny+nz-9; plane++) {
			int klower = max(0, plane-(nx-3)-(ny-3));
			int kupper = min(plane, nz-3);
			int jlowermin = max(0, plane-kupper-(nx-3));
			int juppermax = min(plane, ny-3);

			jacld_blts_kernel<<<kupper-klower+1,juppermax-jlowermin+1>>>(plane, klower, jlowermin, u, rho_i, qs, rsd, nx, ny, nz);
		}
		STOP_TIMER(t_jacld);

		//---------------------------------------------------------------------
		//   form the strictly upper triangular part of the jacobian matrix
		//   perform the upper triangular solution
		//---------------------------------------------------------------------
		START_TIMER(t_jacu);
		for (int plane = nx+ny+nz-9; plane >= 0; plane--) {
			int klower = max(0, plane-(nx-3)-(ny-3));
			int kupper = min(plane, nz-3);
			int jlowermin = max(0, plane-kupper-(nx-3));
			int juppermax = min(plane, ny-3);

			jacu_buts_kernel<<<kupper-klower+1,juppermax-jlowermin+1>>>(plane, klower, jlowermin, u, rho_i, qs, rsd, nx, ny, nz);
		}
		STOP_TIMER(t_jacu);

		//---------------------------------------------------------------------
		//   update the variables
		//---------------------------------------------------------------------
		START_TIMER(t_add);
		ssor_kernel2<<<grid_yz,nx-2>>>(u, rsd, tmp, nx, ny, nz);
		STOP_TIMER(t_add);
	
		//---------------------------------------------------------------------
		//   compute the max-norms of newton iteration corrections
		//---------------------------------------------------------------------
		if (istep % inorm == 0) {
			double delunm[5];
			START_TIMER(t_l2norm);
			l2norm(rsd, delunm);
			STOP_TIMER(t_l2norm);
		}

		//---------------------------------------------------------------------
		//   compute the steady-state residuals
		//---------------------------------------------------------------------
		rhs();

		//---------------------------------------------------------------------
		//   compute the max-norms of newton iteration residuals
		//---------------------------------------------------------------------
		if (istep % inorm == 0) {
			START_TIMER(t_l2norm);
			l2norm(rsd, rsdnm);
			STOP_TIMER(t_l2norm);
		}

		//---------------------------------------------------------------------
		//   check the newton-iteration residuals against the tolerance levels
		//---------------------------------------------------------------------
		if (rsdnm[0] < tolrsd[0] && rsdnm[1] < tolrsd[1] && rsdnm[2] < tolrsd[2] && rsdnm[3] < tolrsd[3] && rsdnm[4] < tolrsd[4]) {
			printf("\n convergence was achieved after %4d pseudo-time steps\n", istep);
			break;
		}
	}

	HANDLE_ERROR(hipDeviceSynchronize());
	timers->timer_stop(0);
	maxtime = timers->timer_read(0);
}

__global__ static void rhs_kernel_init (const double *u, double *rsd, const double *frct, double *qs, double *rho_i, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double tmp;

	k = blockIdx.x;
	j = blockIdx.y;
	i = threadIdx.x;

	for (m = 0; m < 5; m++) rsd(m,i,j,k) = -frct(m,i,j,k);
	rho_i(i,j,k) = tmp = 1.0/u(0,i,j,k);
	qs(i,j,k) = 0.5*(u(1,i,j,k)*u(1,i,j,k) + u(2,i,j,k)*u(2,i,j,k) + u(3,i,j,k)*u(3,i,j,k))*tmp;
}

__global__ static void rhs_kernel_x (const double *u, double *rsd, const double *qs, const double *rho_i, const int nx, const int ny, const int nz) {
	int i, j, k, m, nthreads;
	double q, u21;
	__shared__ double flux[RHSX_BLOCK][5];
	__shared__ double utmp[RHSX_BLOCK*5], rtmp[RHSX_BLOCK*5], rhotmp[RHSX_BLOCK];
	__shared__ double u21i[RHSX_BLOCK], u31i[RHSX_BLOCK], u41i[RHSX_BLOCK], u51i[RHSX_BLOCK];


	k = blockIdx.x+1;
	j = blockIdx.y+1;
	i = threadIdx.x;

	using namespace gpu_mod;

	while (i < nx) {
		// load u, rsd and rho_i using coalesced memory access 
		// first compute number of threads executing this region 
		nthreads = nx-(i-threadIdx.x);
		if (nthreads > blockDim.x) nthreads = blockDim.x;
		m = threadIdx.x;
		utmp[m] = u(m%5, (i-threadIdx.x)+m/5, j, k);
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		m += nthreads;
		utmp[m] = u(m%5, (i-threadIdx.x)+m/5, j, k);
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		m += nthreads;
		utmp[m] = u(m%5, (i-threadIdx.x)+m/5, j, k);
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		m += nthreads;
		utmp[m] = u(m%5, (i-threadIdx.x)+m/5, j, k);
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		m += nthreads;
		utmp[m] = u(m%5, (i-threadIdx.x)+m/5, j, k);
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		rhotmp[threadIdx.x] = rho_i(i,j,k);
		__syncthreads();

		//---------------------------------------------------------------------
		//   xi-direction flux differences
		//---------------------------------------------------------------------
		flux[threadIdx.x][0] = utmp[threadIdx.x*5+1];
		u21 = utmp[threadIdx.x*5+1]*rhotmp[threadIdx.x];
		q = qs(i,j,k);
		flux[threadIdx.x][1] = utmp[threadIdx.x*5+1]*u21 + c2*(utmp[threadIdx.x*5+4]-q);
		flux[threadIdx.x][2] = utmp[threadIdx.x*5+2]*u21;
		flux[threadIdx.x][3] = utmp[threadIdx.x*5+3]*u21;
		flux[threadIdx.x][4] = (c1*utmp[threadIdx.x*5+4]-c2*q)*u21;
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSX_BLOCK-1 && i < nx-1) 
			for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] = rtmp[threadIdx.x*5+m] - tx2*(flux[threadIdx.x+1][m]-flux[threadIdx.x-1][m]);

		u21i[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+1];
		u31i[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+2];
		u41i[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+3];
		u51i[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+4];
		__syncthreads();
	
		if (threadIdx.x >= 1) {
			flux[threadIdx.x][1] = (4.0/3.0)*tx3*(u21i[threadIdx.x]-u21i[threadIdx.x-1]);
			flux[threadIdx.x][2] = tx3*(u31i[threadIdx.x]-u31i[threadIdx.x-1]);
			flux[threadIdx.x][3] = tx3*(u41i[threadIdx.x]-u41i[threadIdx.x-1]);
			flux[threadIdx.x][4] = 0.5*(1.0-c1*c5)*tx3*((u21i[threadIdx.x]*u21i[threadIdx.x]+u31i[threadIdx.x]*u31i[threadIdx.x]+u41i[threadIdx.x]*u41i[threadIdx.x]) - 
							(u21i[threadIdx.x-1]*u21i[threadIdx.x-1]+u31i[threadIdx.x-1]*u31i[threadIdx.x-1]+u41i[threadIdx.x-1]*u41i[threadIdx.x-1])) + 
							(1.0/6.0)*tx3*(u21i[threadIdx.x]*u21i[threadIdx.x]-u21i[threadIdx.x-1]*u21i[threadIdx.x-1]) + c1*c5*tx3*(u51i[threadIdx.x]-u51i[threadIdx.x-1]);
		}
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSX_BLOCK-1 && i < nx-1) {
			rtmp[threadIdx.x*5+0] += dx1*tx1*(utmp[threadIdx.x*5-5]-2.0*utmp[threadIdx.x*5+0]+utmp[threadIdx.x*5+5]);
			rtmp[threadIdx.x*5+1] += tx3*c3*c4*(flux[threadIdx.x+1][1]-flux[threadIdx.x][1]) + dx2*tx1*(utmp[threadIdx.x*5-4]-2.0*utmp[threadIdx.x*5+1]+utmp[threadIdx.x*5+6]);
			rtmp[threadIdx.x*5+2] += tx3*c3*c4*(flux[threadIdx.x+1][2]-flux[threadIdx.x][2]) + dx3*tx1*(utmp[threadIdx.x*5-3]-2.0*utmp[threadIdx.x*5+2]+utmp[threadIdx.x*5+7]);
			rtmp[threadIdx.x*5+3] += tx3*c3*c4*(flux[threadIdx.x+1][3]-flux[threadIdx.x][3]) + dx4*tx1*(utmp[threadIdx.x*5-2]-2.0*utmp[threadIdx.x*5+3]+utmp[threadIdx.x*5+8]);
			rtmp[threadIdx.x*5+4] += tx3*c3*c4*(flux[threadIdx.x+1][4]-flux[threadIdx.x][4]) + dx5*tx1*(utmp[threadIdx.x*5-1]-2.0*utmp[threadIdx.x*5+4]+utmp[threadIdx.x*5+9]);

			//---------------------------------------------------------------------
			//   Fourth-order dissipation
			//---------------------------------------------------------------------
			if (i == 1) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(5.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]+u(m,3,j,k));
			if (i == 2) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(-4.0*utmp[threadIdx.x*5+m-5]+6.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]+u(m,4,j,k));
			if (i >= 3 && i < nx-3) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(u(m,i-2,j,k)-4.0*utmp[threadIdx.x*5+m-5]+6.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]+u(m,i+2,j,k));
			if (i == nx-3) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(u(m,nx-5,j,k)-4.0*utmp[threadIdx.x*5+m-5]+6.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]);
			if (i == nx-2) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(u(m,nx-4,j,k)-4.0*utmp[threadIdx.x*5+m-5]+5.0*utmp[threadIdx.x*5+m]);
		}

		// store the updated rsd values using a coalesced write pattern
		// Note: this stores more values than actually computed but it leads to a more efficient execution
		m = threadIdx.x;
		rsd(m%5, (i-threadIdx.x)+m/5, j, k) = rtmp[m];
		m += nthreads;
		rsd(m%5, (i-threadIdx.x)+m/5, j, k) = rtmp[m];
		m += nthreads;
		rsd(m%5, (i-threadIdx.x)+m/5, j, k) = rtmp[m];
		m += nthreads;
		rsd(m%5, (i-threadIdx.x)+m/5, j, k) = rtmp[m];
		m += nthreads;
		rsd(m%5, (i-threadIdx.x)+m/5, j, k) = rtmp[m];

		i += RHSX_BLOCK-2;
	}
}

__global__ static void rhs_kernel_y (const double *u, double *rsd, const double *qs, const double *rho_i, const int nx, const int ny, const int nz) {
	int i, j, k, m, nthreads;
	double q, u31;
	__shared__ double flux[RHSY_BLOCK][5];
	__shared__ double utmp[RHSY_BLOCK*5], rtmp[RHSY_BLOCK*5], rhotmp[RHSY_BLOCK];
	__shared__ double u21j[RHSY_BLOCK], u31j[RHSY_BLOCK], u41j[RHSY_BLOCK], u51j[RHSY_BLOCK];

	k = blockIdx.x+1;
	i = blockIdx.y+1;
	j = threadIdx.x;

	using namespace gpu_mod;

	while(j < ny) {
		// load u, rsd and rho_i using coalesced memory access along the m-axis
		// first compute number of threads executing this region 
		nthreads = ny-(j-threadIdx.x);
		if (nthreads > blockDim.x) nthreads = blockDim.x;
		m = threadIdx.x;
		utmp[m] = u(m%5, i, (j-threadIdx.x)+m/5, k);
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		m += nthreads;
		utmp[m] = u(m%5, i, (j-threadIdx.x)+m/5, k);
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		m += nthreads;
		utmp[m] = u(m%5, i, (j-threadIdx.x)+m/5, k);
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		m += nthreads;
		utmp[m] = u(m%5, i, (j-threadIdx.x)+m/5, k);
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		m += nthreads;
		utmp[m] = u(m%5, i, (j-threadIdx.x)+m/5, k);
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		rhotmp[threadIdx.x] = rho_i(i,j,k);
		__syncthreads();

		//---------------------------------------------------------------------
		//   eta-direction flux differences
		//---------------------------------------------------------------------
		flux[threadIdx.x][0] = utmp[threadIdx.x*5+2];
		u31 = utmp[threadIdx.x*5+2] * rhotmp[threadIdx.x];
		q = qs(i,j,k);
		flux[threadIdx.x][1] = utmp[threadIdx.x*5+1]*u31;
		flux[threadIdx.x][2] = utmp[threadIdx.x*5+2]*u31 + c2*(utmp[threadIdx.x*5+4]-q);
		flux[threadIdx.x][3] = utmp[threadIdx.x*5+3]*u31;
		flux[threadIdx.x][4] = (c1*utmp[threadIdx.x*5+4]-c2*q)*u31;
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSY_BLOCK-1 && j < ny-1) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] = rtmp[threadIdx.x*5+m] - ty2*(flux[threadIdx.x+1][m]-flux[threadIdx.x-1][m]);
		
		u21j[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+1];
		u31j[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+2];
		u41j[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+3];
		u51j[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+4];
		__syncthreads();

		if (threadIdx.x >= 1) {
			flux[threadIdx.x][1] = ty3*(u21j[threadIdx.x]-u21j[threadIdx.x-1]);
			flux[threadIdx.x][2] = (4.0/3.0)*ty3*(u31j[threadIdx.x]-u31j[threadIdx.x-1]);
			flux[threadIdx.x][3] = ty3*(u41j[threadIdx.x]-u41j[threadIdx.x-1]);
			flux[threadIdx.x][4] = 0.5*(1.0-c1*c5)*ty3*((u21j[threadIdx.x]*u21j[threadIdx.x]+u31j[threadIdx.x]*u31j[threadIdx.x]+u41j[threadIdx.x]*u41j[threadIdx.x]) - 
							(u21j[threadIdx.x-1]*u21j[threadIdx.x-1]+u31j[threadIdx.x-1]*u31j[threadIdx.x-1]+u41j[threadIdx.x-1]*u41j[threadIdx.x-1])) + 
							(1.0/6.0)*ty3*(u31j[threadIdx.x]*u31j[threadIdx.x]-u31j[threadIdx.x-1]*u31j[threadIdx.x-1]) + c1*c5*ty3*(u51j[threadIdx.x]-u51j[threadIdx.x-1]);
		}
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSY_BLOCK-1 && j < ny-1) {
			rtmp[threadIdx.x*5+0] += dy1*ty1*(utmp[5*(threadIdx.x-1)]-2.0*utmp[threadIdx.x*5+0]+utmp[5*(threadIdx.x+1)]);
			rtmp[threadIdx.x*5+1] += ty3*c3*c4*(flux[threadIdx.x+1][1]-flux[threadIdx.x][1]) + dy2*ty1*(utmp[5*threadIdx.x-4]-2.0*utmp[threadIdx.x*5+1]+utmp[5*threadIdx.x+6]);
			rtmp[threadIdx.x*5+2] += ty3*c3*c4*(flux[threadIdx.x+1][2]-flux[threadIdx.x][2]) + dy3*ty1*(utmp[5*threadIdx.x-3]-2.0*utmp[threadIdx.x*5+2]+utmp[5*threadIdx.x+7]);
			rtmp[threadIdx.x*5+3] += ty3*c3*c4*(flux[threadIdx.x+1][3]-flux[threadIdx.x][3]) + dy4*ty1*(utmp[5*threadIdx.x-2]-2.0*utmp[threadIdx.x*5+3]+utmp[5*threadIdx.x+8]);
			rtmp[threadIdx.x*5+4] += ty3*c3*c4*(flux[threadIdx.x+1][4]-flux[threadIdx.x][4]) + dy5*ty1*(utmp[5*threadIdx.x-1]-2.0*utmp[threadIdx.x*5+4]+utmp[5*threadIdx.x+9]);

			//---------------------------------------------------------------------
			//   fourth-order dissipation
			//---------------------------------------------------------------------
			if (j == 1) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(5.0*utmp[threadIdx.x*5+m]-4.0*utmp[5*threadIdx.x+m+5]+u(m,i,3,k));
			if (j == 2) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(-4.0*utmp[threadIdx.x*5+m-5]+6.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]+u(m,i,4,k));
			if (j >= 3 && j < ny-3) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(u(m,i,j-2,k)-4.0*utmp[threadIdx.x*5+m-5]+6.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]+u(m,i,j+2,k));
			if (j == ny-3) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(u(m,i,ny-5,k)-4.0*utmp[threadIdx.x*5+m-5]+6.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]);
			if (j == ny-2) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(u(m,i,ny-4,k)-4.0*utmp[threadIdx.x*5+m-5]+5.0*utmp[threadIdx.x*5+m]);
		}

		// store the updated rsd values using a coalesced write pattern
		// Note: this stores more values than actually computed but it leads to a more efficient execution
		m = threadIdx.x;
		rsd(m%5, i, (j-threadIdx.x)+m/5, k) = rtmp[m];
		m += nthreads;
		rsd(m%5, i, (j-threadIdx.x)+m/5, k) = rtmp[m];
		m += nthreads;
		rsd(m%5, i, (j-threadIdx.x)+m/5, k) = rtmp[m];
		m += nthreads;
		rsd(m%5, i, (j-threadIdx.x)+m/5, k) = rtmp[m];
		m += nthreads;
		rsd(m%5, i, (j-threadIdx.x)+m/5, k) = rtmp[m];

		j += RHSY_BLOCK-2;
	}
}

__global__ static void rhs_kernel_z (const double *u, double *rsd, const double *qs, const double *rho_i, const int nx, const int ny, const int nz) {
	int i, j, k, m, nthreads;
	double q, u41;
	__shared__ double flux[RHSZ_BLOCK][5];
	__shared__ double utmp[RHSZ_BLOCK*5], rtmp[RHSZ_BLOCK*5], rhotmp[RHSZ_BLOCK];
	__shared__ double u21k[RHSZ_BLOCK], u31k[RHSZ_BLOCK], u41k[RHSZ_BLOCK], u51k[RHSZ_BLOCK];

	j = blockIdx.x+1;
	i = blockIdx.y+1;
	k = threadIdx.x;

	using namespace gpu_mod;

	while (k < nz) {
		// load u, rsd and rho_i using coalesced memory access along the m-axis
		// first compute number of threads executing this region 
		nthreads = (nz-(k-threadIdx.x));
		if (nthreads > blockDim.x) nthreads = blockDim.x;
		m = threadIdx.x;
		utmp[m] = u(m%5, i, j, (k-threadIdx.x)+m/5);
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		m += nthreads;
		utmp[m] = u(m%5, i, j, (k-threadIdx.x)+m/5);
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		m += nthreads;
		utmp[m] = u(m%5, i, j, (k-threadIdx.x)+m/5);
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		m += nthreads;
		utmp[m] = u(m%5, i, j, (k-threadIdx.x)+m/5);
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		m += nthreads;
		utmp[m] = u(m%5, i, j, (k-threadIdx.x)+m/5);
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		rhotmp[threadIdx.x] = rho_i(i,j,k);
		__syncthreads();

		//---------------------------------------------------------------------
		//   zeta-direction flux differences
		//---------------------------------------------------------------------
		flux[threadIdx.x][0] = utmp[threadIdx.x*5+3];
		u41 = utmp[threadIdx.x*5+3]*rhotmp[threadIdx.x];
		q = qs(i,j,k);
		flux[threadIdx.x][1] = utmp[threadIdx.x*5+1]*u41;
		flux[threadIdx.x][2] = utmp[threadIdx.x*5+2]*u41;
		flux[threadIdx.x][3] = utmp[threadIdx.x*5+3]*u41 + c2*(utmp[threadIdx.x*5+4]-q);
		flux[threadIdx.x][4] = (c1*utmp[threadIdx.x*5+4]-c2*q)*u41;
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSZ_BLOCK-1 && k < nz-1) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] = rtmp[threadIdx.x*5+m] - tz2*(flux[threadIdx.x+1][m]-flux[threadIdx.x-1][m]);

		u21k[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+1];
		u31k[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+2];
		u41k[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+3];
		u51k[threadIdx.x] = rhotmp[threadIdx.x]*utmp[threadIdx.x*5+4];
		__syncthreads();

		if (threadIdx.x >= 1) {
			flux[threadIdx.x][1] = tz3*(u21k[threadIdx.x]-u21k[threadIdx.x-1]);
			flux[threadIdx.x][2] = tz3*(u31k[threadIdx.x]-u31k[threadIdx.x-1]);
			flux[threadIdx.x][3] = (4.0/3.0)*tz3*(u41k[threadIdx.x]-u41k[threadIdx.x-1]);
			flux[threadIdx.x][4] = 0.5*(1.0-c1*c5)*tz3*((u21k[threadIdx.x]*u21k[threadIdx.x]+u31k[threadIdx.x]*u31k[threadIdx.x]+u41k[threadIdx.x]*u41k[threadIdx.x])-
							(u21k[threadIdx.x-1]*u21k[threadIdx.x-1]+u31k[threadIdx.x-1]*u31k[threadIdx.x-1]+u41k[threadIdx.x-1]*u41k[threadIdx.x-1])) + 
							(1.0/6.0)*tz3*(u41k[threadIdx.x]*u41k[threadIdx.x]-u41k[threadIdx.x-1]*u41k[threadIdx.x-1]) + c1*c5*tz3*(u51k[threadIdx.x]-u51k[threadIdx.x-1]);
		}
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSZ_BLOCK-1 && k < nz-1) {
			rtmp[threadIdx.x*5+0] += dz1*tz1*(utmp[threadIdx.x*5-5]-2.0*utmp[threadIdx.x*5+0]+utmp[threadIdx.x*5+5]);
			rtmp[threadIdx.x*5+1] += tz3*c3*c4*(flux[threadIdx.x+1][1]-flux[threadIdx.x][1]) + dz2*tz1*(utmp[5*threadIdx.x-4]-2.0*utmp[threadIdx.x*5+1]+utmp[threadIdx.x*5+6]);
			rtmp[threadIdx.x*5+2] += tz3*c3*c4*(flux[threadIdx.x+1][2]-flux[threadIdx.x][2]) + dz3*tz1*(utmp[5*threadIdx.x-3]-2.0*utmp[threadIdx.x*5+2]+utmp[threadIdx.x*5+7]);
			rtmp[threadIdx.x*5+3] += tz3*c3*c4*(flux[threadIdx.x+1][3]-flux[threadIdx.x][3]) + dz4*tz1*(utmp[5*threadIdx.x-2]-2.0*utmp[threadIdx.x*5+3]+utmp[threadIdx.x*5+8]);
			rtmp[threadIdx.x*5+4] += tz3*c3*c4*(flux[threadIdx.x+1][4]-flux[threadIdx.x][4]) + dz5*tz1*(utmp[5*threadIdx.x-1]-2.0*utmp[threadIdx.x*5+4]+utmp[threadIdx.x*5+9]);

			//---------------------------------------------------------------------
			//   fourth-order dissipation
			//---------------------------------------------------------------------
			if (k == 1) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(5.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]+u(m,i,j,3));
			if (k == 2) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(-4.0*utmp[threadIdx.x*5+m-5]+6.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]+u(m,i,j,4));
			if (k >= 3 && k < nz-3) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(u(m,i,j,k-2)-4.0*utmp[threadIdx.x*5+m-5]+6.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]+u(m,i,j,k+2));
			if (k == nz-3) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(u(m,i,j,nz-5)-4.0*utmp[threadIdx.x*5+m-5]+6.0*utmp[threadIdx.x*5+m]-4.0*utmp[threadIdx.x*5+m+5]);
			if (k == nz-2) for (m = 0; m < 5; m++) rtmp[threadIdx.x*5+m] -= dssp*(u(m,i,j,nz-4)-4.0*utmp[threadIdx.x*5+m-5]+5.0*utmp[threadIdx.x*5+m]);
		}

		// store the updated rsd values using a coalesced write pattern
		// Note: this stores more values than actually computed but it leads to a more efficient execution
		m = threadIdx.x;
		rsd(m%5, i, j, (k-threadIdx.x)+m/5) = rtmp[m];
		m += nthreads;
		rsd(m%5, i, j, (k-threadIdx.x)+m/5) = rtmp[m];
		m += nthreads;
		rsd(m%5, i, j, (k-threadIdx.x)+m/5) = rtmp[m];
		m += nthreads;
		rsd(m%5, i, j, (k-threadIdx.x)+m/5) = rtmp[m];
		m += nthreads;
		rsd(m%5, i, j, (k-threadIdx.x)+m/5) = rtmp[m];

		k += RHSZ_BLOCK-2;
	}
}

void LU::rhs() {
	dim3 grid(nz,ny);
	dim3 grid_yz(nz-2,ny-2);
	dim3 grid_xz(nz-2,nx-2);
	dim3 grid_xy(ny-2,nx-2);

	START_TIMER(t_rhs);
	rhs_kernel_init<<<grid,nx>>>(u, rsd, frct, qs, rho_i, nx, ny, nz);

	//---------------------------------------------------------------------
	//   xi-direction flux differences
	//---------------------------------------------------------------------
	START_TIMER(t_rhsx);
	rhs_kernel_x<<<grid_yz, min(nx,RHSX_BLOCK)>>>(u, rsd, qs, rho_i, nx, ny, nz);
	STOP_TIMER(t_rhsx);

	//---------------------------------------------------------------------
	//   eta-direction flux differences
	//---------------------------------------------------------------------
	START_TIMER(t_rhsy);
	rhs_kernel_y<<<grid_xz, min(ny,RHSY_BLOCK)>>>(u, rsd, qs, rho_i, nx, ny, nz);
	STOP_TIMER(t_rhsy);

	//---------------------------------------------------------------------
	//   zeta-direction flux differences
	//---------------------------------------------------------------------
	START_TIMER(t_rhsz);
	rhs_kernel_z<<<grid_xy, min(nz,RHSZ_BLOCK)>>>(u, rsd, qs, rho_i, nx, ny, nz);
	STOP_TIMER(t_rhsz);
	STOP_TIMER(t_rhs);
}

__global__ static void l2norm_kernel (const double *v, double *sum, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	__shared__ double sum_loc[5*NORM_BLOCK];

	k = blockIdx.x+1;
	j = blockIdx.y+1;
	i = threadIdx.x+1;
	for (m = 0; m < 5; m++) sum_loc[m+5*threadIdx.x] = 0.0;

	while (i < nx-1) {
		for (m = 0; m < 5; m++)	sum_loc[m+5*threadIdx.x] += v(m,i,j,k)*v(m,i,j,k);
		i += blockDim.x;
	}

	// reduction in x direction
	i = threadIdx.x;
	int loc_max = blockDim.x;
	int dist = (loc_max+1)/2;
	__syncthreads();
	while (loc_max > 1) {
		if (i < dist && i+dist < loc_max)
			for (m = 0; m < 5; m++) sum_loc[m+5*i] += sum_loc[m+5*(i+dist)];
		loc_max = dist;
		dist = (dist+1)/2;
		__syncthreads();
	}

	if (i == 0) for (m = 0; m < 5; m++) sum[m+5*(blockIdx.y+gridDim.y*blockIdx.x)] = sum_loc[m];
}

__global__ static void norm_reduce(double *rms, const int size) {
	int i, m, loc_max, dist;
	__shared__ double buffer[5*NORM_BLOCK];

	i = threadIdx.x;
	for (m = 0; m < 5; m++) buffer[m+5*i] = 0.0;

	while (i < size) {
		for (m = 0; m < 5; m++) buffer[m+5*threadIdx.x] += rms[m+5*i];
		i += blockDim.x;
	}

	loc_max = blockDim.x;
	dist = (loc_max+1)/2;
	i = threadIdx.x;
	__syncthreads();
	while (loc_max > 1) {
		if (i < dist && i+dist < loc_max) 
			for (m = 0; m < 5; m++) buffer[m+5*i] += buffer[m+5*(i+dist)];
		loc_max = dist;
		dist = (dist+1)/2;
		__syncthreads();
	}

	if (threadIdx.x < 5) rms[threadIdx.x] = buffer[threadIdx.x];
}

void LU::l2norm (const double *v, double *sum) {
	dim3 grid(nz-2,ny-2);

	l2norm_kernel<<<grid,min(nx-2,NORM_BLOCK)>>>(v, dev_norm_buf, nx, ny, nz);
	norm_reduce<<<1,NORM_BLOCK>>>(dev_norm_buf, (nz-2)*(ny-2));
	HANDLE_ERROR(hipMemcpy(sum, dev_norm_buf, 5*sizeof(double), hipMemcpyDeviceToHost));

	for (int m = 0; m < 5; m++) sum[m] = sqrt(sum[m]/((double)(nz-2)*(double)(ny-2)*(double)(nx-2)));
}

//---------------------------------------------------------------------
//
//   compute the exact solution at (i,j,k)
//
//---------------------------------------------------------------------
__device__ static void exact_kernel (const int i, const int j, const int k, double *u000ijk, const int nx, const int ny, const int nz) {
	int m;
	double xi, eta, zeta;

	using namespace gpu_mod;

	xi = (double)i/(double)(nx-1);
	eta = (double)j/(double)(ny-1);
	zeta = (double)k/(double)(nz-1);

	for (m = 0; m < 5; m++) u000ijk[m] = ce[m+0*5]+(ce[m+1*5]+(ce[m+4*5]+(ce[m+7*5]+ce[m+10*5]*xi)*xi)*xi)*xi + (ce[m+2*5]+(ce[m+5*5]+(ce[m+8*5]+ce[m+11*5]*eta)*eta)*eta)*eta + (ce[m+3*5]+(ce[m+6*5]+(ce[m+9*5]+ce[m+12*5]*zeta)*zeta)*zeta)*zeta;
}

//---------------------------------------------------------------------
//
//   compute the solution error
//
//---------------------------------------------------------------------
__global__ static void error_kernel (const double *u, double *errnm, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double tmp, u000ijk[5];
	__shared__ double errnm_loc[5*NORM_BLOCK];

	k = blockIdx.x+1;
	j = blockIdx.y+1;
	i = threadIdx.x+1;
	for (m = 0; m < 5; m++) errnm_loc[m+5*threadIdx.x] = 0.0;

	while (i < nx-1) {
		exact_kernel(i, j, k, u000ijk, nx, ny, nz);
		for (m = 0; m < 5; m++) {
			tmp = u000ijk[m]-u(m,i,j,k);
			errnm_loc[m+5*threadIdx.x] += tmp * tmp;
		}
		i += blockDim.x;
	}

	// reduce in x direction
	i = threadIdx.x;
	int loc_max = blockDim.x;
	int dist = (loc_max+1)/2;
	__syncthreads();
	while (loc_max > 1) {
		if (i < dist && i+dist < loc_max)
			for (m = 0; m < 5; m++) errnm_loc[m+5*i] += errnm_loc[m+5*(i+dist)];
		loc_max = dist;
		dist = (dist+1)/2;
		__syncthreads();
	}

	if (i == 0) for (m = 0; m < 5; m++) errnm[m+5*(blockIdx.y+gridDim.y*blockIdx.x)] = errnm_loc[m];
}

void LU::error() {
	dim3 grid(nz-2,ny-2);

	error_kernel<<<grid, min(nx-2,NORM_BLOCK)>>>(u, dev_norm_buf, nx, ny, nz);
	norm_reduce<<<1,NORM_BLOCK>>>(dev_norm_buf, (nz-2)*(ny-2));
	HANDLE_ERROR(hipMemcpy(errnm, dev_norm_buf, 5*sizeof(double), hipMemcpyDeviceToHost));

	for (int m = 0; m < 5; m++) errnm[m] = sqrt(errnm[m]/((double)(nz-2)*(double)(ny-2)*(double)(nx-2)));
}

__global__ static void pintgr_kernel_1 (const double *u, double *frc, const int nx, const int ny, const int nz) {
	int i, j, k;
	__shared__ double phi1[PINTGR_BLOCK][PINTGR_BLOCK], phi2[PINTGR_BLOCK][PINTGR_BLOCK], frc1[PINTGR_BLOCK*PINTGR_BLOCK];

	i = blockIdx.x*(PINTGR_BLOCK-1)+threadIdx.x+1;
	j = blockIdx.y*(PINTGR_BLOCK-1)+threadIdx.y+1;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	//   initialize
	//---------------------------------------------------------------------
	if (j < ny-2 && i < nx-1) {
		k = 2;
		phi1[threadIdx.x][threadIdx.y] = c2*(u(4,i,j,k) - 0.5*(u(1,i,j,k)*u(1,i,j,k)+u(2,i,j,k)*u(2,i,j,k)+u(3,i,j,k)*u(3,i,j,k))/u(0,i,j,k));
		k = nz-2;
		phi2[threadIdx.x][threadIdx.y] = c2*(u(4,i,j,k) - 0.5*(u(1,i,j,k)*u(1,i,j,k)+u(2,i,j,k)*u(2,i,j,k)+u(3,i,j,k)*u(3,i,j,k))/u(0,i,j,k));
	}
	__syncthreads();

	frc1[threadIdx.y*blockDim.x+threadIdx.x] = 0.0;
	if (j < ny-3 && i < nx-2 && threadIdx.x < PINTGR_BLOCK-1 && threadIdx.y < PINTGR_BLOCK-1) 
		frc1[threadIdx.y*blockDim.x+threadIdx.x] = phi1[threadIdx.x][threadIdx.y]+phi1[threadIdx.x+1][threadIdx.y]+phi1[threadIdx.x][threadIdx.y+1]+phi1[threadIdx.x+1][threadIdx.y+1]+
								phi2[threadIdx.x][threadIdx.y]+phi2[threadIdx.x+1][threadIdx.y]+phi2[threadIdx.x][threadIdx.y+1]+phi2[threadIdx.x+1][threadIdx.y+1];

	// reduce
	int loc_max = blockDim.x*blockDim.y;
	int dist = (loc_max+1)/2;
	i = threadIdx.y*blockDim.x+threadIdx.x;
	__syncthreads();
	while (loc_max > 1) {
		if (i < dist && i+dist < loc_max) frc1[i] += frc1[i+dist];
		loc_max = dist;
		dist = (dist+1)/2;
		__syncthreads();
	}
	if (i == 0) frc[blockIdx.y*gridDim.x+blockIdx.x] = frc1[0]*dxi*deta;
}

__global__ static void pintgr_kernel_2 (const double *u, double *frc, const int nx, const int ny, const int nz) {
	int i, j, k, kp, ip;
	__shared__ double phi1[PINTGR_BLOCK][PINTGR_BLOCK], phi2[PINTGR_BLOCK][PINTGR_BLOCK], frc2[PINTGR_BLOCK*PINTGR_BLOCK];

	i = blockIdx.x*(PINTGR_BLOCK-1)+1;
	k = blockIdx.y*(PINTGR_BLOCK-1)+2;
	kp = threadIdx.y;
	ip = threadIdx.x;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	//   initialize
	//---------------------------------------------------------------------
	if (k+kp < nz-1 && i+ip < nx-1) {
		j = 1;
		phi1[kp][ip] = c2*(u(4,i+ip,j,k+kp) - 0.5*(u(1,i+ip,j,k+kp)*u(1,i+ip,j,k+kp)+u(2,i+ip,j,k+kp)*u(2,i+ip,j,k+kp)+u(3,i+ip,j,k+kp)*u(3,i+ip,j,k+kp))/u(0,i+ip,j,k+kp));
		j = ny-3;
		phi2[kp][ip] = c2*(u(4,i+ip,j,k+kp) - 0.5*(u(1,i+ip,j,k+kp)*u(1,i+ip,j,k+kp)+u(2,i+ip,j,k+kp)*u(2,i+ip,j,k+kp)+u(3,i+ip,j,k+kp)*u(3,i+ip,j,k+kp))/u(0,i+ip,j,k+kp));
	}
	__syncthreads();

	frc2[kp*PINTGR_BLOCK+ip] = 0.0;
	if (k+kp < nz-2 && i+ip < nx-2 && kp < PINTGR_BLOCK-1 && ip < PINTGR_BLOCK-1)
		frc2[kp*PINTGR_BLOCK+ip] += phi1[kp][ip] + phi1[kp+1][ip] + phi1[kp][ip+1] + phi1[kp+1][ip+1] + phi2[kp][ip] + phi2[kp+1][ip] + phi2[kp][ip+1] + phi2[kp+1][ip+1];

	// reduce
	int loc_max = blockDim.x*blockDim.y;
	int dist = (loc_max+1)/2;
	i = threadIdx.y*blockDim.x+threadIdx.x;
	__syncthreads();
	while (loc_max > 1) {
		if (i < dist && i+dist < loc_max) frc2[i] += frc2[i+dist];
		loc_max = dist;
		dist = (dist+1)/2;
		__syncthreads();
	}
	if (i == 0) frc[blockIdx.y*gridDim.x+blockIdx.x] = frc2[0]*dxi*dzeta;
}

__global__ static void pintgr_kernel_3 (const double *u, double *frc, const int nx, const int ny, const int nz) {
	int j, k, jp, kp;
	__shared__ double phi1[PINTGR_BLOCK][PINTGR_BLOCK], phi2[PINTGR_BLOCK][PINTGR_BLOCK], frc3[PINTGR_BLOCK*PINTGR_BLOCK];

	j = blockIdx.x*(PINTGR_BLOCK-1)+1;
	k = blockIdx.y*(PINTGR_BLOCK-1)+2;
	kp = threadIdx.y;
	jp = threadIdx.x;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	//   initialize
	//---------------------------------------------------------------------
	if (k+kp < nz-1 && j+jp < ny-2) {
		phi1[kp][jp] = c2*(u(4,1,j+jp,k+kp) - 0.5*(u(1,1,j+jp,k+kp)*u(1,1,j+jp,k+kp)+u(2,1,j+jp,k+kp)*u(2,1,j+jp,k+kp)+u(3,1,j+jp,k+kp)*u(3,1,j+jp,k+kp))/u(0,1,j+jp,k+kp));
		phi2[kp][jp] = c2*(u(4,nx-2,j+jp,k+kp) - 0.5*(u(1,nx-2,j+jp,k+kp)*u(1,nx-2,j+jp,k+kp)+u(2,nx-2,j+jp,k+kp)*u(2,nx-2,j+jp,k+kp)+u(3,nx-2,j+jp,k+kp)*u(3,nx-2,j+jp,k+kp))/u(0,nx-2,j+jp,k+kp));
	}
	__syncthreads();

	frc3[kp*PINTGR_BLOCK+jp] = 0.0;
	if (k+kp < nz-2 && j+jp < ny-3 && kp < PINTGR_BLOCK-1 && jp < PINTGR_BLOCK-1)
		frc3[kp*PINTGR_BLOCK+jp] = phi1[kp][jp] + phi1[kp+1][jp] + phi1[kp][jp+1] + phi1[kp+1][jp+1] + phi2[kp][jp] + phi2[kp+1][jp] + phi2[kp][jp+1] + phi2[kp+1][jp+1];

	// reduce
	int loc_max = blockDim.x*blockDim.y;
	int dist = (loc_max+1)/2;
	j = threadIdx.y*blockDim.x+threadIdx.x;
	__syncthreads();
	while (loc_max > 1) {
		if (j < dist && j+dist < loc_max) frc3[j] += frc3[j+dist];
		loc_max = dist;
		dist = (dist+1)/2;
		__syncthreads();
	}
	if (j == 0) frc[blockIdx.y*gridDim.x+blockIdx.x] = frc3[0]*deta*dzeta;
}

__global__ static void pintgr_reduce (double *frc, const int num) {
	int i, loc_max, dist;
	__shared__ double buffer[PINTGR_BLOCK*PINTGR_BLOCK];

	i = threadIdx.x;
	buffer[i] = 0.0;

	while (i < num) {
		buffer[threadIdx.x] += frc[i];
		i += blockDim.x;
	}

	loc_max = blockDim.x;
	dist = (loc_max+1)/2;
	i = threadIdx.x;
	__syncthreads();
	while (loc_max > 1) {
		if (i < dist && i+dist < loc_max) buffer[i] += buffer[i+dist];
		loc_max = dist;
		dist = (dist+1)/2;
		__syncthreads();
	}

	if (i == 0) frc[0] = .25*buffer[0];
}

void LU::pintgr() {
	dim3 grid(PINTGR_BLOCK,PINTGR_BLOCK);
	dim3 grid_xy((nx-3+PINTGR_BLOCK-2)/(PINTGR_BLOCK-1), (ny-4+PINTGR_BLOCK-2)/(PINTGR_BLOCK-1));
	int grid1_size = grid_xy.x*grid_xy.y;
	pintgr_kernel_1<<<grid_xy,grid>>>(u, dev_norm_buf, nx, ny, nz);

	dim3 grid_xz((nx-3+PINTGR_BLOCK-2)/(PINTGR_BLOCK-1), (nz-4+PINTGR_BLOCK-2)/(PINTGR_BLOCK-1));
	int grid2_size = grid_xz.x*grid_xz.y;
	pintgr_kernel_2<<<grid_xz,grid>>>(u, dev_norm_buf+grid1_size, nx, ny, nz);

	dim3 grid_yz((ny-4+PINTGR_BLOCK-2)/(PINTGR_BLOCK-1), (nz-4+PINTGR_BLOCK-2)/(PINTGR_BLOCK-1));
	int grid3_size = grid_yz.x*grid_yz.y;
	pintgr_kernel_3<<<grid_yz,grid>>>(u, dev_norm_buf+grid1_size+grid2_size, nx, ny, nz);

	pintgr_reduce<<<1,PINTGR_BLOCK*PINTGR_BLOCK>>>(dev_norm_buf, grid1_size+grid2_size+grid3_size);
	HANDLE_ERROR(hipMemcpy(&frc, dev_norm_buf, sizeof(double), hipMemcpyDeviceToHost));
}

__global__ static void setbv_kernel_x (double *u, const int nx, const int ny, const int nz) {
	int j, k, m;
	double temp1[5], temp2[5];

	k = blockIdx.x;
	j = threadIdx.x;

	//---------------------------------------------------------------------
	//   set the dependent variable values along east and west faces
	//---------------------------------------------------------------------
	exact_kernel(0, j, k, temp1, nx, ny, nz);
	exact_kernel(nx-1, j, k, temp2, nx, ny, nz);
	for (m = 0; m < 5; m++) {
		u(m,0,j,k) = temp1[m];
		u(m,nx-1,j,k) = temp2[m];
	}
}

__global__ static void setbv_kernel_y (double *u, const int nx, const int ny, const int nz) {
	int i, k, m;
	double temp1[5], temp2[5];

	k = blockIdx.x;
	i = threadIdx.x;

	//---------------------------------------------------------------------
	//   set the dependent variable values along north and south faces
	//---------------------------------------------------------------------
	exact_kernel(i, 0, k, temp1, nx, ny, nz);
	exact_kernel(i, ny-1, k, temp2, nx, ny, nz);
	for (m = 0; m < 5; m++) {
		u(m,i,0,k) = temp1[m];
		u(m,i,ny-1,k) = temp2[m];
	}
}

__global__ static void setbv_kernel_z (double *u, const int nx, const int ny, const int nz) {
	int i, j, m;
	double temp1[5], temp2[5];

	j = blockIdx.x;
	i = threadIdx.x;

	//---------------------------------------------------------------------
	//   set the dependent variable values along the top and bottom faces
	//---------------------------------------------------------------------
	exact_kernel(i, j, 0, temp1, nx, ny, nz);
	exact_kernel(i, j, nz-1, temp2, nx, ny, nz);
	for (m = 0; m < 5; m++) {
		u(m,i,j,0) = temp1[m];
		u(m,i,j,nz-1) = temp2[m];
	}
}

//---------------------------------------------------------------------
//   set the boundary values of dependent variables
//---------------------------------------------------------------------
void LU::setbv() {
	setbv_kernel_z<<<ny,nx>>>(u, nx, ny, nz);
	setbv_kernel_y<<<nz,nx>>>(u, nx, ny, nz);
	setbv_kernel_x<<<nz,ny>>>(u, nx, ny, nz);
}

__global__ static void setiv_kernel (double *u, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, pxi, peta, pzeta;
	double ue_1jk[5], ue_nx0jk[5], ue_i1k[5], ue_iny0k[5], ue_ij1[5], ue_ijnz[5];

	k = blockIdx.x+1;
	j = blockIdx.y+1;
	i = threadIdx.x+1;

	zeta = (double)k/(double)(nz-1);
	eta = (double)j/(double)(ny-1);
	xi = (double)i/(double)(nx-1);
	exact_kernel(0, j, k, ue_1jk, nx, ny, nz);
	exact_kernel(nx-1, j, k, ue_nx0jk, nx, ny, nz);
	exact_kernel(i, 0, k, ue_i1k, nx, ny, nz);
	exact_kernel(i, ny-1, k, ue_iny0k, nx, ny, nz);
	exact_kernel(i, j, 0, ue_ij1, nx, ny, nz);
	exact_kernel(i, j, nz-1, ue_ijnz, nx, ny, nz);
	for (m = 0; m < 5; m++) {
		pxi = (1.0-xi)*ue_1jk[m] + xi*ue_nx0jk[m];
		peta = (1.0-eta)*ue_i1k[m] + eta*ue_iny0k[m];
		pzeta = (1.0-zeta)*ue_ij1[m] + zeta*ue_ijnz[m];

		u(m,i,j,k) = pxi + peta + pzeta - pxi*peta - peta*pzeta - pzeta*pxi + pxi*peta*pzeta;
	}
}

//---------------------------------------------------------------------
//
//   set the initial values of independent variables based on tri-linear
//   interpolation of boundary values in the computational space.
//
//---------------------------------------------------------------------
void LU::setiv() {
	dim3 grid(nz-2, ny-2);
	setiv_kernel<<<grid,nx-2>>>(u, nx, ny, nz);
}

__global__ static void erhs_kernel_init (double *frct, double *rsd, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta;

	k = blockIdx.x;
	j = blockIdx.y;
	i = threadIdx.x;

	using namespace gpu_mod;

	for (m = 0; m < 5; m++) frct(m,i,j,k) = 0.0;

	zeta = (double)k/((double)(nz-1));
	eta = (double)j/((double)(ny-1));
	xi = (double)i/((double)(nx-1));
	for (m = 0; m < 5; m++) rsd(m,i,j,k) = ce[m+0*5] + (ce[m+1*5]+(ce[m+4*5]+(ce[m+7*5]+ce[m+10*5]*xi)*xi)*xi)*xi + (ce[m+2*5]+(ce[m+5*5]+(ce[m+8*5]+ce[m+11*5]*eta)*eta)*eta)*eta + (ce[m+3*5]+(ce[m+6*5]+(ce[m+9*5]+ce[m+12*5]*zeta)*zeta)*zeta)*zeta;
}

__global__ static void erhs_kernel_x (double *frct, const double *rsd, const int nx, const int ny, const int nz) {
	int i, j, k, m, nthreads;
	double q, u21;
	__shared__ double flux[RHSX_BLOCK][5];
	__shared__ double rtmp[RHSX_BLOCK*5];
	__shared__ double u21i[RHSX_BLOCK], u31i[RHSX_BLOCK], u41i[RHSX_BLOCK], u51i[RHSX_BLOCK];
	double utmp[5];

	k = blockIdx.x+1;
	j = blockIdx.y+1;
	i = threadIdx.x;

	using namespace gpu_mod;

	while (i < nx) {
		// load rsd using coalesced memory access 
		// first compute number of threads executing this region 
		nthreads = nx-(i-threadIdx.x);
		if (nthreads > blockDim.x) nthreads = blockDim.x;
		m = threadIdx.x;
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		m += nthreads;
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		m += nthreads;
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		m += nthreads;
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		m += nthreads;
		rtmp[m] = rsd(m%5, (i-threadIdx.x)+m/5, j, k);
		__syncthreads();

		//---------------------------------------------------------------------
		//   xi-direction flux differences
		//---------------------------------------------------------------------
		flux[threadIdx.x][0] = rtmp[threadIdx.x*5+1];
		u21 = rtmp[threadIdx.x*5+1]/rtmp[threadIdx.x*5+0];
		q = 0.5*(rtmp[threadIdx.x*5+1]*rtmp[threadIdx.x*5+1] + rtmp[threadIdx.x*5+2]*rtmp[threadIdx.x*5+2] + rtmp[threadIdx.x*5+3]*rtmp[threadIdx.x*5+3])/rtmp[threadIdx.x*5+0];
		flux[threadIdx.x][1] = rtmp[threadIdx.x*5+1]*u21 + c2*(rtmp[threadIdx.x*5+4]-q);
		flux[threadIdx.x][2] = rtmp[threadIdx.x*5+2]*u21;
		flux[threadIdx.x][3] = rtmp[threadIdx.x*5+3]*u21;
		flux[threadIdx.x][4] = (c1*rtmp[threadIdx.x*5+4] - c2*q)*u21;
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSX_BLOCK-1 && i < nx-1)
			for (m = 0; m < 5; m++) utmp[m] = frct(m,i,j,k) - tx2*(flux[threadIdx.x+1][m]-flux[threadIdx.x-1][m]);

		u21 = 1.0/rtmp[threadIdx.x*5+0];
		u21i[threadIdx.x] = u21*rtmp[threadIdx.x*5+1];
		u31i[threadIdx.x] = u21*rtmp[threadIdx.x*5+2];
		u41i[threadIdx.x] = u21*rtmp[threadIdx.x*5+3];
		u51i[threadIdx.x] = u21*rtmp[threadIdx.x*5+4];
		__syncthreads();

		if (threadIdx.x >= 1) {
			flux[threadIdx.x][1] = (4.0/3.0)*tx3*(u21i[threadIdx.x]-u21i[threadIdx.x-1]);
			flux[threadIdx.x][2] = tx3*(u31i[threadIdx.x]-u31i[threadIdx.x-1]);
			flux[threadIdx.x][3] = tx3*(u41i[threadIdx.x]-u41i[threadIdx.x-1]);
			flux[threadIdx.x][4] = 0.5*(1.0-c1*c5)*tx3*((u21i[threadIdx.x]*u21i[threadIdx.x]+u31i[threadIdx.x]*u31i[threadIdx.x]+u41i[threadIdx.x]*u41i[threadIdx.x]) - 
							(u21i[threadIdx.x-1]*u21i[threadIdx.x-1]+u31i[threadIdx.x-1]*u31i[threadIdx.x-1]+u41i[threadIdx.x-1]*u41i[threadIdx.x-1])) + 
							(1.0/6.0)*tx3*(u21i[threadIdx.x]*u21i[threadIdx.x]-u21i[threadIdx.x-1]*u21i[threadIdx.x-1]) + c1*c5*tx3*(u51i[threadIdx.x]-u51i[threadIdx.x-1]);
		}
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSX_BLOCK-1 && i < nx-1) {
			utmp[0] += dx1*tx1*(rtmp[threadIdx.x*5-5]-2.0*rtmp[threadIdx.x*5+0]+rtmp[threadIdx.x*5+5]);
			utmp[1] += tx3*c3*c4*(flux[threadIdx.x+1][1]-flux[threadIdx.x][1]) + dx2*tx1*(rtmp[threadIdx.x*5-4]-2.0*rtmp[threadIdx.x*5+1]+rtmp[threadIdx.x*5+6]);
			utmp[2] += tx3*c3*c4*(flux[threadIdx.x+1][2]-flux[threadIdx.x][2]) + dx3*tx1*(rtmp[threadIdx.x*5-3]-2.0*rtmp[threadIdx.x*5+2]+rtmp[threadIdx.x*5+7]);
			utmp[3] += tx3*c3*c4*(flux[threadIdx.x+1][3]-flux[threadIdx.x][3]) + dx4*tx1*(rtmp[threadIdx.x*5-2]-2.0*rtmp[threadIdx.x*5+3]+rtmp[threadIdx.x*5+8]);
			utmp[4] += tx3*c3*c4*(flux[threadIdx.x+1][4]-flux[threadIdx.x][4]) + dx5*tx1*(rtmp[threadIdx.x*5-1]-2.0*rtmp[threadIdx.x*5+4]+rtmp[threadIdx.x*5+9]);
		
			//---------------------------------------------------------------------
			//   Fourth-order dissipation
			//---------------------------------------------------------------------
			if (i == 1) for (m = 0; m < 5; m++) frct(m,1,j,k) = utmp[m] - dssp*(+5.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]+rsd(m,3,j,k));
			if (i == 2) for (m = 0; m < 5; m++) frct(m,2,j,k) = utmp[m] - dssp*(-4.0*rtmp[threadIdx.x*5+m-5]+6.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]+rsd(m,4,j,k));
			if (i >= 3 && i < nx-3) for (m = 0; m < 5; m++) frct(m,i,j,k) = utmp[m] - dssp*(rsd(m,i-2,j,k)-4.0*rtmp[threadIdx.x*5+m-5]+6.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]+rsd(m,i+2,j,k));
			if (i == nx-3) for (m = 0; m < 5; m++) frct(m,nx-3,j,k) = utmp[m] - dssp*(rsd(m,nx-5,j,k)-4.0*rtmp[threadIdx.x*5+m-5]+6.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]);
			if (i == nx-2) for (m = 0; m < 5; m++) frct(m,nx-2,j,k) = utmp[m] - dssp*(rsd(m,nx-4,j,k)-4.0*rtmp[threadIdx.x*5+m-5]+5.0*rtmp[threadIdx.x*5+m]);
		}

		i += RHSX_BLOCK-2;
	}
}

__global__ static void erhs_kernel_y (double *frct, const double *rsd, const int nx, const int ny, const int nz) {
	int i, j, k, m, nthreads;
	double q, u31;
	__shared__ double flux[RHSY_BLOCK][5];
	__shared__ double rtmp[RHSY_BLOCK*5];
	__shared__ double u21j[RHSY_BLOCK], u31j[RHSY_BLOCK], u41j[RHSY_BLOCK], u51j[RHSY_BLOCK];
	double utmp[5];;

	k = blockIdx.x+1;
	i = blockIdx.y+1;
	j = threadIdx.x;

	using namespace gpu_mod;

	while (j < ny) {
		// load u, rsd and rho_i using coalesced memory access along the m-axis
		// first compute number of threads executing this region 
		nthreads = ny-(j-threadIdx.x);
		if (nthreads > blockDim.x) nthreads = blockDim.x;
		m = threadIdx.x;
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		m += nthreads;
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		m += nthreads;
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		m += nthreads;
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		m += nthreads;
		rtmp[m] = rsd(m%5, i, (j-threadIdx.x)+m/5, k);
		__syncthreads();

		//---------------------------------------------------------------------
		//   eta-direction flux differences
		//---------------------------------------------------------------------
		flux[threadIdx.x][0] = rtmp[threadIdx.x*5+2];
		u31 = rtmp[threadIdx.x*5+2]/rtmp[threadIdx.x*5+0];
		q = 0.5*(rtmp[threadIdx.x*5+1]*rtmp[threadIdx.x*5+1] + rtmp[threadIdx.x*5+2]*rtmp[threadIdx.x*5+2] + rtmp[threadIdx.x*5+3]*rtmp[threadIdx.x*5+3])/rtmp[threadIdx.x*5+0];
		flux[threadIdx.x][1] = rtmp[threadIdx.x*5+1]*u31;
		flux[threadIdx.x][2] = rtmp[threadIdx.x*5+2]*u31 + c2*(rtmp[threadIdx.x*5+4]-q);
		flux[threadIdx.x][3] = rtmp[threadIdx.x*5+3]*u31;
		flux[threadIdx.x][4] = (c1*rtmp[threadIdx.x*5+4]-c2*q)*u31;
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSY_BLOCK-1 && j < ny-1) 
			for (m = 0; m < 5; m++) utmp[m] = frct(m,i,j,k) - ty2*(flux[threadIdx.x+1][m]-flux[threadIdx.x-1][m]);
		u31 = 1.0/rtmp[threadIdx.x*5+0];
		u21j[threadIdx.x] = u31*rtmp[threadIdx.x*5+1];
		u31j[threadIdx.x] = u31*rtmp[threadIdx.x*5+2];
		u41j[threadIdx.x] = u31*rtmp[threadIdx.x*5+3];
		u51j[threadIdx.x] = u31*rtmp[threadIdx.x*5+4];

		__syncthreads();

		if (threadIdx.x >= 1) {
			flux[threadIdx.x][1] = ty3*(u21j[threadIdx.x]-u21j[threadIdx.x-1]);
			flux[threadIdx.x][2] = (4.0/3.0)*ty3*(u31j[threadIdx.x]-u31j[threadIdx.x-1]);
			flux[threadIdx.x][3] = ty3*(u41j[threadIdx.x]-u41j[threadIdx.x-1]);
			flux[threadIdx.x][4] = 0.5*(1.0-c1*c5)*ty3*((u21j[threadIdx.x]*u21j[threadIdx.x]+u31j[threadIdx.x]*u31j[threadIdx.x]+u41j[threadIdx.x]*u41j[threadIdx.x]) - 
							(u21j[threadIdx.x-1]*u21j[threadIdx.x-1]+u31j[threadIdx.x-1]*u31j[threadIdx.x-1]+u41j[threadIdx.x-1]*u41j[threadIdx.x-1])) + 
							(1.0/6.0)*ty3*(u31j[threadIdx.x]*u31j[threadIdx.x]-u31j[threadIdx.x-1]*u31j[threadIdx.x-1]) + c1*c5*ty3*(u51j[threadIdx.x]-u51j[threadIdx.x-1]);
		}
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSY_BLOCK-1 && j < ny-1) {
			utmp[0] += dy1*ty1*(rtmp[threadIdx.x*5-5]-2.0*rtmp[threadIdx.x*5+0]+rtmp[threadIdx.x*5+5]);
			utmp[1] += ty3*c3*c4*(flux[threadIdx.x+1][1]-flux[threadIdx.x][1]) + dy2*ty1*(rtmp[threadIdx.x*5-4]-2.0*rtmp[threadIdx.x*5+1]+rtmp[threadIdx.x*5+6]);
			utmp[2] += ty3*c3*c4*(flux[threadIdx.x+1][2]-flux[threadIdx.x][2]) + dy3*ty1*(rtmp[threadIdx.x*5-3]-2.0*rtmp[threadIdx.x*5+2]+rtmp[threadIdx.x*5+7]);
			utmp[3] += ty3*c3*c4*(flux[threadIdx.x+1][3]-flux[threadIdx.x][3]) + dy4*ty1*(rtmp[threadIdx.x*5-2]-2.0*rtmp[threadIdx.x*5+3]+rtmp[threadIdx.x*5+8]);
			utmp[4] += ty3*c3*c4*(flux[threadIdx.x+1][4]-flux[threadIdx.x][4]) + dy5*ty1*(rtmp[threadIdx.x*5-1]-2.0*rtmp[threadIdx.x*5+4]+rtmp[threadIdx.x*5+9]);

			//---------------------------------------------------------------------
			//   fourth-order dissipation
			//---------------------------------------------------------------------
			if (j == 1) for (m = 0; m < 5; m++) frct(m,i,1,k) = utmp[m] - dssp*(+5.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]+rsd(m,i,3,k));
			if (j == 2) for (m = 0; m < 5; m++) frct(m,i,2,k) = utmp[m] - dssp*(-4.0*rtmp[threadIdx.x*5+m-5]+6.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]+rsd(m,i,4,k));
			if (j >= 3 && j < ny-3) for (m = 0; m < 5; m++) frct(m,i,j,k) = utmp[m] - dssp*(rsd(m,i,j-2,k)-4.0*rtmp[threadIdx.x*5+m-5]+6.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]+rsd(m,i,j+2,k));
			if (j == ny-3) for (m = 0; m < 5; m++) frct(m,i,ny-3,k) = utmp[m] - dssp*(rsd(m,i,ny-5,k)-4.0*rtmp[threadIdx.x*5+m-5]+6.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]);
			if (j == ny-2) for (m = 0; m < 5; m++) frct(m,i,ny-2,k) = utmp[m] - dssp*(rsd(m,i,ny-4,k)-4.0*rtmp[threadIdx.x*5+m-5]+5.0*rtmp[threadIdx.x*5+m]);
		}

		j += RHSY_BLOCK-2;
	}
}

__global__ static void erhs_kernel_z (double *frct, const double *rsd, const int nx, const int ny, const int nz) {
	int i, j, k, m, nthreads;
	double q, u41;
	__shared__ double flux[RHSZ_BLOCK][5];
	__shared__ double rtmp[RHSZ_BLOCK*5];
	__shared__ double u21k[RHSZ_BLOCK], u31k[RHSZ_BLOCK], u41k[RHSZ_BLOCK], u51k[RHSZ_BLOCK];
	double utmp[5];

	j = blockIdx.x+1;
	i = blockIdx.y+1;
	k = threadIdx.x;
	
	using namespace gpu_mod;

	while (k < nz) {
		// load rsd using coalesced memory access along the m-axis
		// first compute number of threads executing this region 
		nthreads = (nz-(k-threadIdx.x));
		if (nthreads > blockDim.x) nthreads = blockDim.x;
		m = threadIdx.x;
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		m += nthreads;
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		m += nthreads;
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		m += nthreads;
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		m += nthreads;
		rtmp[m] = rsd(m%5, i, j, (k-threadIdx.x)+m/5);
		__syncthreads();

		//---------------------------------------------------------------------
		//   zeta-direction flux differences
		//---------------------------------------------------------------------
		flux[threadIdx.x][0] = rtmp[threadIdx.x*5+3];
		u41 = rtmp[threadIdx.x*5+3]/rtmp[threadIdx.x*5+0];
		q = 0.5*(rtmp[threadIdx.x*5+1]*rtmp[threadIdx.x*5+1]+rtmp[threadIdx.x*5+2]*rtmp[threadIdx.x*5+2]+rtmp[threadIdx.x*5+3]*rtmp[threadIdx.x*5+3])/rtmp[threadIdx.x*5+0];
		flux[threadIdx.x][1] = rtmp[threadIdx.x*5+1]*u41;
		flux[threadIdx.x][2] = rtmp[threadIdx.x*5+2]*u41;
		flux[threadIdx.x][3] = rtmp[threadIdx.x*5+3]*u41 + c2*(rtmp[threadIdx.x*5+4]-q);
		flux[threadIdx.x][4] = (c1*rtmp[threadIdx.x*5+4]-c2*q)*u41;
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSZ_BLOCK-1 && k < nz-1)
			for (m = 0; m < 5; m++) utmp[m] = frct(m,i,j,k) - tz2*(flux[threadIdx.x+1][m]-flux[threadIdx.x-1][m]);

		u41 = 1.0/rtmp[threadIdx.x*5+0];
		u21k[threadIdx.x] = u41*rtmp[threadIdx.x*5+1];
		u31k[threadIdx.x] = u41*rtmp[threadIdx.x*5+2];
		u41k[threadIdx.x] = u41*rtmp[threadIdx.x*5+3];
		u51k[threadIdx.x] = u41*rtmp[threadIdx.x*5+4];
		__syncthreads();

		if (threadIdx.x >= 1) {
			flux[threadIdx.x][1] = tz3*(u21k[threadIdx.x]-u21k[threadIdx.x-1]);
			flux[threadIdx.x][2] = tz3*(u31k[threadIdx.x]-u31k[threadIdx.x-1]);
			flux[threadIdx.x][3] = (4.0/3.0)*tz3*(u41k[threadIdx.x]-u41k[threadIdx.x-1]);
			flux[threadIdx.x][4] = 0.5*(1.0-c1*c5)*tz3*((u21k[threadIdx.x]*u21k[threadIdx.x]+u31k[threadIdx.x]*u31k[threadIdx.x]+u41k[threadIdx.x]*u41k[threadIdx.x]) - 
							(u21k[threadIdx.x-1]*u21k[threadIdx.x-1]+u31k[threadIdx.x-1]*u31k[threadIdx.x-1]+u41k[threadIdx.x-1]*u41k[threadIdx.x-1])) + 
							(1.0/6.0)*tz3*(u41k[threadIdx.x]*u41k[threadIdx.x]-u41k[threadIdx.x-1]*u41k[threadIdx.x-1]) + c1*c5*tz3*(u51k[threadIdx.x]-u51k[threadIdx.x-1]);
		}
		__syncthreads();

		if (threadIdx.x >= 1 && threadIdx.x < RHSZ_BLOCK-1 && k < nz-1) {
			utmp[0] += dz1*tz1*(rtmp[threadIdx.x*5-5]-2.0*rtmp[threadIdx.x*5+0]+rtmp[threadIdx.x*5+5]);
			utmp[1] += tz3*c3*c4*(flux[threadIdx.x+1][1]-flux[threadIdx.x][1]) + dz2*tz1*(rtmp[threadIdx.x*5-4]-2.0*rtmp[threadIdx.x*5+1]+rtmp[threadIdx.x*5+6]);
			utmp[2] += tz3*c3*c4*(flux[threadIdx.x+1][2]-flux[threadIdx.x][2]) + dz3*tz1*(rtmp[threadIdx.x*5-3]-2.0*rtmp[threadIdx.x*5+2]+rtmp[threadIdx.x*5+7]);
			utmp[3] += tz3*c3*c4*(flux[threadIdx.x+1][3]-flux[threadIdx.x][3]) + dz4*tz1*(rtmp[threadIdx.x*5-2]-2.0*rtmp[threadIdx.x*5+3]+rtmp[threadIdx.x*5+8]);
			utmp[4] += tz3*c3*c4*(flux[threadIdx.x+1][4]-flux[threadIdx.x][4]) + dz5*tz1*(rtmp[threadIdx.x*5-1]-2.0*rtmp[threadIdx.x*5+4]+rtmp[threadIdx.x*5+9]);

			//---------------------------------------------------------------------
			//   fourth-order dissipation
			//---------------------------------------------------------------------
			if (k == 1) for (m = 0; m < 5; m++) frct(m,i,j,1) = utmp[m] - dssp*(+5.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]+rsd(m,i,j,3));
			if (k == 2) for (m = 0; m < 5; m++) frct(m,i,j,2) = utmp[m] - dssp*(-4.0*rtmp[threadIdx.x*5+m-5]+6.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]+rsd(m,i,j,4));
			if (k >= 3 && k < nz-3) for (m = 0; m < 5; m++) frct(m,i,j,k) = utmp[m] - dssp*(rsd(m,i,j,k-2)-4.0*rtmp[threadIdx.x*5+m-5]+6.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]+rsd(m,i,j,k+2));
			if (k == nz-3) for (m = 0; m < 5; m++) frct(m,i,j,nz-3) = utmp[m] - dssp*(rsd(m,i,j,nz-5)-4.0*rtmp[threadIdx.x*5+m-5]+6.0*rtmp[threadIdx.x*5+m]-4.0*rtmp[threadIdx.x*5+m+5]);
			if (k == nz-2) for (m = 0; m < 5; m++) frct(m,i,j,nz-2) = utmp[m] - dssp*(rsd(m,i,j,nz-4)-4.0*rtmp[threadIdx.x*5+m-5]+5.0*rtmp[threadIdx.x*5+m]);
		}

		k += RHSZ_BLOCK-2;
	}
}

//---------------------------------------------------------------------
//
//   compute the right hand side based on exact solution
//
//---------------------------------------------------------------------
void LU::erhs() {
	dim3 grid_full(nz,ny);
	dim3 grid_x(nz-2,ny-2);
	dim3 grid_y(nz-2,nx-2);
	dim3 grid_z(ny-2,nx-2);
	
	erhs_kernel_init<<<grid_full,nx>>>(frct, rsd, nx, ny, nz);
	erhs_kernel_x<<<grid_x, min(nx,RHSX_BLOCK)>>>(frct, rsd, nx, ny, nz);
	erhs_kernel_y<<<grid_y, min(ny,RHSY_BLOCK)>>>(frct, rsd, nx, ny, nz);
	erhs_kernel_z<<<grid_z, min(nz,RHSZ_BLOCK)>>>(frct, rsd, nx, ny, nz);
}

//---------------------------------------------------------------------
//   set up coefficients
//---------------------------------------------------------------------
void LU::setcoeff() {
	double dxi = 1.0/((double)nx-1.0);
	double deta = 1.0/((double)nx-1.0);
	double dzeta = 1.0/((double)nz-1.0);

	double tx1 = 1.0/(dxi*dxi);
	double tx2 = 1.0/(2.0*dxi);
	double tx3 = 1.0/dxi;

	double ty1 = 1.0/(deta*deta);
	double ty2 = 1.0/(2.0*deta);
	double ty3 = 1.0/deta;

	double tz1 = 1.0/(dzeta*dzeta);
	double tz2 = 1.0/(2.0*dzeta);
	double tz3 = 1.0/dzeta;

	//---------------------------------------------------------------------
	//   coefficients of the exact solution to the first pde
	//---------------------------------------------------------------------
	double ce[5*13];
	ce[0+0*5] = 2.0;
	ce[0+1*5] = 0.0;
	ce[0+2*5] = 0.0;
	ce[0+3*5] = 4.0;
	ce[0+4*5] = 5.0;
	ce[0+5*5] = 3.0;
	ce[0+6*5] = 0.5;
	ce[0+7*5] = 0.02;
	ce[0+8*5] = 0.01;
	ce[0+9*5] = 0.03;
	ce[0+10*5] = 0.5;
	ce[0+11*5] = 0.4;
	ce[0+12*5] = 0.3;

	//---------------------------------------------------------------------
	//   coefficients of the exact solution to the second pde
	//---------------------------------------------------------------------
	ce[1+0*5] = 1.0;
	ce[1+1*5] = 0.0;
	ce[1+2*5] = 0.0;
	ce[1+3*5] = 0.0;
	ce[1+4*5] = 1.0;
	ce[1+5*5] = 2.0;
	ce[1+6*5] = 3.0;
	ce[1+7*5] = 0.01;
	ce[1+8*5] = 0.03;
	ce[1+9*5] = 0.02;
	ce[1+10*5] = 0.4;
	ce[1+11*5] = 0.3;
	ce[1+12*5] = 0.5;

	//---------------------------------------------------------------------
	//   coefficients of the exact solution to the third pde
	//---------------------------------------------------------------------
	ce[2+0*5] = 2.0;
	ce[2+1*5] = 2.0;
	ce[2+2*5] = 0.0;
	ce[2+3*5] = 0.0;
	ce[2+4*5] = 0.0;
	ce[2+5*5] = 2.0;
	ce[2+6*5] = 3.0;
	ce[2+7*5] = 0.04;
	ce[2+8*5] = 0.03;
	ce[2+9*5] = 0.05;
	ce[2+10*5] = 0.3;
	ce[2+11*5] = 0.5;
	ce[2+12*5] = 0.4;

	//---------------------------------------------------------------------
	//   coefficients of the exact solution to the fourth pde
	//---------------------------------------------------------------------
	ce[3+0*5] = 2.0;
	ce[3+1*5] = 2.0;
	ce[3+2*5] = 0.0;
	ce[3+3*5] = 0.0;
	ce[3+4*5] = 0.0;
	ce[3+5*5] = 2.0;
	ce[3+6*5] = 3.0;
	ce[3+7*5] = 0.03;
	ce[3+8*5] = 0.05;
	ce[3+9*5] = 0.04;
	ce[3+10*5] = 0.2;
	ce[3+11*5] = 0.1;
	ce[3+12*5] = 0.3;

	//---------------------------------------------------------------------
	//   coefficients of the exact solution to the fifth pde
	//---------------------------------------------------------------------
	ce[4+0*5] = 5.0;
	ce[4+1*5] = 4.0;
	ce[4+2*5] = 3.0;
	ce[4+3*5] = 2.0;
	ce[4+4*5] = 0.1;
	ce[4+5*5] = 0.4;
	ce[4+6*5] = 0.3;
	ce[4+7*5] = 0.05;
	ce[4+8*5] = 0.04;
	ce[4+9*5] = 0.03;
	ce[4+10*5] = 0.1;
	ce[4+11*5] = 0.3;
	ce[4+12*5] = 0.2;

	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dxi), &dxi, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::deta), &deta, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dzeta), &dzeta, sizeof(double)));

	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tx1), &tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tx2), &tx2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tx3), &tx3, sizeof(double)));

	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ty1), &ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ty2), &ty2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ty3), &ty3, sizeof(double)));

	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tz1), &tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tz2), &tz2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tz3), &tz3, sizeof(double)));

	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ce), &ce, 13*5*sizeof(double)));

	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dt), &dt, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::omega), &omega, sizeof(double)));
}

void LU::allocate_device_memory() {
	int gridsize = nx*ny*nz;
	int norm_buf_size = max(5*(ny-2)*(nz-2), ((nx-3)*(ny-3)+(nx-3)*(nz-3)+(ny-3)*(nz-3))/((PINTGR_BLOCK-1)*(PINTGR_BLOCK-1))+3);

	HANDLE_ERROR(hipMalloc((void **)&u, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rsd, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&frct, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rho_i, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&qs, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&dev_norm_buf, norm_buf_size*sizeof(double)));
}

void LU::free_device_memory() {
	HANDLE_ERROR(hipFree(u));
	HANDLE_ERROR(hipFree(rsd));
	HANDLE_ERROR(hipFree(frct));
	HANDLE_ERROR(hipFree(rho_i));
	HANDLE_ERROR(hipFree(qs));
	HANDLE_ERROR(hipFree(dev_norm_buf));
}

void LU::get_cuda_info() {
	int count;
	hipDeviceProp_t prop;

	HANDLE_ERROR(hipGetDeviceCount(&count));
	if (count == 0) {
		printf ("No CUDA devices found.\n");
		exit(EXIT_FAILURE);
	}

	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	strncpy (CUDAname, prop.name, 256);
	CUDAmp = prop.multiProcessorCount;
	CUDAclock = prop.clockRate;
	CUDAmem = prop.totalGlobalMem;
	CUDAmemclock = prop.memoryClockRate;
	CUDAl2cache = prop.l2CacheSize;
}
