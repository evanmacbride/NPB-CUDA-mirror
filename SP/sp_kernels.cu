#include "hip/hip_runtime.h"
#include <stdio.h>
#include "main.h"

namespace gpu_mod {
__constant__ double tx1, tx2, tx3, ty1, ty2, ty3, tz1, tz2, tz3;
__constant__ double bt, dt, dtdssp;
__constant__ double dnxm1, dnym1, dnzm1;
__constant__ double dttx1, dttx2, dtty1, dtty2, dttz1, dttz2, c2dttx1, c2dtty1, c2dttz1;
__constant__ double comz1, comz4, comz5, comz6, c3c4tx3, c3c4ty3, c3c4tz3;
__constant__ double xxcon1, xxcon2, xxcon3, xxcon4, xxcon5, dx1tx1, dx2tx1, dx3tx1, dx4tx1, dx5tx1;
__constant__ double yycon1, yycon2, yycon3, yycon4, yycon5, dy1ty1, dy2ty1, dy3ty1, dy4ty1, dy5ty1;
__constant__ double zzcon1, zzcon2, zzcon3, zzcon4, zzcon5, dz1tz1, dz2tz1, dz3tz1, dz4tz1, dz5tz1;
__constant__ double ce[13][5];
}

static void inline HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define START_TIMER(timer) if (timeron) { HANDLE_ERROR(hipDeviceSynchronize()); timers->timer_start(timer); }
#define STOP_TIMER(timer) if (timeron) { HANDLE_ERROR(hipDeviceSynchronize()); timers->timer_stop(timer); }

void SP::adi (bool singlestep) {

	for (int i = 0; i < t_last; i++) timers->timer_clear(i);
	HANDLE_ERROR(hipDeviceSynchronize());
	timers->timer_start(0);

	int itmax = singlestep ? 1 : niter;
	for (int step = 1; step <= itmax; step++) {
		if (step % 20 == 0 || step == 1 && !singlestep)
			printf(" Time step %4d\n", step);

		compute_rhs();
		txinvr();
		x_solve();
		y_solve();
		z_solve();
		add();
	}

	HANDLE_ERROR(hipDeviceSynchronize());
	timers->timer_stop(0);
	tmax = timers->timer_read(0);
}

//---------------------------------------------------------------------
// 	addition of update to the vector u
//---------------------------------------------------------------------
__global__ static void add_kernel (double *u, const double *rhs, const int nx, const int ny, const int nz) {
	int i, j, k, m;

	k = blockIdx.y+1;
	j = blockIdx.x+1;
	i = threadIdx.x+1;
	m = threadIdx.y;

	u(m,i,j,k) += rhs(m,i,j,k);
}

void SP::add () {
	dim3 grid(ny-2,nz-2);
	dim3 block(nx-2,5);

	START_TIMER(t_add);
	add_kernel<<<grid,block>>>(u, rhs, nx, ny, nz);
	STOP_TIMER(t_add);
}

__global__ static void compute_rhs_kernel_1 (double *rho_i, double *us, double *vs, double *ws, double *speed, double *qs, double *square, const double *u, const int nx, const int ny, const int nz) {
	int i, j, k;
	k = blockIdx.y;
	j = blockIdx.x;
	i = threadIdx.x;
	//---------------------------------------------------------------------
	//      compute the reciprocal of density, and the kinetic energy, 
	//      and the speed of sound. 
	//---------------------------------------------------------------------
	double rho_inv = 1.0/u(0,i,j,k);
	double square_ijk;
	rho_i(i,j,k) = rho_inv;
	us(i,j,k) = u(1,i,j,k) * rho_inv;
	vs(i,j,k) = u(2,i,j,k) * rho_inv;
	ws(i,j,k) = u(3,i,j,k) * rho_inv;
	square(i,j,k) = square_ijk = 0.5*(u(1,i,j,k)*u(1,i,j,k) + u(2,i,j,k)*u(2,i,j,k) + u(3,i,j,k)*u(3,i,j,k)) * rho_inv;
	qs(i,j,k) = square_ijk * rho_inv;
	//---------------------------------------------------------------------
	//               (don't need speed and ainx until the lhs computation)
	//---------------------------------------------------------------------
	speed(i,j,k) = sqrt(c1c2*rho_inv*(u(4,i,j,k) - square_ijk));
}

__global__ static void compute_rhs_kernel_2 (const double *rho_i, const double *us, const double *vs, const double *ws, const double *qs, const double *square, double *rhs, const double *forcing, const double *u, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	k = blockIdx.y;
	j = blockIdx.x;
	i = threadIdx.x;
	double rtmp[5];

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	// copy the exact forcing term to the right hand side;  because 
	// this forcing term is known, we can store it on the whole zone
	// including the boundary                   
	//---------------------------------------------------------------------
	for (m = 0; m < 5; m++) rtmp[m] = forcing(m,i,j,k);
	
	//---------------------------------------------------------------------
	//      compute xi-direction fluxes 
	//---------------------------------------------------------------------
	if (k >= 1 && k < nz-1 && j >= 1 && j < ny-1 && i >= 1 && i < nx-1) {
		double uijk = us(i,j,k);
		double up1 = us(i+1,j,k);
		double um1 = us(i-1,j,k);
				
		rtmp[0] = rtmp[0] + dx1tx1*(u(0,i+1,j,k) - 2.0*u(0,i,j,k) + u(0,i-1,j,k)) - tx2*(u(1,i+1,j,k)-u(1,i-1,j,k));
		rtmp[1] = rtmp[1] + dx2tx1*(u(1,i+1,j,k) - 2.0*u(1,i,j,k) + u(1,i-1,j,k)) + xxcon2*con43*(up1-2.0*uijk+um1) - tx2*(u(1,i+1,j,k)*up1 - u(1,i-1,j,k)*um1 + (u(4,i+1,j,k)-square(i+1,j,k)-u(4,i-1,j,k)+square(i-1,j,k))*c2);
		rtmp[2] = rtmp[2] + dx3tx1*(u(2,i+1,j,k) - 2.0*u(2,i,j,k) + u(2,i-1,j,k)) + xxcon2*(vs(i+1,j,k)-2.0*vs(i,j,k)+vs(i-1,j,k)) - tx2*(u(2,i+1,j,k)*up1 - u(2,i-1,j,k)*um1);
		rtmp[3] = rtmp[3] + dx4tx1*(u(3,i+1,j,k) - 2.0*u(3,i,j,k) + u(3,i-1,j,k)) + xxcon2*(ws(i+1,j,k)-2.0*ws(i,j,k)+ws(i-1,j,k)) - tx2*(u(3,i+1,j,k)*up1 - u(3,i-1,j,k)*um1);
		rtmp[4] = rtmp[4] + dx5tx1*(u(4,i+1,j,k) - 2.0*u(4,i,j,k) + u(4,i-1,j,k)) + xxcon3*(qs(i+1,j,k)-2.0*qs(i,j,k)+qs(i-1,j,k))+ xxcon4*(up1*up1-2.0*uijk*uijk+um1*um1) +
				xxcon5*(u(4,i+1,j,k)*rho_i(i+1,j,k) - 2.0*u(4,i,j,k)*rho_i(i,j,k) + u(4,i-1,j,k)*rho_i(i-1,j,k)) - tx2*((c1*u(4,i+1,j,k) - c2*square(i+1,j,k))*up1 - (c1*u(4,i-1,j,k) - c2*square(i-1,j,k))*um1 );
		//---------------------------------------------------------------------
		//      add fourth order xi-direction dissipation               
		//---------------------------------------------------------------------
		if (i == 1) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (5.0*u(m,i,j,k)-4.0*u(m,i+1,j,k)+u(m,i+2,j,k));
		} else if (i == 2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (-4.0*u(m,i-1,j,k)+6.0*u(m,i,j,k)-4.0*u(m,i+1,j,k)+u(m,i+2,j,k));
		} else if (i >= 3 && i < nx-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * ( u(m,i-2,j,k)-4.0*u(m,i-1,j,k)+6.0*u(m,i,j,k)-4.0*u(m,i+1,j,k)+u(m,i+2,j,k));
		} else if (i == nx-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (u(m,i-2,j,k)-4.0*u(m,i-1,j,k)+6.0*u(m,i,j,k)-4.0*u(m,i+1,j,k) );
		} else if (i == nx-2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp * (u(m,i-2,j,k)-4.0*u(m,i-1,j,k) + 5.0*u(m,i,j,k));
		}
		//---------------------------------------------------------------------
		//      compute eta-direction fluxes 
		//---------------------------------------------------------------------
		double vijk = vs(i,j,k);
		double vp1 = vs(i,j+1,k);
		double vm1 = vs(i,j-1,k);
		rtmp[0] = rtmp[0] + dy1ty1*(u(0,i,j+1,k) - 2.0*u(0,i,j,k) + u(0,i,j-1,k)) - ty2*(u(2,i,j+1,k)-u(2,i,j-1,k));
		rtmp[1] = rtmp[1] + dy2ty1*(u(1,i,j+1,k) - 2.0*u(1,i,j,k) + u(1,i,j-1,k)) + yycon2*(us(i,j+1,k)-2.0*us(i,j,k)+us(i,j-1,k)) - ty2*(u(1,i,j+1,k)*vp1-u(1,i,j-1,k)*vm1);
		rtmp[2] = rtmp[2] + dy3ty1*(u(2,i,j+1,k) - 2.0*u(2,i,j,k) + u(2,i,j-1,k)) + yycon2*con43*(vp1-2.0*vijk+vm1) - ty2*(u(2,i,j+1,k)*vp1-u(2,i,j-1,k)*vm1+(u(4,i,j+1,k)-square(i,j+1,k)-u(4,i,j-1,k)+square(i,j-1,k))*c2);
		rtmp[3] = rtmp[3] + dy4ty1*(u(3,i,j+1,k) - 2.0*u(3,i,j,k) + u(3,i,j-1,k)) + yycon2*(ws(i,j+1,k)-2.0*ws(i,j,k)+ws(i,j-1,k))-ty2*(u(3,i,j+1,k)*vp1-u(3,i,j-1,k)*vm1);
		rtmp[4] = rtmp[4] + dy5ty1*(u(4,i,j+1,k) - 2.0*u(4,i,j,k) + u(4,i,j-1,k)) + yycon3*(qs(i,j+1,k)-2.0*qs(i,j,k)+qs(i,j-1,k)) + yycon4*(vp1*vp1-2.0*vijk*vijk+vm1*vm1) +
				yycon5*(u(4,i,j+1,k)*rho_i(i,j+1,k)-2.0*u(4,i,j,k)*rho_i(i,j,k)+u(4,i,j-1,k)*rho_i(i,j-1,k)) - ty2*((c1*u(4,i,j+1,k)-c2*square(i,j+1,k))*vp1 - (c1*u(4,i,j-1,k)-c2*square(i,j-1,k))*vm1);
		//---------------------------------------------------------------------
		//      add fourth order eta-direction dissipation         
		//---------------------------------------------------------------------
		if (j == 1) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(5.0*u(m,i,j,k)-4.0*u(m,i,j+1,k)+u(m,i,j+2,k));
		} else if (j == 2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(-4.0*u(m,i,j-1,k)+6.0*u(m,i,j,k)-4.0*u(m,i,j+1,k)+u(m,i,j+2,k));
		} else if (j >= 3 && j < ny-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j-2,k)-4.0*u(m,i,j-1,k)+6.0*u(m,i,j,k)-4.0*u(m,i,j+1,k)+u(m,i,j+2,k));
		} else if (j == ny-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j-2,k)-4.0*u(m,i,j-1,k)+6.0*u(m,i,j,k)-4.0*u(m,i,j+1,k));
		} else if (j == ny-2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j-2,k)-4.0*u(m,i,j-1,k)+5.0*u(m,i,j,k));
		}
		//---------------------------------------------------------------------
		//      compute zeta-direction fluxes 
		//---------------------------------------------------------------------
		double wijk = ws(i,j,k);
		double wp1 = ws(i,j,k+1);
		double wm1 = ws(i,j,k-1);

		rtmp[0] = rtmp[0] + dz1tz1*(u(0,i,j,k+1)-2.0*u(0,i,j,k)+u(0,i,j,k-1)) - tz2*(u(3,i,j,k+1)-u(3,i,j,k-1));
		rtmp[1] = rtmp[1] + dz2tz1*(u(1,i,j,k+1)-2.0*u(1,i,j,k)+u(1,i,j,k-1)) + zzcon2*(us(i,j,k+1)-2.0*us(i,j,k)+us(i,j,k-1)) - tz2*(u(1,i,j,k+1)*wp1-u(1,i,j,k-1)*wm1);
		rtmp[2] = rtmp[2] + dz3tz1*(u(2,i,j,k+1)-2.0*u(2,i,j,k)+u(2,i,j,k-1)) + zzcon2*(vs(i,j,k+1)-2.0*vs(i,j,k)+vs(i,j,k-1)) - tz2*(u(2,i,j,k+1)*wp1-u(2,i,j,k-1)*wm1);
		rtmp[3] = rtmp[3] + dz4tz1*(u(3,i,j,k+1)-2.0*u(3,i,j,k)+u(3,i,j,k-1)) + zzcon2*con43*(wp1-2.0*wijk+wm1) - tz2*(u(3,i,j,k+1)*wp1-u(3,i,j,k-1)*wm1+(u(4,i,j,k+1)-square(i,j,k+1)-u(4,i,j,k-1)+square(i,j,k-1))*c2);
		rtmp[4] = rtmp[4] + dz5tz1*(u(4,i,j,k+1)-2.0*u(4,i,j,k)+u(4,i,j,k-1)) + zzcon3*(qs(i,j,k+1)-2.0*qs(i,j,k)+qs(i,j,k-1)) + zzcon4*(wp1*wp1-2.0*wijk*wijk+wm1*wm1) +
			zzcon5*(u(4,i,j,k+1)*rho_i(i,j,k+1)-2.0*u(4,i,j,k)*rho_i(i,j,k)+u(4,i,j,k-1)*rho_i(i,j,k-1)) - tz2*((c1*u(4,i,j,k+1)-c2*square(i,j,k+1))*wp1-(c1*u(4,i,j,k-1)-c2*square(i,j,k-1))*wm1);
		//---------------------------------------------------------------------
		//      add fourth order zeta-direction dissipation                
		//---------------------------------------------------------------------
		if (k == 1) {
			for (m = 0; m < 5; m++)	rtmp[m] = rtmp[m] - dssp*(5.0*u(m,i,j,k)-4.0*u(m,i,j,k+1)+u(m,i,j,k+2));
		} else if (k == 2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(-4.0*u(m,i,j,k-1)+6.0*u(m,i,j,k)-4.0*u(m,i,j,k+1)+u(m,i,j,k+2));
		} else if (k >= 3 && k < nz-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j,k-2)-4.0*u(m,i,j,k-1)+6.0*u(m,i,j,k)-4.0*u(m,i,j,k+1)+u(m,i,j,k+2));
		} else if (k == nz-3) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j,k-2)-4.0*u(m,i,j,k-1)+6.0*u(m,i,j,k)-4.0*u(m,i,j,k+1));
		} else if (k == nz-2) {
			for (m = 0; m < 5; m++) rtmp[m] = rtmp[m] - dssp*(u(m,i,j,k-2)-4.0*u(m,i,j,k-1)+5.0*u(m,i,j,k));
		}

		for (m = 0; m < 5; m++) rtmp[m] *= dt;
	}

	for (m = 0; m < 5; m++) rhs(m,i,j,k) = rtmp[m];
}

void SP::compute_rhs () {
	dim3 grid1(ny,nz);

	START_TIMER(t_rhs);
	compute_rhs_kernel_1<<<grid1,nx>>>(rho_i, us, vs, ws, speed, qs, square, u, nx, ny, nz);

	START_TIMER(t_rhsx);
	compute_rhs_kernel_2<<<grid1,nx>>>(rho_i, us, vs, ws, qs, square, rhs, forcing, u, nx, ny, nz);
	STOP_TIMER(t_rhsx);

	STOP_TIMER(t_rhs);
}

__global__ static void txinvr_kernel (const double *rho_i, const double *us, const double *vs, const double *ws, const double *speed, const double *qs, double *rhs, const int nx, const int ny, const int nz) {
	int i, j, k;

	k = blockIdx.y+1;
	j = blockIdx.x+1;
	i = threadIdx.x+1;

	using namespace gpu_mod;

	double ru1 = rho_i(i,j,k);
	double uu = us(i,j,k);
	double vv = vs(i,j,k);
	double ww = ws(i,j,k);
	double ac = speed(i,j,k);
	double ac2inv = 1.0/( ac*ac );

	double r1 = rhs(0,i,j,k);
	double r2 = rhs(1,i,j,k);
	double r3 = rhs(2,i,j,k);
	double r4 = rhs(3,i,j,k);
	double r5 = rhs(4,i,j,k);

	double t1 = c2*ac2inv*(qs(i,j,k)*r1 - uu*r2  - vv*r3 - ww*r4 + r5);
	double t2 = bt * ru1 * ( uu * r1 - r2 );
	double t3 = ( bt * ru1 * ac ) * t1;

	rhs(0,i,j,k) = r1 - t1;
	rhs(1,i,j,k) = -ru1*(ww*r1-r4);
	rhs(2,i,j,k) = ru1*(vv*r1-r3);
	rhs(3,i,j,k) = -t2+t3;
	rhs(4,i,j,k) = t2+t3;
}

void SP::txinvr () {
	dim3 grid (ny-2,nz-2);

	START_TIMER(t_txinvr);
	txinvr_kernel<<<grid,nx-2>>> (rho_i, us, vs, ws, speed, qs, rhs, nx, ny, nz);
	STOP_TIMER(t_txinvr);
}

//---------------------------------------------------------------------
// Computes the left hand side for the three x-factors  
//---------------------------------------------------------------------
#define lhs(m,i,j,k) lhs[(j-1)+(ny-2)*((k-1)+(nz-2)*((i)+nx*(m-3)))]
#define lhsp(m,i,j,k) lhs[(j-1)+(ny-2)*((k-1)+(nz-2)*((i)+nx*(m+4)))]
#define lhsm(m,i,j,k) lhs[(j-1)+(ny-2)*((k-1)+(nz-2)*((i)+nx*(m-3+2)))]
#define rtmp(m,i,j,k) rhstmp[(j)+ny*((k)+nz*((i)+nx*(m)))]
__global__ static void x_solve_kernel (const double *rho_i, const double *us, const double *speed, double *rhs, double *lhs, double *rhstmp, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double rhon[3], cv[3], _lhs[3][5], _lhsp[3][5], _rhs[3][5], fac1;

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	j = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (k >= nz-1 || j >= ny-1) return;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	// Computes the left hand side for the three x-factors  
	//---------------------------------------------------------------------
	//---------------------------------------------------------------------
	//     zap the whole left hand side for starters
	//---------------------------------------------------------------------
	_lhs[0][0] = lhsp(0,0,j,k) = 0.0;
	_lhs[0][1] = lhsp(1,0,j,k) = 0.0;
	_lhs[0][2] = lhsp(2,0,j,k) = 1.0;
	_lhs[0][3] = lhsp(3,0,j,k) = 0.0;
	_lhs[0][4] = lhsp(4,0,j,k) = 0.0;

	//---------------------------------------------------------------------
	// first fill the lhs for the u-eigenvalue                          
	//---------------------------------------------------------------------
	for (i = 0; i < 3; i++) {
		fac1 = c3c4*rho_i(i,j,k);
		rhon[i] = max(max(max(dx2+con43*fac1, dx5+c1c5*fac1), dxmax+fac1), dx1);
		cv[i] = us(i,j,k);
	}
	_lhs[1][0] = 0.0;
	_lhs[1][1] = - dttx2 * cv[0] - dttx1 * rhon[0];
	_lhs[1][2] = 1.0 + c2dttx1 * rhon[1];
	_lhs[1][3] = dttx2 * cv[2] - dttx1 * rhon[2];
	_lhs[1][4] = 0.0;
	_lhs[1][2] += comz5;
	_lhs[1][3] -= comz4;
	_lhs[1][4] += comz1;
	for (m = 0; m < 5; m++) lhsp(m,1,j,k) = _lhs[1][m];
	rhon[0] = rhon[1]; rhon[1] = rhon[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	for (m = 0; m < 3; m++) {
		_rhs[0][m] = rhs(m,0,j,k);
		_rhs[1][m] = rhs(m,1,j,k);
	}

	//---------------------------------------------------------------------
	//      perform the Thomas algorithm; first, FORWARD ELIMINATION     
	//---------------------------------------------------------------------
	for (i = 0; i < nx-2; i++) {
		//---------------------------------------------------------------------
		// first fill the lhs for the u-eigenvalue                          
		//---------------------------------------------------------------------
		if (i+2 == nx-1) {
			_lhs[2][0] = lhsp(0,i+2,j,k) = 0.0;
			_lhs[2][1] = lhsp(1,i+2,j,k) = 0.0;
			_lhs[2][2] = lhsp(2,i+2,j,k) = 1.0;
			_lhs[2][3] = lhsp(3,i+2,j,k) = 0.0;
			_lhs[2][4] = lhsp(4,i+2,j,k) = 0.0;
		} else {
			fac1 = c3c4*rho_i(i+3,j,k);
			rhon[2] = max(max(max(dx2+con43*fac1, dx5+c1c5*fac1), dxmax+fac1), dx1);
			cv[2] = us(i+3,j,k);
			_lhs[2][0] = 0.0;
			_lhs[2][1] = - dttx2 * cv[0] - dttx1 * rhon[0];
			_lhs[2][2] = 1.0 + c2dttx1 * rhon[1];
			_lhs[2][3] = dttx2 * cv[2] - dttx1 * rhon[2];
			_lhs[2][4] = 0.0;
			//---------------------------------------------------------------------
			//      add fourth order dissipation                                  
			//---------------------------------------------------------------------
			if (i+2 == 2) {
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (i+2 >= 3 && i+2 < nx-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (i+2 == nx-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
			} else if (i+2 == nx-2) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz5;
			}

			//---------------------------------------------------------------------
			//      store computed lhs for later reuse
			//---------------------------------------------------------------------
			for (m = 0; m < 5; m++) lhsp(m,i+2,j,k) = _lhs[2][m];
			rhon[0] = rhon[1]; rhon[1] = rhon[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		//---------------------------------------------------------------------
		//      load rhs values for current iteration
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[2][m] = rhs(m,i+2,j,k);

		//---------------------------------------------------------------------
		//      perform current iteration
		//---------------------------------------------------------------------
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		lhs(3,i,j,k) = _lhs[0][3];
		lhs(4,i,j,k) = _lhs[0][4];
		for (m = 0; m < 5; m++) {
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
		for (m = 0; m < 3; m++) {
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
	}

	//---------------------------------------------------------------------
	//      The last two rows in this zone are a bit different, 
	//      since they do not have two more rows available for the
	//      elimination of off-diagonal entries
	//---------------------------------------------------------------------
	i = nx-2;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//            scale the last row immediately 
	//---------------------------------------------------------------------
	fac1 = 1.0/_lhs[1][2];
	for (m = 0; m < 3; m++) _rhs[1][m] *= fac1;
	lhs(3,nx-2,j,k) = _lhs[0][3];
	lhs(4,nx-2,j,k) = _lhs[0][4];

	//---------------------------------------------------------------------
	//      subsequently, fill the other factors (u+c), (u-c) 
	//---------------------------------------------------------------------
	for (i = 0; i < 3; i++) cv[i] = speed(i,j,k);
	for (m = 0; m < 5; m++) {
		_lhsp[0][m] = _lhs[0][m] = lhsp(m,0,j,k);
		_lhsp[1][m] = _lhs[1][m] = lhsp(m,1,j,k);
	}
	_lhsp[1][1] -= dttx2 * cv[0];
	_lhsp[1][3] += dttx2 * cv[2];
	_lhs[1][1] += dttx2 * cv[0];
	_lhs[1][3] -= dttx2 * cv[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	_rhs[0][3] = rhs(3,0,j,k);
	_rhs[0][4] = rhs(4,0,j,k);
	_rhs[1][3] = rhs(3,1,j,k);
	_rhs[1][4] = rhs(4,1,j,k);
	//---------------------------------------------------------------------
	//      do the u+c and the u-c factors               
	//---------------------------------------------------------------------
	for (i = 0; i < nx-2; i++) {
		//---------------------------------------------------------------------
		//      first, fill the other factors (u+c), (u-c) 
		//---------------------------------------------------------------------
		for (m = 0; m < 5; m++) {
			_lhsp[2][m] = _lhs[2][m] = lhsp(m,i+2,j,k);
		}
		_rhs[2][3] = rhs(3,i+2,j,k);
		_rhs[2][4] = rhs(4,i+2,j,k);

		if (i+2 < nx-1) {
			cv[2] = speed(i+3,j,k);
			_lhsp[2][1] -= dttx2 * cv[0];
			_lhsp[2][3] += dttx2 * cv[2];
			_lhs[2][1] += dttx2 * cv[0];
			_lhs[2][3] -= dttx2 * cv[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		m = 3;
		fac1 = 1.0/_lhsp[0][2];
		_lhsp[0][3] *= fac1;
		_lhsp[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhsp[1][2] -= _lhsp[1][1]*_lhsp[0][3];
		_lhsp[1][3] -= _lhsp[1][1]*_lhsp[0][4];
		_rhs[1][m] -= _lhsp[1][1]*_rhs[0][m];
		_lhsp[2][1] -= _lhsp[2][0]*_lhsp[0][3];
		_lhsp[2][2] -= _lhsp[2][0]*_lhsp[0][4];
		_rhs[2][m] -= _lhsp[2][0]*_rhs[0][m];

		m = 4;
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1]*_lhs[0][3];
		_lhs[1][3] -= _lhs[1][1]*_lhs[0][4];
		_rhs[1][m] -= _lhs[1][1]*_rhs[0][m];
		_lhs[2][1] -= _lhs[2][0]*_lhs[0][3];
		_lhs[2][2] -= _lhs[2][0]*_lhs[0][4];
		_rhs[2][m] -= _lhs[2][0]*_rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		for (m = 3; m < 5; m++) {
			lhsp(m,i,j,k) = _lhsp[0][m];
			lhsm(m,i,j,k) = _lhs[0][m];
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
		for (m = 0; m < 5; m++) {
			_lhsp[0][m] = _lhsp[1][m];
			_lhsp[1][m] = _lhsp[2][m];
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//         And again the last two rows separately
	//---------------------------------------------------------------------
	i = nx-2;
	m = 3;
	fac1 = 1.0/_lhsp[0][2];
	_lhsp[0][3] *= fac1;
	_lhsp[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhsp[1][2] -= _lhsp[1][1]*_lhsp[0][3];
	_lhsp[1][3] -= _lhsp[1][1]*_lhsp[0][4];
	_rhs[1][m] -= _lhsp[1][1]*_rhs[0][m];

	m = 4;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1]*_lhs[0][3];
	_lhs[1][3] -= _lhs[1][1]*_lhs[0][4];
	_rhs[1][m] -= _lhs[1][1]*_rhs[0][m];

	//---------------------------------------------------------------------
	//               Scale the last row immediately
	//---------------------------------------------------------------------
	_rhs[1][3] /= _lhsp[1][2];
	_rhs[1][4] /= _lhs[1][2];

	//---------------------------------------------------------------------
	//                         BACKSUBSTITUTION 
	//---------------------------------------------------------------------
	for (m = 0; m < 3; m++) _rhs[0][m] -= lhs(3,nx-2,j,k)*_rhs[1][m];
	_rhs[0][3] -= _lhsp[0][3]*_rhs[1][3];
	_rhs[0][4] -= _lhs[0][3]*_rhs[1][4];
	for (m = 0; m < 5; m++) {
		_rhs[2][m] = _rhs[1][m];
		_rhs[1][m] = _rhs[0][m];
	}

	for (i = nx-3; i >= 0; i--) {
		//---------------------------------------------------------------------
		//      The first three factors
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[0][m] = rtmp(m,i,j,k) - lhs(3,i,j,k)*_rhs[1][m] - lhs(4,i,j,k)*_rhs[2][m];
		//---------------------------------------------------------------------
		//      And the remaining two
		//---------------------------------------------------------------------
		_rhs[0][3] = rtmp(3,i,j,k) - lhsp(3,i,j,k)*_rhs[1][3] - lhsp(4,i,j,k)*_rhs[2][3];
		_rhs[0][4] = rtmp(4,i,j,k) - lhsm(3,i,j,k)*_rhs[1][4] - lhsm(4,i,j,k)*_rhs[2][4];

		if (i+2 < nx-1) {
			//---------------------------------------------------------------------
			//      Do the block-diagonal inversion          
			//---------------------------------------------------------------------
				double r1 = _rhs[2][0];
				double r2 = _rhs[2][1];
				double r3 = _rhs[2][2];
				double r4 = _rhs[2][3];
				double r5 = _rhs[2][4];
				double t1 = bt * r3;
				double t2 = 0.5 * (r4+r5);

				_rhs[2][0] = -r2;
				_rhs[2][1] =  r1;
				_rhs[2][2] = bt * ( r4 - r5 );
				_rhs[2][3] = -t1 + t2;
				_rhs[2][4] =  t1 + t2;
		}

		for (m = 0; m < 5; m++) {
			rhs(m,i+2,j,k) = _rhs[2][m];
			_rhs[2][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[0][m];
		}
	}

	//---------------------------------------------------------------------
	//      Do the block-diagonal inversion          
	//---------------------------------------------------------------------
	double t1 = bt * _rhs[2][2];
	double t2 = 0.5 * (_rhs[2][3]+_rhs[2][4]);
	rhs(0,1,j,k) = -_rhs[2][1];
	rhs(1,1,j,k) =  _rhs[2][0];
	rhs(2,1,j,k) = bt * ( _rhs[2][3] - _rhs[2][4] );
	rhs(3,1,j,k) = -t1 + t2;
	rhs(4,1,j,k) =  t1 + t2;

	for (m = 0; m < 5; m++) rhs(m,0,j,k) = _rhs[1][m];
}
#undef lhs
#undef lhsp
#undef lhsm
#undef rtmp

void SP::x_solve () {
	int yblock = min(SOLVE_BLOCK,ny);
	int ygrid = (ny+yblock-1)/yblock;
	int zblock = min(SOLVE_BLOCK/yblock,nz);
	int zgrid = (nz+zblock-1)/zblock;
	dim3 grid(zgrid,ygrid), block(zblock,yblock);

	START_TIMER(t_xsolve);
	x_solve_kernel<<<grid,block>>>(rho_i, us, speed, rhs, lhs, rhstmp, nx, ny, nz);
	STOP_TIMER(t_xsolve);
}

//---------------------------------------------------------------------
// this function performs the solution of the approximate factorization
// step in the y-direction for all five matrix components
// simultaneously. The Thomas algorithm is employed to solve the
// systems for the y-lines. Boundary conditions are non-periodic
//---------------------------------------------------------------------
#define lhs(m,i,j,k) lhs[(i-1)+(nx-2)*((k-1)+(nz-2)*((j)+ny*(m-3)))]
#define lhsp(m,i,j,k) lhs[(i-1)+(nx-2)*((k-1)+(nz-2)*((j)+ny*(m+4)))]
#define lhsm(m,i,j,k) lhs[(i-1)+(nx-2)*((k-1)+(nz-2)*((j)+ny*(m-3+2)))]
#define rtmp(m,i,j,k) rhstmp[(i)+nx*((k)+nz*((j)+ny*(m)))]
__global__ static void y_solve_kernel (const double *rho_i, const double *vs, const double *speed, double *rhs, double *lhs, double *rhstmp, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double rhoq[3], cv[3], _lhs[3][5], _lhsp[3][5], _rhs[3][5], fac1;

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (k >= nz-1 || i >= nx-1) return;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	// Computes the left hand side for the three y-factors   
	//---------------------------------------------------------------------
	//---------------------------------------------------------------------
	//     zap the whole left hand side for starters
	//---------------------------------------------------------------------
	_lhs[0][0] = lhsp(0,i,0,k) = 0.0;
	_lhs[0][1] = lhsp(1,i,0,k) = 0.0;
	_lhs[0][2] = lhsp(2,i,0,k) = 1.0;
	_lhs[0][3] = lhsp(3,i,0,k) = 0.0;
	_lhs[0][4] = lhsp(4,i,0,k) = 0.0;

	//---------------------------------------------------------------------
	//      first fill the lhs for the u-eigenvalue         
	//---------------------------------------------------------------------
	for (j = 0; j < 3; j++) {
		fac1 = c3c4*rho_i(i,j,k);
		rhoq[j] = max(max(max(dy3+con43*fac1, dy5+c1c5*fac1), dymax+fac1), dy1);
		cv[j] = vs(i,j,k);
	}
	_lhs[1][0] =  0.0;
	_lhs[1][1] = -dtty2*cv[0]-dtty1 * rhoq[0];
	_lhs[1][2] =  1.0 + c2dtty1 * rhoq[1];
	_lhs[1][3] =  dtty2*cv[2]-dtty1 * rhoq[2];
	_lhs[1][4] =  0.0;
	_lhs[1][2] += comz5;
	_lhs[1][3] -= comz4;
	_lhs[1][4] += comz1;
	for (m = 0; m < 5; m++) lhsp(m,i,1,k) = _lhs[1][m];
	rhoq[0] = rhoq[1]; rhoq[1] = rhoq[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	for (m = 0; m < 3; m++) {
		_rhs[0][m] = rhs(m,i,0,k);
		_rhs[1][m] = rhs(m,i,1,k);
	}

	//---------------------------------------------------------------------
	//                          FORWARD ELIMINATION  
	//---------------------------------------------------------------------
	for (j = 0; j < ny-2; j++) {
		//---------------------------------------------------------------------
		// first fill the lhs for the u-eigenvalue                          
		//---------------------------------------------------------------------
		if (j+2 == ny-1) {
			_lhs[2][0] = lhsp(0,i,j+2,k) = 0.0;
			_lhs[2][1] = lhsp(1,i,j+2,k) = 0.0;
			_lhs[2][2] = lhsp(2,i,j+2,k) = 1.0;
			_lhs[2][3] = lhsp(3,i,j+2,k) = 0.0;
			_lhs[2][4] = lhsp(4,i,j+2,k) = 0.0;
		} else {
			fac1 = c3c4*rho_i(i,j+3,k);
			rhoq[2] = max(max(max(dy3+con43*fac1, dy5+c1c5*fac1), dymax+fac1), dy1);
			cv[2] = vs(i,j+3,k);
			_lhs[2][0] =  0.0;
			_lhs[2][1] = -dtty2*cv[0]-dtty1 * rhoq[0];
			_lhs[2][2] =  1.0 + c2dtty1 * rhoq[1];
			_lhs[2][3] =  dtty2*cv[2]-dtty1 * rhoq[2];
			_lhs[2][4] =  0.0;
			//---------------------------------------------------------------------
			//      add fourth order dissipation                             
			//---------------------------------------------------------------------
			if (j+2 == 2) {
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (j+2 >= 3 && j+2 < ny-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (j+2 == ny-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
			} else if (j+2 == ny-2) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz5;
			}

			//---------------------------------------------------------------------
			//      store computed lhs for later reuse
			//---------------------------------------------------------------------
			for (m = 0; m < 5; m++) lhsp(m,i,j+2,k) = _lhs[2][m];
			rhoq[0] = rhoq[1]; rhoq[1] = rhoq[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		//---------------------------------------------------------------------
		//      load rhs values for current iteration
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[2][m] = rhs(m,i,j+2,k);

		//---------------------------------------------------------------------
		//      perform current iteration
		//---------------------------------------------------------------------
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		lhs(3,i,j,k) = _lhs[0][3];
		lhs(4,i,j,k) = _lhs[0][4];
		for (m = 0; m < 5; m++) {
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
		for (m = 0; m < 3; m++) {
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//      The last two rows in this zone are a bit different, 
	//      since they do not have two more rows available for the
	//      elimination of off-diagonal entries
	//---------------------------------------------------------------------
	j = ny-2;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//            scale the last row immediately 
	//---------------------------------------------------------------------
	fac1 = 1.0/_lhs[1][2];
	for (m = 0; m < 3; m++) _rhs[1][m] *= fac1;
	lhs(3,i,ny-2,k) = _lhs[0][3];
	lhs(4,i,ny-2,k) = _lhs[0][4];

	//---------------------------------------------------------------------
	//      do the u+c and the u-c factors                 
	//---------------------------------------------------------------------
	for (j = 0; j < 3; j++) cv[j] = speed(i,j,k);
	for (m = 0; m < 5; m++) {
		_lhsp[0][m] = _lhs[0][m] = lhsp(m,i,0,k);
		_lhsp[1][m] = _lhs[1][m] = lhsp(m,i,1,k);
	}
	_lhsp[1][1] -= dtty2*cv[0];
	_lhsp[1][3] += dtty2*cv[2];
	_lhs[1][1] += dtty2*cv[0];
	_lhs[1][3] -= dtty2*cv[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	_rhs[0][3] = rhs(3,i,0,k);
	_rhs[0][4] = rhs(4,i,0,k);
	_rhs[1][3] = rhs(3,i,1,k);
	_rhs[1][4] = rhs(4,i,1,k);
	for (j = 0; j < ny-2; j++) {
		for (m = 0; m < 5; m++) {
			_lhsp[2][m] = _lhs[2][m] = lhsp(m,i,j+2,k);
		}
		_rhs[2][3] = rhs(3,i,j+2,k);
		_rhs[2][4] = rhs(4,i,j+2,k);
		if (j+2 < ny-1) {
			cv[2] = speed(i,j+3,k);
			_lhsp[2][1] -= dtty2*cv[0];
			_lhsp[2][3] += dtty2*cv[2];
			_lhs[2][1] += dtty2*cv[0];
			_lhs[2][3] -= dtty2*cv[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		fac1 = 1.0/_lhsp[0][2];
		m = 3;
		_lhsp[0][3] *= fac1;
		_lhsp[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhsp[1][2] -= _lhsp[1][1] * _lhsp[0][3];
		_lhsp[1][3] -= _lhsp[1][1] * _lhsp[0][4];
		_rhs[1][m] -= _lhsp[1][1] * _rhs[0][m];
		_lhsp[2][1] -= _lhsp[2][0] * _lhsp[0][3];
		_lhsp[2][2] -= _lhsp[2][0] * _lhsp[0][4];
		_rhs[2][m] -= _lhsp[2][0] * _rhs[0][m];

		m = 4;
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		_rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		_rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		for (m = 3; m < 5; m++) {
			lhsp(m,i,j,k) = _lhsp[0][m];
			lhsm(m,i,j,k) = _lhs[0][m];
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
		for (m = 0; m < 5; m++) {
			_lhsp[0][m] = _lhsp[1][m];
			_lhsp[1][m] = _lhsp[2][m];
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//         And again the last two rows separately
	//---------------------------------------------------------------------
	j = ny-2;
	m = 3;
	fac1 = 1.0/_lhsp[0][2];
	_lhsp[0][3] *= fac1;
	_lhsp[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhsp[1][2] -= _lhsp[1][1] * _lhsp[0][3];
	_lhsp[1][3] -= _lhsp[1][1] * _lhsp[0][4];
	_rhs[1][m] -= _lhsp[1][1] * _rhs[0][m];

	m = 4;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	_rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//               Scale the last row immediately 
	//---------------------------------------------------------------------
	_rhs[1][3] /= _lhsp[1][2];
	_rhs[1][4] /= _lhs[1][2];

	//---------------------------------------------------------------------
	//                         BACKSUBSTITUTION 
	//---------------------------------------------------------------------
	for (m = 0; m < 3; m++) _rhs[0][m] -= lhs(3,i,ny-2,k) * _rhs[1][m];
	_rhs[0][3] -= _lhsp[0][3] * _rhs[1][3];
	_rhs[0][4] -= _lhs[0][3] * _rhs[1][4];
	for (m = 0; m < 5; m++) {
		_rhs[2][m] = _rhs[1][m];
		_rhs[1][m] = _rhs[0][m];
	}
	for (j = ny-3; j >= 0; j--) {
		//---------------------------------------------------------------------
		//      The first three factors
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[0][m] = rtmp(m,i,j,k) - lhs(3,i,j,k)*_rhs[1][m] - lhs(4,i,j,k)*_rhs[2][m];
		//---------------------------------------------------------------------
		//      And the remaining two
		//---------------------------------------------------------------------
		_rhs[0][3] = rtmp(3,i,j,k) - lhsp(3,i,j,k)*_rhs[1][3] - lhsp(4,i,j,k)*_rhs[2][3];
		_rhs[0][4] = rtmp(4,i,j,k) - lhsm(3,i,j,k)*_rhs[1][4] - lhsm(4,i,j,k)*_rhs[2][4];
	
		if (j+2 < ny-1) {
			//---------------------------------------------------------------------
			//   block-diagonal matrix-vector multiplication                       
			//---------------------------------------------------------------------
			double r1 = _rhs[2][0];
			double r2 = _rhs[2][1];
			double r3 = _rhs[2][2];
			double r4 = _rhs[2][3];
			double r5 = _rhs[2][4];

			double t1 = bt * r1;
			double t2 = 0.5 * ( r4 + r5 );

			_rhs[2][0] =  bt * ( r4 - r5 );
			_rhs[2][1] = -r3;
			_rhs[2][2] =  r2;
			_rhs[2][3] = -t1 + t2;
			_rhs[2][4] =  t1 + t2;
		}

		for (m = 0; m < 5; m++) {
			rhs(m,i,j+2,k) = _rhs[2][m];
			_rhs[2][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[0][m];
		}
	}

	//---------------------------------------------------------------------
	//   block-diagonal matrix-vector multiplication                       
	//---------------------------------------------------------------------
	double t1 = bt * _rhs[2][0];
	double t2 = 0.5 * ( _rhs[2][3] + _rhs[2][4] );
	rhs(0,i,1,k) =  bt * ( _rhs[2][3] - _rhs[2][4] );
	rhs(1,i,1,k) = -_rhs[2][2];
	rhs(2,i,1,k) =  _rhs[2][1];
	rhs(3,i,1,k) = -t1 + t2;
	rhs(4,i,1,k) =  t1 + t2;

	for (m = 0; m < 5; m++) rhs(m,i,0,k) = _rhs[1][m];
}
#undef lhs
#undef lhsp
#undef lhsm
#undef rtmp

void SP::y_solve () {
	int xblock = min(SOLVE_BLOCK,nx);
	int xgrid = (nx+xblock-1)/xblock;
	int zblock = min(SOLVE_BLOCK/xblock,nz);
	int zgrid = (nz+zblock-1)/zblock;
	dim3 grid(zgrid,xgrid), block(zblock,xblock);

	START_TIMER(t_ysolve);
	y_solve_kernel<<<grid,block>>>(rho_i, vs, speed, rhs, lhs, rhstmp, nx, ny, nz);
	STOP_TIMER(t_ysolve);
}

//---------------------------------------------------------------------
// this function performs the solution of the approximate factorization
// step in the z-direction for all five matrix components
// simultaneously. The Thomas algorithm is employed to solve the
// systems for the z-lines. Boundary conditions are non-periodic
//---------------------------------------------------------------------
#define lhs(m,i,j,k) lhs[(i-1)+(nx-2)*((j-1)+(ny-2)*((k)+nz*(m-3)))]
#define lhsp(m,i,j,k) lhs[(i-1)+(nx-2)*((j-1)+(ny-2)*((k)+nz*(m+4)))]
#define lhsm(m,i,j,k) lhs[(i-1)+(nx-2)*((j-1)+(ny-2)*((k)+nz*(m-3+2)))]
#define rtmp(m,i,j,k) rhstmp[(i)+nx*((j)+ny*((k)+nz*(m)))]
__global__ static void z_solve_kernel (const double *rho_i, const double *us, const double *vs, const double *ws, const double *speed, const double *qs, const double *u, double *rhs, double *lhs, double *rhstmp, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double rhos[3], cv[3], _lhs[3][5], _lhsp[3][5], _rhs[3][5], fac1;

	j = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (j >= ny-1 || i >= nx-1) return;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	// Computes the left hand side for the three z-factors   
	//---------------------------------------------------------------------
	//---------------------------------------------------------------------
	//     zap the whole left hand side for starters
	//---------------------------------------------------------------------
	_lhs[0][0] = lhsp(0,i,j,0) = 0.0;
	_lhs[0][1] = lhsp(1,i,j,0) = 0.0;
	_lhs[0][2] = lhsp(2,i,j,0) = 1.0;
	_lhs[0][3] = lhsp(3,i,j,0) = 0.0;
	_lhs[0][4] = lhsp(4,i,j,0) = 0.0;

	//---------------------------------------------------------------------
	// first fill the lhs for the u-eigenvalue                          
	//---------------------------------------------------------------------
	for (k = 0; k < 3; k++) {
		fac1 = c3c4*rho_i(i,j,k);
		rhos[k] = max(max(max(dz4+con43*fac1, dz5+c1c5*fac1), dzmax+fac1), dz1);
		cv[k] = ws(i,j,k);
	}
	_lhs[1][0] =  0.0;
	_lhs[1][1] = -dttz2*cv[0] - dttz1*rhos[0];
	_lhs[1][2] =  1.0 + c2dttz1 * rhos[1];
	_lhs[1][3] =  dttz2*cv[2] - dttz1*rhos[2];
	_lhs[1][4]=  0.0;
	_lhs[1][2] += comz5;
	_lhs[1][3] -= comz4;
	_lhs[1][4] += comz1;
	for (m = 0; m < 5; m++) lhsp(m,i,j,1) = _lhs[1][m];
	rhos[0] = rhos[1]; rhos[1] = rhos[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	for (m = 0; m < 3; m++) {
		_rhs[0][m] = rhs(m,i,j,0);
		_rhs[1][m] = rhs(m,i,j,1);
	}

	//---------------------------------------------------------------------
	//                          FORWARD ELIMINATION  
	//---------------------------------------------------------------------
	for (k = 0; k < nz-2; k++) {
		//---------------------------------------------------------------------
		// first fill the lhs for the u-eigenvalue                          
		//---------------------------------------------------------------------
		if (k+2 == nz-1) {
			_lhs[2][0] = lhsp(0,i,j,k+2) = 0.0;
			_lhs[2][1] = lhsp(1,i,j,k+2) = 0.0;
			_lhs[2][2] = lhsp(2,i,j,k+2) = 1.0;
			_lhs[2][3] = lhsp(3,i,j,k+2) = 0.0;
			_lhs[2][4] = lhsp(4,i,j,k+2) = 0.0;
		} else {
			fac1 = c3c4*rho_i(i,j,k+3);
			rhos[2] = max(max(max(dz4+con43*fac1, dz5+c1c5*fac1), dzmax+fac1), dz1);
			cv[2] = ws(i,j,k+3);
			_lhs[2][0] =  0.0;
			_lhs[2][1] = -dttz2*cv[0] - dttz1*rhos[0];
			_lhs[2][2] =  1.0 + c2dttz1 * rhos[1];
			_lhs[2][3] =  dttz2*cv[2] - dttz1*rhos[2];
			_lhs[2][4] =  0.0;
			//---------------------------------------------------------------------
			//      add fourth order dissipation                                  
			//---------------------------------------------------------------------
			if (k+2 == 2) {
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (k+2 >= 3 && k+2 < nz-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
				_lhs[2][4] += comz1;
			} else if (k+2 == nz-3) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz6;
				_lhs[2][3] -= comz4;
			} else if (k+2 == nz-2) {
				_lhs[2][0] += comz1;
				_lhs[2][1] -= comz4;
				_lhs[2][2] += comz5;
			}

			//---------------------------------------------------------------------
			//      store computed lhs for later reuse
			//---------------------------------------------------------------------
			for (m = 0; m < 5; m++) lhsp(m,i,j,k+2) = _lhs[2][m];
			rhos[0] = rhos[1]; rhos[1] = rhos[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		//---------------------------------------------------------------------
		//      load rhs values for current iteration
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[2][m] = rhs(m,i,j,k+2);

		//---------------------------------------------------------------------
		//      perform current iteration
		//---------------------------------------------------------------------
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		for (m = 0; m < 3; m++) _rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		lhs(3,i,j,k) = _lhs[0][3];
		lhs(4,i,j,k) = _lhs[0][4];
		for (m = 0; m < 5; m++) {
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
		for (m = 0; m < 3; m++) {
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//      The last two rows in this zone are a bit different, 
	//      since they do not have two more rows available for the
	//      elimination of off-diagonal entries
	//---------------------------------------------------------------------
	k = nz-2;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	for (m = 0; m < 3; m++) _rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	for (m = 0; m < 3; m++) _rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//               scale the last row immediately
	//---------------------------------------------------------------------
	fac1 = 1.0/_lhs[1][2];
	for (m = 0; m < 3; m++) _rhs[1][m] *= fac1;
	lhs(3,i,j,k) = _lhs[0][3];
	lhs(4,i,j,k) = _lhs[0][4];

	//---------------------------------------------------------------------
	//      subsequently, fill the other factors (u+c), (u-c) 
	//---------------------------------------------------------------------
	for (k = 0; k < 3; k++) cv[k] = speed(i,j,k);
	for (m = 0; m < 5; m++) {
		_lhsp[0][m] = _lhs[0][m] = lhsp(m,i,j,0);
		_lhsp[1][m] = _lhs[1][m] = lhsp(m,i,j,1);
	}
	_lhsp[1][1] -= dttz2*cv[0];
	_lhsp[1][3] += dttz2*cv[2];
	_lhs[1][1] += dttz2*cv[0];
	_lhs[1][3] -= dttz2*cv[2];
	cv[0] = cv[1]; cv[1] = cv[2];
	_rhs[0][3] = rhs(3,i,j,0);
	_rhs[0][4] = rhs(4,i,j,0);
	_rhs[1][3] = rhs(3,i,j,1);
	_rhs[1][4] = rhs(4,i,j,1);
	//---------------------------------------------------------------------
	//      do the u+c and the u-c factors               
	//---------------------------------------------------------------------
	for (k = 0; k < nz-2; k++) {
		//---------------------------------------------------------------------
		//      first, fill the other factors (u+c), (u-c) 
		//---------------------------------------------------------------------
		for (m = 0; m < 5; m++) {
			_lhsp[2][m] = _lhs[2][m] = lhsp(m,i,j,k+2);
		}
		_rhs[2][3] = rhs(3,i,j,k+2);
		_rhs[2][4] = rhs(4,i,j,k+2);
		if (k+2 < nz-1) {
			cv[2] = speed(i,j,k+3);
			_lhsp[2][1] -= dttz2*cv[0];
			_lhsp[2][3] += dttz2*cv[2];
			_lhs[2][1] += dttz2*cv[0];
			_lhs[2][3] -= dttz2*cv[2];
			cv[0] = cv[1]; cv[1] = cv[2];
		}

		m = 3;
		fac1 = 1.0/_lhsp[0][2];
		_lhsp[0][3] *= fac1;
		_lhsp[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhsp[1][2] -= _lhsp[1][1] * _lhsp[0][3];
		_lhsp[1][3] -= _lhsp[1][1] * _lhsp[0][4];
		_rhs[1][m] -= _lhsp[1][1] * _rhs[0][m];
		_lhsp[2][1] -= _lhsp[2][0] * _lhsp[0][3];
		_lhsp[2][2] -= _lhsp[2][0] * _lhsp[0][4];
		_rhs[2][m] -= _lhsp[2][0] * _rhs[0][m];

		m = 4;
		fac1 = 1.0/_lhs[0][2];
		_lhs[0][3] *= fac1;
		_lhs[0][4] *= fac1;
		_rhs[0][m] *= fac1;
		_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
		_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
		_rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
		_lhs[2][1] -= _lhs[2][0] * _lhs[0][3];
		_lhs[2][2] -= _lhs[2][0] * _lhs[0][4];
		_rhs[2][m] -= _lhs[2][0] * _rhs[0][m];

		//---------------------------------------------------------------------
		//      store computed lhs and prepare data for next iteration
		//	rhs is stored in a temp array such that write accesses are coalesced
		//---------------------------------------------------------------------
		for (m = 3; m < 5; m++) {
			lhsp(m,i,j,k) = _lhsp[0][m];
			lhsm(m,i,j,k) = _lhs[0][m];
			rtmp(m,i,j,k) = _rhs[0][m];
			_rhs[0][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[2][m];
		}
		for (m = 0; m < 5; m++) {
			_lhsp[0][m] = _lhsp[1][m];
			_lhsp[1][m] = _lhsp[2][m];
			_lhs[0][m] = _lhs[1][m];
			_lhs[1][m] = _lhs[2][m];
		}
	}
	//---------------------------------------------------------------------
	//         And again the last two rows separately
	//---------------------------------------------------------------------
	k = nz-2;
	m = 3;
	fac1 = 1.0/_lhsp[0][2];
	_lhsp[0][3] *= fac1;
	_lhsp[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhsp[1][2] -= _lhsp[1][1] * _lhsp[0][3];
	_lhsp[1][3] -= _lhsp[1][1] * _lhsp[0][4];
	_rhs[1][m] -= _lhsp[1][1] * _rhs[0][m];

	m = 4;
	fac1 = 1.0/_lhs[0][2];
	_lhs[0][3] *= fac1;
	_lhs[0][4] *= fac1;
	_rhs[0][m] *= fac1;
	_lhs[1][2] -= _lhs[1][1] * _lhs[0][3];
	_lhs[1][3] -= _lhs[1][1] * _lhs[0][4];
	_rhs[1][m] -= _lhs[1][1] * _rhs[0][m];
	//---------------------------------------------------------------------
	//               Scale the last row immediately (some of this is overkill
	//               if this is the last cell)
	//---------------------------------------------------------------------
	_rhs[1][3] /= _lhsp[1][2];
	_rhs[1][4] /= _lhs[1][2];
		
	//---------------------------------------------------------------------
	//                         BACKSUBSTITUTION 
	//---------------------------------------------------------------------
	for (m = 0; m < 3; m++) _rhs[0][m] -= lhs(3,i,j,nz-2) * _rhs[1][m];
	_rhs[0][3] -= _lhsp[0][3] * _rhs[1][3];
	_rhs[0][4] -= _lhs[0][3] * _rhs[1][4];
	for (m = 0; m < 5; m++) {
		_rhs[2][m] = _rhs[1][m];
		_rhs[1][m] = _rhs[0][m];
	}
	
	for (k = nz-3; k >= 0; k--) {
		//---------------------------------------------------------------------
		//      The first three factors
		//---------------------------------------------------------------------
		for (m = 0; m < 3; m++) _rhs[0][m] = rtmp(m,i,j,k) - lhs(3,i,j,k)*_rhs[1][m] - lhs(4,i,j,k)*_rhs[2][m];
		//---------------------------------------------------------------------
		//      And the remaining two
		//---------------------------------------------------------------------
		_rhs[0][3] = rtmp(3,i,j,k) - lhsp(3,i,j,k)*_rhs[1][3] - lhsp(4,i,j,k)*_rhs[2][3];
		_rhs[0][4] = rtmp(4,i,j,k) - lhsm(3,i,j,k)*_rhs[1][4] - lhsm(4,i,j,k)*_rhs[2][4];

		if (k+2 < nz-1) {
			//---------------------------------------------------------------------
			//   block-diagonal matrix-vector multiplication (tzetar)
			//---------------------------------------------------------------------
			double xvel = us(i,j,k+2);
			double yvel = vs(i,j,k+2);
			double zvel = ws(i,j,k+2);
			double ac = speed(i,j,k+2);
			double uzik1 = u(0,i,j,k+2);
			double t1 = (bt*uzik1)/ac * (_rhs[2][3] + _rhs[2][4]);
			double t2 = _rhs[2][2] + t1;
			double t3 = bt*uzik1 * (_rhs[2][3] - _rhs[2][4]);

			_rhs[2][4] =  uzik1*(-xvel*_rhs[2][1] + yvel*_rhs[2][0]) + qs(i,j,k+2)*t2 + c2iv*(ac*ac)*t1 + zvel*t3;
			_rhs[2][3] =  zvel*t2  + t3;
			_rhs[2][2] =  uzik1*_rhs[2][0] + yvel*t2;
			_rhs[2][1] = -uzik1*_rhs[2][1] + xvel*t2;
			_rhs[2][0] = t2;
		}

		for (m = 0; m < 5; m++) {
			rhs(m,i,j,k+2) = _rhs[2][m];
			_rhs[2][m] = _rhs[1][m];
			_rhs[1][m] = _rhs[0][m];
		}
	}

	//---------------------------------------------------------------------
	//   block-diagonal matrix-vector multiplication (tzetar)
	//---------------------------------------------------------------------
	double xvel = us(i,j,1);
	double yvel = vs(i,j,1);
	double zvel = ws(i,j,1);
	double ac = speed(i,j,1);
	double uzik1 = u(0,i,j,1);
	double t1 = (bt*uzik1)/ac * (_rhs[2][3] + _rhs[2][4]);
	double t2 = _rhs[2][2] + t1;
	double t3 = bt*uzik1 * (_rhs[2][3] - _rhs[2][4]);

	rhs(4,i,j,1) =  uzik1*(-xvel*_rhs[2][1] + yvel*_rhs[2][0]) + qs(i,j,1)*t2 + c2iv*(ac*ac)*t1 + zvel*t3;
	rhs(3,i,j,1) =  zvel*t2  + t3;
	rhs(2,i,j,1) =  uzik1*_rhs[2][0] + yvel*t2;
	rhs(1,i,j,1) = -uzik1*_rhs[2][1] + xvel*t2;
	rhs(0,i,j,1) = t2;

	for (m = 0; m < 5; m++) rhs(m,i,j,0) = _rhs[1][m];
}
#undef lhs
#undef lhsp
#undef lhsm
#undef rtmp

void SP::z_solve () {
	int xblock = min(SOLVE_BLOCK,nx);
	int xgrid = (nx+xblock-1)/xblock;
	int yblock = min(SOLVE_BLOCK/xblock,ny);
	int ygrid = (ny+yblock-1)/yblock;
	dim3 grid(ygrid,xgrid), block(yblock,xblock);

	START_TIMER(t_zsolve);
	z_solve_kernel<<<grid,block>>>(rho_i, us, vs, ws, speed, qs, u, rhs, lhs, rhstmp, nx, ny, nz);
	STOP_TIMER(t_zsolve);
}

//---------------------------------------------------------------------
// this function returns the exact solution at point xi, eta, zeta  
//---------------------------------------------------------------------
__device__ static void exact_solution_kernel (const double xi, const double eta, const double zeta, double *dtemp) {
	using namespace gpu_mod;
	for (int m = 0; m < 5; m++)
		dtemp[m] = ce[0][m] + xi*(ce[1][m] + xi*(ce[4][m] + xi*(ce[7][m] + xi*ce[10][m]))) +
				eta*(ce[2][m] + eta*(ce[5][m] + eta*(ce[8][m] + eta*ce[11][m])))+
				zeta*(ce[3][m] + zeta*(ce[6][m] + zeta*(ce[9][m] + zeta*ce[12][m])));
}

//---------------------------------------------------------------------
// compute the right hand side based on exact solution
//---------------------------------------------------------------------
__global__ static void exact_rhs_kernel_init (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	k = blockIdx.y;
	j = blockIdx.x;
	i = threadIdx.x;
	for (m = 0; m < 5; m++) forcing(m,i,j,k) = 0.0;
}

__global__ static void exact_rhs_kernel_x (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, dtemp[5], dtpp;
	double ue[5][5], buf[3][5], cuf[3], q[3];

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	j = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (k >= nz-1 || j >= ny-1) return;

	using namespace gpu_mod;

	zeta = (double)k * dnzm1;
	eta = (double)j * dnym1;
	//---------------------------------------------------------------------
	//      xi-direction flux differences                      
	//---------------------------------------------------------------------
	for (i = 0; i < 3; i++) {
		xi = (double)i * dnxm1;
		exact_solution_kernel(xi, eta, zeta, dtemp);
		for (m = 0; m < 5; m++) ue[i+1][m] = dtemp[m];
		dtpp = 1.0/dtemp[0];
		for (m = 1; m < 5; m++) buf[i][m] = dtpp*dtemp[m];
		cuf[i] = buf[i][1] * buf[i][1];
		buf[i][0] = cuf[i] + buf[i][2] * buf[i][2] + buf[i][3] * buf[i][3];
		q[i] = 0.5 * (buf[i][1]*ue[i+1][1] + buf[i][2]*ue[i+1][2] + buf[i][3]*ue[i+1][3]);
	}
	for (i = 1; i < nx-1; i++) {
		if (i+2 < nx) {
			xi = (double)(i+2) * dnxm1;
			exact_solution_kernel(xi, eta, zeta, dtemp);
			for (m = 0; m < 5; m++) ue[4][m] = dtemp[m];
		}
		
		dtemp[0] = 0.0 - tx2*(ue[3][1]-ue[1][1])+ dx1tx1*(ue[3][0]-2.0*ue[2][0]+ue[1][0]);
		dtemp[1] = 0.0 - tx2*((ue[3][1]*buf[2][1]+c2*(ue[3][4]-q[2]))-(ue[1][1]*buf[0][1]+c2*(ue[1][4]-q[0])))+xxcon1*(buf[2][1]-2.0*buf[1][1]+buf[0][1])+dx2tx1*(ue[3][1]-2.0*ue[2][1]+ue[1][1]);
		dtemp[2] = 0.0 - tx2*(ue[3][2]*buf[2][1]-ue[1][2]*buf[0][1])+xxcon2*(buf[2][2]-2.0*buf[1][2]+buf[0][2])+dx3tx1*(ue[3][2]-2.0*ue[2][2]+ue[1][2]);
		dtemp[3] = 0.0 - tx2*(ue[3][3]*buf[2][1]-ue[1][3]*buf[0][1])+xxcon2*(buf[2][3]-2.0*buf[1][3]+buf[0][3])+dx4tx1*(ue[3][3]-2.0*ue[2][3]+ue[1][3]);
		dtemp[4] = 0.0 - tx2*(buf[2][1]*(c1*ue[3][4]-c2*q[2])-buf[0][1]*(c1*ue[1][4]-c2*q[0]))+0.5*xxcon3*(buf[2][0]-2.0*buf[1][0]+buf[0][0])+xxcon4*(cuf[2]-2.0*cuf[1]+cuf[0])+
					xxcon5*(buf[2][4]-2.0*buf[1][4]+buf[0][4])+dx5tx1*(ue[3][4]-2.0*ue[2][4]+ ue[1][4]);
		//---------------------------------------------------------------------
		//            Fourth-order dissipation                         
		//---------------------------------------------------------------------
		if (i == 1) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(5.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (i == 2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(-4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (i >= 3 && i < nx-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m]+6.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (i == nx-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m] +6.0*ue[2][m] - 4.0*ue[3][m]);
		} else if (i == nx-2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m] + 5.0*ue[2][m]);
		}

		for (m = 0; m < 5; m++) {
			ue[0][m] = ue[1][m]; 
			ue[1][m] = ue[2][m];
			ue[2][m] = ue[3][m];
			ue[3][m] = ue[4][m];
			buf[0][m] = buf[1][m];
			buf[1][m] = buf[2][m];
		}
		cuf[0] = cuf[1]; cuf[1] = cuf[2];
		q[0] = q[1]; q[1] = q[2];

		if (i < nx-2) {
			dtpp = 1.0/ue[3][0];
			for (m = 1; m < 5; m++) buf[2][m] = dtpp*ue[3][m];
			cuf[2] = buf[2][1] * buf[2][1];
			buf[2][0] = cuf[2] + buf[2][2] * buf[2][2] + buf[2][3] * buf[2][3];
			q[2] = 0.5 * (buf[2][1]*ue[3][1] + buf[2][2]*ue[3][2] + buf[2][3]*ue[3][3]);
		}
	}
}

__global__ static void exact_rhs_kernel_y (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, dtemp[5], dtpp;
	double ue[5][5], buf[3][5], cuf[3], q[3];

	k = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (k >= nz-1 || i >= nx-1) return;

	using namespace gpu_mod;

	zeta = (double)k * dnzm1;
	xi = (double)i * dnxm1;
	//---------------------------------------------------------------------
	//  eta-direction flux differences             
	//---------------------------------------------------------------------
	for (j = 0; j < 3; j++) {
		eta = (double)j * dnym1;
		exact_solution_kernel(xi, eta, zeta, dtemp);
		for (m = 0; m < 5; m++) ue[j+1][m] = dtemp[m];
		dtpp = 1.0/dtemp[0];
		for (m = 1; m < 5; m++) buf[j][m] = dtpp * dtemp[m];
		cuf[j] = buf[j][2] * buf[j][2];
		buf[j][0] = cuf[j] + buf[j][1] * buf[j][1] + buf[j][3] * buf[j][3];
		q[j] = 0.5*(buf[j][1]*ue[j+1][1] + buf[j][2]*ue[j+1][2] + buf[j][3]*ue[j+1][3]);
	}

	for (j = 1; j < ny-1; j++) {
		if (j+2 < ny) {
			eta = (double)(j+2) * dnym1;
			exact_solution_kernel(xi, eta, zeta, dtemp);
			for (m = 0; m < 5; m++) ue[4][m] = dtemp[m];
		}

		dtemp[0] = forcing(0,i,j,k) - ty2*(ue[3][2]-ue[1][2])+ dy1ty1*(ue[3][0]-2.0*ue[2][0]+ue[1][0]);
		dtemp[1] = forcing(1,i,j,k) - ty2*(ue[3][1]*buf[2][2]-ue[1][1]*buf[0][2])+yycon2*(buf[2][1]-2.0*buf[1][1]+buf[0][1])+dy2ty1*(ue[3][1]-2.0*ue[2][1]+ ue[1][1]);
		dtemp[2] = forcing(2,i,j,k) - ty2*((ue[3][2]*buf[2][2]+c2*(ue[3][4]-q[2]))-(ue[1][2]*buf[0][2]+c2*(ue[1][4]-q[0])))+yycon1*(buf[2][2]-2.0*buf[1][2]+buf[0][2])+dy3ty1*( ue[3][2]-2.0*ue[2][2] +ue[1][2]);
		dtemp[3] = forcing(3,i,j,k) - ty2*(ue[3][3]*buf[2][2]-ue[1][3]*buf[0][2])+yycon2*(buf[2][3]-2.0*buf[1][3]+buf[0][3])+dy4ty1*( ue[3][3]-2.0*ue[2][3]+ ue[1][3]);
		dtemp[4] = forcing(4,i,j,k) - ty2*(buf[2][2]*(c1*ue[3][4]-c2*q[2])-buf[0][2]*(c1*ue[1][4]-c2*q[0]))+0.5*yycon3*(buf[2][0]-2.0*buf[1][0]+buf[0][0])+yycon4*(cuf[2]-2.0*cuf[1]+cuf[0])+
					yycon5*(buf[2][4]-2.0*buf[1][4]+buf[0][4])+dy5ty1*(ue[3][4]-2.0*ue[2][4]+ue[1][4]);
		//---------------------------------------------------------------------
		//            Fourth-order dissipation                      
		//---------------------------------------------------------------------
		if (j == 1) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (5.0*ue[2][m] - 4.0*ue[3][m] +ue[4][m]);
		} else if (j == 2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (-4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m] +       ue[4][m]);
		} else if (j >= 3 && j < ny-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp*(ue[0][m] - 4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m] + ue[4][m]);
		} else if (j == ny-3) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (ue[0][m] - 4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m]);
		} else if (j == ny-2) {
			for (m = 0; m < 5; m++) forcing(m,i,j,k) = dtemp[m] - dssp * (ue[0][m] - 4.0*ue[1][m] + 5.0*ue[2][m]);
		}

		for (m = 0; m < 5; m++) {
			ue[0][m] = ue[1][m]; 
			ue[1][m] = ue[2][m];
			ue[2][m] = ue[3][m];
			ue[3][m] = ue[4][m];
			buf[0][m] = buf[1][m];
			buf[1][m] = buf[2][m];
		}
		cuf[0] = cuf[1]; cuf[1] = cuf[2];
		q[0] = q[1]; q[1] = q[2];

		if (j < ny-2) {
			dtpp = 1.0/ue[3][0];
			for (m = 1; m < 5; m++) buf[2][m] = dtpp * ue[3][m];
			cuf[2] = buf[2][2] * buf[2][2];
			buf[2][0] = cuf[2] + buf[2][1] * buf[2][1] + buf[2][3] * buf[2][3];
			q[2] = 0.5*(buf[2][1]*ue[3][1] + buf[2][2]*ue[3][2] + buf[2][3]*ue[3][3]);
		}
	}
}

__global__ static void exact_rhs_kernel_z (double *forcing, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, dtpp, dtemp[5];
	double ue[5][5], buf[3][5], cuf[3], q[3];

	j = blockIdx.x*blockDim.x+threadIdx.x+1;
	i = blockIdx.y*blockDim.y+threadIdx.y+1;
	if (j >= ny-1 || i >= nx-1) return;

	using namespace gpu_mod;

	eta = (double)j * dnym1;
	xi = (double)i * dnxm1;
	//---------------------------------------------------------------------
	//      zeta-direction flux differences                      
	//---------------------------------------------------------------------
	for (k = 0; k < 3; k++) {
		zeta = (double)k * dnzm1;
		exact_solution_kernel(xi, eta, zeta, dtemp);
		for (m = 0; m < 5; m++) ue[k+1][m] = dtemp[m];
		dtpp = 1.0/dtemp[0];
		for (m = 1; m < 5; m++) buf[k][m] = dtpp * dtemp[m];
		cuf[k] = buf[k][3] * buf[k][3];
		buf[k][0] = cuf[k] + buf[k][1] * buf[k][1] + buf[k][2] * buf[k][2];
		q[k] = 0.5*(buf[k][1]*ue[k+1][1] + buf[k][2]*ue[k+1][2] + buf[k][3]*ue[k+1][3]);
	}

	for (k = 1; k < nz-1; k++) {
		if (k+2 < nz) {
			zeta = (double)(k+2) * dnzm1;
			exact_solution_kernel(xi, eta, zeta, dtemp);
			for (m = 0; m < 5; m++) ue[4][m] = dtemp[m];
		}

		dtemp[0] = forcing(0,i,j,k) - tz2*(ue[3][3]-ue[1][3])+dz1tz1*(ue[3][0]-2.0*ue[2][0]+ue[1][0]);
		dtemp[1] = forcing(1,i,j,k) - tz2*(ue[3][1]*buf[2][3]-ue[1][1]*buf[0][3])+zzcon2*(buf[2][1]-2.0*buf[1][1]+buf[0][1])+dz2tz1*(ue[3][1]-2.0*ue[2][1]+ue[1][1]);
		dtemp[2] = forcing(2,i,j,k) - tz2*(ue[3][2]*buf[2][3]-ue[1][2]*buf[0][3])+zzcon2*(buf[2][2]-2.0*buf[1][2]+buf[0][2])+dz3tz1*(ue[3][2]-2.0*ue[2][2]+ue[1][2]);
		dtemp[3] = forcing(3,i,j,k) - tz2*((ue[3][3]*buf[2][3]+c2*(ue[3][4]-q[2]))-(ue[1][3]*buf[0][3]+c2*(ue[1][4]-q[0])))+zzcon1*(buf[2][3]-2.0*buf[1][3]+buf[0][3])+dz4tz1*(ue[3][3]-2.0*ue[2][3] +ue[1][3]);
		dtemp[4] = forcing(4,i,j,k) - tz2*(buf[2][3]*(c1*ue[3][4]-c2*q[2])-buf[0][3]*(c1*ue[1][4]-c2*q[0]))+0.5*zzcon3*(buf[2][0]-2.0*buf[1][0]+buf[0][0])+
					zzcon4*(cuf[2]-2.0*cuf[1]+cuf[0])+zzcon5*(buf[2][4]-2.0*buf[1][4]+buf[0][4])+dz5tz1*(ue[3][4]-2.0*ue[2][4]+ue[1][4]);
		//---------------------------------------------------------------------
		//            Fourth-order dissipation
		//---------------------------------------------------------------------
		if (k == 1) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(5.0*ue[2][m]-4.0*ue[3][m]+ue[4][m]);
		} else if (k == 2) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(-4.0*ue[1][m]+6.0*ue[2][m]-4.0*ue[3][m]+ue[4][m]);
		} else if (k >= 3 && k < nz-3) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(ue[0][m]-4.0*ue[1][m]+6.0*ue[2][m]-4.0*ue[3][m]+ue[4][m]);
		} else if (k == nz-3) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(ue[0][m]-4.0*ue[1][m] + 6.0*ue[2][m] - 4.0*ue[3][m]);
		} else if (k == nz-2) {
			for (m = 0; m < 5; m++) dtemp[m] = dtemp[m] - dssp*(ue[0][m]-4.0*ue[1][m]+5.0*ue[2][m]);
		}
		//---------------------------------------------------------------------
		// now change the sign of the forcing function, 
		//---------------------------------------------------------------------
		for (m = 0; m < 5; m++) forcing(m,i,j,k) = -1.0 * dtemp[m];

		for (m = 0; m < 5; m++) {
			ue[0][m] = ue[1][m]; 
			ue[1][m] = ue[2][m];
			ue[2][m] = ue[3][m];
			ue[3][m] = ue[4][m];
			buf[0][m] = buf[1][m];
			buf[1][m] = buf[2][m];
		}
		cuf[0] = cuf[1]; cuf[1] = cuf[2];
		q[0] = q[1]; q[1] = q[2];

		if (k < nz-2) {
			dtpp = 1.0/ue[3][0];
			for (m = 1; m < 5; m++) buf[2][m] = dtpp * ue[3][m];
			cuf[2] = buf[2][3] * buf[2][3];
			buf[2][0] = cuf[2] + buf[2][1] * buf[2][1] + buf[2][2] * buf[2][2];
			q[2] = 0.5*(buf[2][1]*ue[3][1] + buf[2][2]*ue[3][2] + buf[2][3]*ue[3][3]);
		}
	}
}

void SP::exact_rhs () {
	dim3 gridinit(ny,nz);
	exact_rhs_kernel_init<<<gridinit,nx>>>(forcing, nx, ny, nz);

	int yblock = min(ERHS_BLOCK,ny);
	int ygrid = (ny+yblock-1)/yblock;
	int zblock_y = min(ERHS_BLOCK/yblock,nz);
	int zgrid_y = (nz+zblock_y-1)/zblock_y;
	dim3 grid_x(zgrid_y,ygrid), block_x(zblock_y,yblock);
	exact_rhs_kernel_x<<<grid_x,block_x>>>(forcing, nx, ny, nz);

	int xblock = min(ERHS_BLOCK,nx);
	int xgrid = (nx+xblock-1)/xblock;
	int zblock_x = min(ERHS_BLOCK/xblock,nz);
	int zgrid_x = (nz+zblock_x-1)/zblock_x;
	dim3 grid_y(zgrid_x,xgrid), block_y(zblock_x,xblock);
	exact_rhs_kernel_y<<<grid_y,block_y>>>(forcing, nx, ny, nz);

	int yblock_x = min(ERHS_BLOCK/xblock,ny);
	int ygrid_x = (ny+yblock_x-1)/yblock_x;
	dim3 grid_z(ygrid_x,xgrid), block_z(yblock_x,xblock);
	exact_rhs_kernel_z<<<grid_z,block_z>>>(forcing, nx, ny, nz);
}

//---------------------------------------------------------------------
// This subroutine initializes the field variable u using 
// tri-linear transfinite interpolation of the boundary values     
//---------------------------------------------------------------------
__global__ static void initialize_kernel (double *u, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, temp[5];
	double Pface11[5], Pface12[5], Pface21[5], Pface22[5], Pface31[5], Pface32[5];

	k = blockIdx.x;
	j = blockIdx.y;
	i = threadIdx.x;

	using namespace gpu_mod;

	//---------------------------------------------------------------------
	//  Later (in compute_rhs) we compute 1/u for every element. A few of 
	//  the corner elements are not used, but it convenient (and faster) 
	//  to compute the whole thing with a simple loop. Make sure those 
	//  values are nonzero by initializing the whole thing here. 
	//---------------------------------------------------------------------
	u(0,i,j,k) = 1.0;
	u(1,i,j,k) = 0.0;
	u(2,i,j,k) = 0.0;
	u(3,i,j,k) = 0.0;
	u(4,i,j,k) = 1.0;

	//---------------------------------------------------------------------
	// first store the "interpolated" values everywhere on the zone    
	//---------------------------------------------------------------------
	zeta = (double)k * dnzm1;
	eta = (double)j * dnym1;
	xi = (double)i * dnxm1;
	exact_solution_kernel (0.0, eta, zeta, Pface11);
	exact_solution_kernel (1.0, eta, zeta, Pface12);
	exact_solution_kernel (xi, 0.0, zeta, Pface21);
	exact_solution_kernel (xi, 1.0, zeta, Pface22);
	exact_solution_kernel (xi, eta, 0.0, Pface31);
	exact_solution_kernel (xi, eta, 1.0, Pface32);
	for (m = 0; m < 5; m++) {
		double Pxi = xi * Pface12[m] + (1.0-xi)*Pface11[m];
		double Peta = eta * Pface22[m] + (1.0-eta)*Pface21[m];
		double Pzeta = zeta * Pface32[m] + (1.0-zeta)*Pface31[m];
		u(m,i,j,k) = Pxi + Peta + Pzeta - Pxi*Peta - Pxi*Pzeta - Peta*Pzeta + Pxi*Peta*Pzeta;
	}

	//---------------------------------------------------------------------
	// now store the exact values on the boundaries        
	//---------------------------------------------------------------------

	//---------------------------------------------------------------------
	// west face                                                  
	//---------------------------------------------------------------------
	xi = 0.0;
	if (i == 0) {
		zeta = (double)k * dnzm1;
		eta = (double)j * dnym1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// east face                                                      
	//---------------------------------------------------------------------
	xi = 1.0;
	if (i == nx-1) {
		zeta = (double)k * dnzm1;
		eta = (double)j * dnym1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// south face                                                 
	//---------------------------------------------------------------------
	eta = 0.0;
	if (j == 0) {
		zeta = (double)k * dnzm1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi,eta,zeta,temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// north face                                    
	//---------------------------------------------------------------------
	eta = 1.0;
	if (j == ny-1) {
		zeta = (double)k * dnzm1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi,eta,zeta,temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// bottom face                                       
	//---------------------------------------------------------------------
	zeta = 0.0;
	if (k == 0) {
		eta = (double)j * dnym1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
	//---------------------------------------------------------------------
	// top face     
	//---------------------------------------------------------------------
	zeta = 1.0;
	if (k == nz-1) {
		eta = (double)j * dnym1;
		xi = (double)i * dnxm1;
		exact_solution_kernel (xi, eta, zeta, temp);
		for (m = 0; m < 5; m++) u(m,i,j,k) = temp[m];
	}
}

void SP::initialize () {
	dim3 grid(nz,ny);
	initialize_kernel<<<grid,nx>>> (u, nx, ny, nz);
}

//---------------------------------------------------------------------
//     this function computes the norm of the difference between the
//     computed solution and the exact solution
//---------------------------------------------------------------------
__global__ static void error_norm_kernel (double *rms, const double *u, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double xi, eta, zeta, u_exact[5], rms_loc[5];

	j = blockIdx.x*blockDim.x+threadIdx.x;
	i = blockIdx.y*blockDim.y+threadIdx.y;
	if (j >= ny || i >= nx) return;

	using namespace gpu_mod;

	for (m = 0; m < 5; m++) rms_loc[m] = 0.0;

	xi = (double)i * dnxm1;
	eta = (double)j * dnym1;

	for (k = 0; k < nz; k++) {
		zeta = (double)k * dnzm1;
		exact_solution_kernel (xi, eta, zeta, u_exact);
		for (m = 0; m < 5; m++) {
			double add = u(m,i,j,k) - u_exact[m];
			rms_loc[m] += add*add;
		}
	}

	for (m = 0; m < 5; m++) rms[i+nx*(j+ny*m)] = rms_loc[m];
}

__global__ static void reduce_norm_kernel (double *rms, const int nx, const int ny, const int nz) {
	int i, m, maxpos, dist;
	__shared__ double buffer[NORM_BLOCK][5];

	i = threadIdx.x;
	for (m = 0; m < 5; m++) buffer[i][m] = 0.0;

	while (i < nx*ny) {
		for (m = 0; m < 5; m++) buffer[threadIdx.x][m] += rms[i+nx*ny*m];
		i += blockDim.x;
	}

	maxpos = blockDim.x;
	dist = (maxpos+1)/2;
	i = threadIdx.x;
	__syncthreads();
	while (maxpos > 1) {
		if (i < dist && i+dist < maxpos)
			for (m = 0; m < 5; m++) buffer[i][m] += buffer[i+dist][m];
		maxpos = dist;
		dist = (dist+1)/2;
		__syncthreads();
	}
	
	m = threadIdx.x;
	if (m < 5) rms[m] = sqrt(buffer[0][m]/((double)(nz-2)*(double)(ny-2)*(double)(nx-2)));
}

void SP::error_norm () {
	int xblock = min(64,nx);
	int xgrid = (nx+xblock-1)/xblock;
	int yblock = min(64/xblock,ny);
	int ygrid = (ny+yblock-1)/yblock;
	dim3 grid(ygrid,xgrid), block(yblock,xblock);

	error_norm_kernel<<<grid,block>>>(rmsbuf, u, nx, ny, nz);
	reduce_norm_kernel<<<1,NORM_BLOCK>>>(rmsbuf, nx, ny, nz);
	HANDLE_ERROR(hipMemcpy(xce, rmsbuf, 5*sizeof(double), hipMemcpyDeviceToHost));
}

__global__ static void rhs_norm_kernel (double *rms, const double *rhs, const int nx, const int ny, const int nz) {
	int i, j, k, m;
	double rms_loc[5];

	j = blockIdx.x*blockDim.x+threadIdx.x;
	i = blockIdx.y*blockDim.y+threadIdx.y;
	if (j >= ny || i >= nx) return;

	for (m = 0; m < 5; m++) rms_loc[m] = 0.0;
	if (i >= 1 && i < nx-1 && j >= 1 && j < ny-1) {
		for (k = 1; k < nz-1; k++) {
			for (int m = 0; m < 5; m++) {
				double add = rhs(m,i,j,k);
				rms_loc[m] += add*add;
			}
		}
	}

	for (m = 0; m < 5; m++) rms[i+nx*(j+ny*m)] = rms_loc[m];

}
void SP::rhs_norm () {
	int xblock = min(64,nx);
	int xgrid = (nx+xblock-1)/xblock;
	int yblock = min(64/xblock,ny);
	int ygrid = (ny+yblock-1)/yblock;
	dim3 grid(ygrid,xgrid), block(yblock,xblock);

	rhs_norm_kernel<<<grid,block>>>(rmsbuf, rhs, nx, ny, nz);
	reduce_norm_kernel<<<1,NORM_BLOCK>>>(rmsbuf, nx, ny, nz);
	HANDLE_ERROR(hipMemcpy(xcr, rmsbuf, 5*sizeof(double), hipMemcpyDeviceToHost));
}

void SP::set_constants() {

	double ce[13][5];
	ce[0][0] = 2.0;
	ce[1][0] = 0.0;
	ce[2][0] = 0.0;
	ce[3][0] = 4.0;
	ce[4][0] = 5.0;
	ce[5][0] = 3.0;
	ce[6][0] = 0.5;
	ce[7][0] = 0.02;
	ce[8][0] = 0.01;
	ce[9][0] = 0.03;
	ce[10][0] = 0.5;
	ce[11][0] = 0.4;
	ce[12][0] = 0.3;

	ce[0][1] = 1.0;
	ce[1][1] = 0.0;
	ce[2][1] = 0.0;
	ce[3][1] = 0.0;
	ce[4][1] = 1.0;
	ce[5][1] = 2.0;
	ce[6][1] = 3.0;
	ce[7][1] = 0.01;
	ce[8][1] = 0.03;
	ce[9][1] = 0.02;
	ce[10][1] = 0.4;
	ce[11][1] = 0.3;
	ce[12][1] = 0.5;

	ce[0][2] = 2.0;
	ce[1][2] = 2.0;
	ce[2][2] = 0.0;
	ce[3][2] = 0.0;
	ce[4][2] = 0.0;
	ce[5][2] = 2.0;
	ce[6][2] = 3.0;
	ce[7][2] = 0.04;
	ce[8][2] = 0.03;
	ce[9][2] = 0.05;
	ce[10][2] = 0.3;
	ce[11][2] = 0.5;
	ce[12][2] = 0.4;

	ce[0][3] = 2.0;
	ce[1][3] = 2.0;
	ce[2][3] = 0.0;
	ce[3][3] = 0.0;
	ce[4][3] = 0.0;
	ce[5][3] = 2.0;
	ce[6][3] = 3.0;
	ce[7][3] = 0.03;
	ce[8][3] = 0.05;
	ce[9][3] = 0.04;
	ce[10][3] = 0.2;
	ce[11][3] = 0.1;
	ce[12][3] = 0.3;

	ce[0][4] = 5.0;
	ce[1][4] = 4.0;
	ce[2][4] = 3.0;
	ce[3][4] = 2.0;
	ce[4][4] = 0.1;
	ce[5][4] = 0.4;
	ce[6][4] = 0.3;
	ce[7][4] = 0.05;
	ce[8][4] = 0.04;
	ce[9][4] = 0.03;
	ce[10][4] = 0.1;
	ce[11][4] = 0.3;
	ce[12][4] = 0.2;

	double bt = sqrt(0.5);

	double dnxm1 = 1.0/((double)nx-1.0);
	double dnym1 = 1.0/((double)ny-1.0);
	double dnzm1 = 1.0/((double)nz-1.0);

	double tx1 = 1.0 / (dnxm1 * dnxm1);
	double tx2 = 1.0 / (2.0 * dnxm1);
	double tx3 = 1.0 / dnxm1;

	double ty1 = 1.0 / (dnym1 * dnym1);
	double ty2 = 1.0 / (2.0 * dnym1);
	double ty3 = 1.0 / dnym1;
 
	double tz1 = 1.0 / (dnzm1 * dnzm1);
	double tz2 = 1.0 / (2.0 * dnzm1);
	double tz3 = 1.0 / dnzm1;

	double dttx1 = dt*tx1;
	double dttx2 = dt*tx2;
	double dtty1 = dt*ty1;
	double dtty2 = dt*ty2;
	double dttz1 = dt*tz1;
	double dttz2 = dt*tz2;

	double c2dttx1 = 2.0*dttx1;
	double c2dtty1 = 2.0*dtty1;
	double c2dttz1 = 2.0*dttz1;

	double dtdssp = dt*dssp;

	double comz1  = dtdssp;
	double comz4  = 4.0*dtdssp;
	double comz5  = 5.0*dtdssp;
	double comz6  = 6.0*dtdssp;

	double c3c4tx3 = c3c4*tx3;
	double c3c4ty3 = c3c4*ty3;
	double c3c4tz3 = c3c4*tz3;

	double dx1tx1 = dx1*tx1;
	double dx2tx1 = dx2*tx1;
	double dx3tx1 = dx3*tx1;
	double dx4tx1 = dx4*tx1;
	double dx5tx1 = dx5*tx1;

	double dy1ty1 = dy1*ty1;
	double dy2ty1 = dy2*ty1;
	double dy3ty1 = dy3*ty1;
	double dy4ty1 = dy4*ty1;
	double dy5ty1 = dy5*ty1;

	double dz1tz1 = dz1*tz1;
	double dz2tz1 = dz2*tz1;
	double dz3tz1 = dz3*tz1;
	double dz4tz1 = dz4*tz1;
	double dz5tz1 = dz5*tz1;

	double xxcon1 = c3c4tx3*con43*tx3;
	double xxcon2 = c3c4tx3*tx3;
	double xxcon3 = c3c4tx3*conz1*tx3;
	double xxcon4 = c3c4tx3*con16*tx3;
	double xxcon5 = c3c4tx3*c1c5*tx3;

	double yycon1 = c3c4ty3*con43*ty3;
	double yycon2 = c3c4ty3*ty3;
	double yycon3 = c3c4ty3*conz1*ty3;
	double yycon4 = c3c4ty3*con16*ty3;
	double yycon5 = c3c4ty3*c1c5*ty3;

	double zzcon1 = c3c4tz3*con43*tz3;
	double zzcon2 = c3c4tz3*tz3;
	double zzcon3 = c3c4tz3*conz1*tz3;
	double zzcon4 = c3c4tz3*con16*tz3;
	double zzcon5 = c3c4tz3*c1c5*tz3;

	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ce), &ce, 13*5*sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::bt), &bt, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dnxm1), &dnxm1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dnym1), &dnym1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dnzm1), &dnzm1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tx1), &tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tx2), &tx2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tx3), &tx3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ty1), &ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ty2), &ty2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::ty3), &ty3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tz1), &tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tz2), &tz2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::tz3), &tz3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dttx1), &dttx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dttx2), &dttx2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dtty1), &dtty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dtty2), &dtty2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dttz1), &dttz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dttz2), &dttz2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c2dttx1), &c2dttx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c2dtty1), &c2dtty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c2dttz1), &c2dttz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dt), &dt, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dtdssp), &dtdssp, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::comz1), &comz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::comz4), &comz4, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::comz5), &comz5, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::comz6), &comz6, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c3c4tx3), &c3c4tx3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c3c4ty3), &c3c4ty3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::c3c4tz3), &c3c4tz3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx1tx1), &dx1tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx2tx1), &dx2tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx3tx1), &dx3tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx4tx1), &dx4tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dx5tx1), &dx5tx1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy1ty1), &dy1ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy2ty1), &dy2ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy3ty1), &dy3ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy4ty1), &dy4ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dy5ty1), &dy5ty1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz1tz1), &dz1tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz2tz1), &dz2tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz3tz1), &dz3tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz4tz1), &dz4tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::dz5tz1), &dz5tz1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon1), &xxcon1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon2), &xxcon2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon3), &xxcon3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon4), &xxcon4, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::xxcon5), &xxcon5, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon1), &yycon1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon2), &yycon2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon3), &yycon3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon4), &yycon4, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::yycon5), &yycon5, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon1), &zzcon1, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon2), &zzcon2, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon3), &zzcon3, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon4), &zzcon4, sizeof(double)));
	HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(gpu_mod::zzcon5), &zzcon5, sizeof(double)));
}

void SP::allocate_device_memory() {
	int gridsize = nx*ny*nz;
	int facesize = max(max(nx*ny, nx*nz), ny*nz);

	HANDLE_ERROR(hipMalloc((void **)&u, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&forcing, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rhs, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rho_i, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&us, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&vs, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&ws, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&qs, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&speed, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&square, gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&lhs, 9*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rhstmp, 5*gridsize*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void **)&rmsbuf, 5*facesize*sizeof(double)));
}

void SP::free_device_memory() {
	HANDLE_ERROR(hipFree(u));
	HANDLE_ERROR(hipFree(forcing));
	HANDLE_ERROR(hipFree(rhs));
	HANDLE_ERROR(hipFree(rho_i));
	HANDLE_ERROR(hipFree(us));
	HANDLE_ERROR(hipFree(vs));
	HANDLE_ERROR(hipFree(ws));
	HANDLE_ERROR(hipFree(qs));
	HANDLE_ERROR(hipFree(speed));
	HANDLE_ERROR(hipFree(square));
	HANDLE_ERROR(hipFree(lhs));
	HANDLE_ERROR(hipFree(rhstmp));
	HANDLE_ERROR(hipFree(rmsbuf));
}

void SP::get_cuda_info() {
	int count;
	hipDeviceProp_t prop;

	HANDLE_ERROR(hipGetDeviceCount(&count));
	if (count == 0) {
		printf ("No CUDA devices found.\n");
		exit(EXIT_FAILURE);
	}

	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	strncpy (CUDAname, prop.name, 256);
	CUDAmp = prop.multiProcessorCount;
	CUDAclock = prop.clockRate;
	CUDAmem = prop.totalGlobalMem;
	CUDAmemclock = prop.memoryClockRate;
	CUDAl2cache = prop.l2CacheSize;
}
